#include "hip/hip_runtime.h"
#include "fdtd-2d_kernel.hu"
__global__ void kernel0(double *ey, double *hz, int ny, int tmax, int nx, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    for (int c1 = 16 * b0; c1 < nx - 1; c1 += 4096)
      if (nx >= t0 + c1 + 2)
        for (int c2 = 16 * b1; c2 < ny; c2 += 4096)
          if (ny >= t1 + c2 + 1)
            ey[(t0 + c1 + 1) * ny + (t1 + c2)] = (ey[(t0 + c1 + 1) * ny + (t1 + c2)] - (0.5 * (hz[(t0 + c1 + 1) * ny + (t1 + c2)] - hz[(t0 + c1) * ny + (t1 + c2)])));
}
__global__ void kernel1(double *ex, double *hz, int ny, int tmax, int nx, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    for (int c1 = 16 * b0; c1 < nx; c1 += 4096)
      if (nx >= t0 + c1 + 1)
        for (int c2 = 16 * b1; c2 < ny - 1; c2 += 4096)
          if (ny >= t1 + c2 + 2)
            ex[(t0 + c1) * ny + (t1 + c2 + 1)] = (ex[(t0 + c1) * ny + (t1 + c2 + 1)] - (0.5 * (hz[(t0 + c1) * ny + (t1 + c2 + 1)] - hz[(t0 + c1) * ny + (t1 + c2)])));
}
__global__ void kernel2(double *_fict_, double *ex, double *ey, double *hz, int ny, int tmax, int nx, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared__fict_[1];
    __shared__ double shared_ey[17][16];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      if (t0 == 0 && t1 == 0)
        shared__fict_[0] = _fict_[c0];
      __syncthreads();
      if (ny >= 16 * b1 + 2)
        for (int c1 = 16 * b0; c1 < nx - 1; c1 += 4096) {
          for (int c2 = 16 * b1; c2 < ny - 1; c2 += 4096) {
            if (ny >= t1 + c2 + 1)
              for (int c3 = t0; c3 <= ppcg_min(16, nx - c1 - 1); c3 += 16)
                shared_ey[c3][t1] = ey[(c1 + c3) * ny + (t1 + c2)];
            __syncthreads();
            if (b0 == 0 && t0 == 0 && c1 == 0 && ny >= t1 + c2 + 1)
              shared_ey[0][t1] = shared__fict_[0];
            if (nx >= t0 + c1 + 2 && ny >= t1 + c2 + 2)
              hz[(t0 + c1) * ny + (t1 + c2)] = (hz[(t0 + c1) * ny + (t1 + c2)] - (0.69999999999999996 * (((ex[(t0 + c1) * ny + (t1 + c2 + 1)] - ex[(t0 + c1) * ny + (t1 + c2)]) + shared_ey[t0 + 1][t1]) - shared_ey[t0][t1])));
            __syncthreads();
            if (b0 == 0 && t0 == 0 && c0 + 1 == tmax && c1 == 0 && ny >= t1 + c2 + 1)
              ey[0 * ny + (t1 + c2)] = shared_ey[0][t1];
          }
          if (b0 == 0 && c1 == 0 && (-ny + 16 * b1 + 1) % 4096 == 0) {
            if (t1 == 0)
              for (int c3 = t0; c3 <= ppcg_min(16, nx - 1); c3 += 16)
                shared_ey[c3][0] = ey[c3 * ny + (ny - 1)];
            __syncthreads();
            if (t0 == 0 && t1 == 0)
              shared_ey[0][0] = shared__fict_[0];
            __syncthreads();
            if (t0 == 0 && t1 == 0 && c0 + 1 == tmax)
              ey[0 * ny + (ny - 1)] = shared_ey[0][0];
          }
        }
      if (nx <= 1 && b0 == 0) {
        for (int c2 = 16 * b1; c2 < ny; c2 += 4096) {
          if (t0 == 0 && ny >= t1 + c2 + 1)
            shared_ey[0][t1] = ey[0 * ny + (t1 + c2)];
          __syncthreads();
          if (t0 == 0 && ny >= t1 + c2 + 1)
            shared_ey[0][t1] = shared__fict_[0];
          __syncthreads();
          if (t0 == 0 && c0 + 1 == tmax && ny >= t1 + c2 + 1)
            ey[0 * ny + (t1 + c2)] = shared_ey[0][t1];
        }
      } else if (nx >= 2 && b0 == 0 && 16 * b1 + 1 == ny) {
        if (t1 == 0)
          for (int c3 = t0; c3 <= ppcg_min(16, nx - 1); c3 += 16)
            shared_ey[c3][0] = ey[c3 * ny + (ny - 1)];
        __syncthreads();
        if (t0 == 0 && t1 == 0)
          shared_ey[0][0] = shared__fict_[0];
        __syncthreads();
        if (t0 == 0 && t1 == 0 && c0 + 1 == tmax)
          ey[0 * ny + (ny - 1)] = shared_ey[0][0];
      }
    }
}
