#include "hip/hip_runtime.h"
#include "fdtd-2d_kernel.hu"
__global__ void kernel0(double *ey, double *hz, int ny, int tmax, int nx, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 16 * b0; c1 < nx - 1; c1 += 4096)
      for (int c2 = 16 * b1; c2 < ny; c2 += 4096)
        for (int c3 = t0; c3 <= ppcg_min(15, nx - c1 - 2); c3 += 8)
          for (int c4 = t1; c4 <= ppcg_min(15, ny - c2 - 1); c4 += 8)
            ey[(c1 + c3 + 1) * ny + (c2 + c4)] = (ey[(c1 + c3 + 1) * ny + (c2 + c4)] - (0.5 * (hz[(c1 + c3 + 1) * ny + (c2 + c4)] - hz[(c1 + c3) * ny + (c2 + c4)])));
}
__global__ void kernel1(double *ex, double *hz, int ny, int tmax, int nx, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c1 = 16 * b0; c1 < nx; c1 += 4096)
      for (int c2 = 16 * b1; c2 < ny - 1; c2 += 4096)
        for (int c3 = t0; c3 <= ppcg_min(15, nx - c1 - 1); c3 += 8)
          for (int c4 = t1; c4 <= ppcg_min(15, ny - c2 - 2); c4 += 8)
            ex[(c1 + c3) * ny + (c2 + c4 + 1)] = (ex[(c1 + c3) * ny + (c2 + c4 + 1)] - (0.5 * (hz[(c1 + c3) * ny + (c2 + c4 + 1)] - hz[(c1 + c3) * ny + (c2 + c4)])));
}
__global__ void kernel2(double *_fict_, double *ex, double *ey, double *hz, int ny, int tmax, int nx, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared__fict_[1];
    __shared__ double shared_ey[17][16];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    {
      if (t0 == 0 && t1 == 0)
        shared__fict_[0] = _fict_[c0];
      __syncthreads();
      if (ny >= 16 * b1 + 2)
        for (int c1 = 16 * b0; c1 < nx - 1; c1 += 4096) {
          for (int c2 = 16 * b1; c2 < ny - 1; c2 += 4096) {
            for (int c3 = t0; c3 <= ppcg_min(16, nx - c1 - 1); c3 += 8)
              for (int c4 = t1; c4 <= ppcg_min(15, ny - c2 - 1); c4 += 8)
                shared_ey[c3][c4] = ey[(c1 + c3) * ny + (c2 + c4)];
            __syncthreads();
            if (ny >= t1 + c2 + 1)
              for (int c3 = t0; c3 <= ppcg_min(15, nx - c1 - 2); c3 += 8) {
                for (int c4 = t1; c4 <= ppcg_min(15, ny - c2 - 2); c4 += 8) {
                  if (b0 == 0 && t0 == 0 && c1 == 0 && c3 == 0)
                    shared_ey[0][c4] = shared__fict_[0];
                  hz[(c1 + c3) * ny + (c2 + c4)] = (hz[(c1 + c3) * ny + (c2 + c4)] - (0.69999999999999996 * (((ex[(c1 + c3) * ny + (c2 + c4 + 1)] - ex[(c1 + c3) * ny + (c2 + c4)]) + shared_ey[c3 + 1][c4]) - shared_ey[c3][c4])));
                }
                if (b0 == 0 && t0 == 0 && c1 == 0 && c2 + 16 >= ny && c3 == 0 && (-ny + t1 + 1) % 8 == 0)
                  shared_ey[0][ny - c2 - 1] = shared__fict_[0];
              }
            __syncthreads();
            if (b0 == 0 && t0 == 0 && c0 + 1 == tmax && c1 == 0)
              for (int c4 = t1; c4 <= ppcg_min(15, ny - c2 - 1); c4 += 8)
                ey[0 * ny + (c2 + c4)] = shared_ey[0][c4];
          }
          if (b0 == 0 && c1 == 0 && (-ny + 16 * b1 + 1) % 4096 == 0) {
            if (t1 == 0)
              for (int c3 = t0; c3 <= ppcg_min(16, nx - 1); c3 += 8)
                shared_ey[c3][0] = ey[c3 * ny + (ny - 1)];
            __syncthreads();
            if (t0 == 0 && t1 == 0)
              shared_ey[0][0] = shared__fict_[0];
            __syncthreads();
            if (t0 == 0 && t1 == 0 && c0 + 1 == tmax)
              ey[0 * ny + (ny - 1)] = shared_ey[0][0];
          }
        }
      if (nx <= 1 && b0 == 0) {
        for (int c2 = 16 * b1; c2 < ny; c2 += 4096) {
          if (t0 == 0)
            for (int c4 = t1; c4 <= ppcg_min(15, ny - c2 - 1); c4 += 8)
              shared_ey[0][c4] = ey[0 * ny + (c2 + c4)];
          __syncthreads();
          if (t0 == 0)
            for (int c4 = t1; c4 <= ppcg_min(15, ny - c2 - 1); c4 += 8)
              shared_ey[0][c4] = shared__fict_[0];
          __syncthreads();
          if (t0 == 0 && c0 + 1 == tmax)
            for (int c4 = t1; c4 <= ppcg_min(15, ny - c2 - 1); c4 += 8)
              ey[0 * ny + (c2 + c4)] = shared_ey[0][c4];
        }
      } else if (nx >= 2 && b0 == 0 && 16 * b1 + 1 == ny) {
        if (t1 == 0)
          for (int c3 = t0; c3 <= ppcg_min(16, nx - 1); c3 += 8)
            shared_ey[c3][0] = ey[c3 * ny + (ny - 1)];
        __syncthreads();
        if (t0 == 0 && t1 == 0)
          shared_ey[0][0] = shared__fict_[0];
        __syncthreads();
        if (t0 == 0 && t1 == 0 && c0 + 1 == tmax)
          ey[0 * ny + (ny - 1)] = shared_ey[0][0];
      }
    }
}
