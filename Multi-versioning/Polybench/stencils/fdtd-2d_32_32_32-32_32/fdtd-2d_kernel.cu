#include "hip/hip_runtime.h"
#include "fdtd-2d_kernel.hu"
__global__ void kernel0(double *ey, double *hz, int ny, int tmax, int nx, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    for (int c1 = 32 * b0; c1 < nx - 1; c1 += 8192)
      if (nx >= t0 + c1 + 2)
        for (int c2 = 32 * b1; c2 < ny; c2 += 8192)
          if (ny >= t1 + c2 + 1)
            ey[(t0 + c1 + 1) * ny + (t1 + c2)] = (ey[(t0 + c1 + 1) * ny + (t1 + c2)] - (0.5 * (hz[(t0 + c1 + 1) * ny + (t1 + c2)] - hz[(t0 + c1) * ny + (t1 + c2)])));
}
__global__ void kernel1(double *ex, double *hz, int ny, int tmax, int nx, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;

    for (int c1 = 32 * b0; c1 < nx; c1 += 8192)
      if (nx >= t0 + c1 + 1)
        for (int c2 = 32 * b1; c2 < ny - 1; c2 += 8192)
          if (ny >= t1 + c2 + 2)
            ex[(t0 + c1) * ny + (t1 + c2 + 1)] = (ex[(t0 + c1) * ny + (t1 + c2 + 1)] - (0.5 * (hz[(t0 + c1) * ny + (t1 + c2 + 1)] - hz[(t0 + c1) * ny + (t1 + c2)])));
}
__global__ void kernel2(double *_fict_, double *ex, double *ey, double *hz, int ny, int tmax, int nx, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared__fict_[1];

    {
      if (t0 == 0 && t1 == 0)
        shared__fict_[0] = _fict_[c0];
      __syncthreads();
      if (ny >= 32 * b1 + 2)
        for (int c1 = 32 * b0; c1 < nx - 1; c1 += 8192)
          if (nx >= t0 + c1 + 2) {
            for (int c2 = 32 * b1; c2 < ny - 1; c2 += 8192) {
              if (b0 == 0 && t0 == 0 && c1 == 0 && ny >= t1 + c2 + 1)
                ey[0 * ny + (t1 + c2)] = shared__fict_[0];
              if (ny >= t1 + c2 + 2)
                hz[(t0 + c1) * ny + (t1 + c2)] = (hz[(t0 + c1) * ny + (t1 + c2)] - (0.69999999999999996 * (((ex[(t0 + c1) * ny + (t1 + c2 + 1)] - ex[(t0 + c1) * ny + (t1 + c2)]) + ey[(t0 + c1 + 1) * ny + (t1 + c2)]) - ey[(t0 + c1) * ny + (t1 + c2)])));
            }
            if (b0 == 0 && t0 == 0 && t1 == 0 && c1 == 0 && (-ny + 32 * b1 + 1) % 8192 == 0)
              ey[0 * ny + (ny - 1)] = shared__fict_[0];
          }
      if (nx <= 1 && b0 == 0 && t0 == 0) {
        for (int c2 = 32 * b1; c2 < ny; c2 += 8192)
          if (ny >= t1 + c2 + 1)
            ey[0 * ny + (t1 + c2)] = shared__fict_[0];
      } else if (nx >= 2 && b0 == 0 && 32 * b1 + 1 == ny && t0 == 0 && t1 == 0) {
        ey[0 * ny + (ny - 1)] = shared__fict_[0];
      }
    }
}
