#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int nj, int nk, int ni)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_A[32][32];
    double private_C[2][2];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < ni; c0 += 8192)
      for (int c1 = 32 * b1; c1 < nj; c1 += 8192) {
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          private_C[0][0] = C[(t0 + c0) * nj + (t1 + c1)];
          if (nj >= t1 + c1 + 17)
            private_C[0][1] = C[(t0 + c0) * nj + (t1 + c1 + 16)];
          if (ni >= t0 + c0 + 17) {
            private_C[1][0] = C[(t0 + c0 + 16) * nj + (t1 + c1)];
            if (nj >= t1 + c1 + 17)
              private_C[1][1] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 16)];
          }
        }
        for (int c2 = 0; c2 < nk; c2 += 32) {
          for (int c3 = t0; c3 <= ppcg_min(31, ni - c0 - 1); c3 += 16)
            for (int c4 = t1; c4 <= ppcg_min(31, nk - c2 - 1); c4 += 16)
              shared_A[c3][c4] = A[(c0 + c3) * nk + (c2 + c4)];
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1 && c2 == 0) {
            private_C[0][0] *= beta;
            if (nj >= t1 + c1 + 17)
              private_C[0][1] *= beta;
            if (ni >= t0 + c0 + 17) {
              private_C[1][0] *= beta;
              if (nj >= t1 + c1 + 17)
                private_C[1][1] *= beta;
            }
          }
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1)
            for (int c3 = 0; c3 <= ppcg_min(31, nk - c2 - 1); c3 += 1) {
              private_C[0][0] += ((alpha * shared_A[t0][c3]) * B[(c2 + c3) * nj + (t1 + c1)]);
              if (nj >= t1 + c1 + 17)
                private_C[0][1] += ((alpha * shared_A[t0][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
              if (ni >= t0 + c0 + 17) {
                private_C[1][0] += ((alpha * shared_A[t0 + 16][c3]) * B[(c2 + c3) * nj + (t1 + c1)]);
                if (nj >= t1 + c1 + 17)
                  private_C[1][1] += ((alpha * shared_A[t0 + 16][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
              }
            }
          __syncthreads();
        }
        if (nk == 0) {
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
            private_C[0][0] *= beta;
            if (nj >= t1 + c1 + 17)
              private_C[0][1] *= beta;
            if (ni >= t0 + c0 + 17) {
              private_C[1][0] *= beta;
              if (nj >= t1 + c1 + 17)
                private_C[1][1] *= beta;
            }
          }
          __syncthreads();
        }
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          C[(t0 + c0) * nj + (t1 + c1)] = private_C[0][0];
          if (nj >= t1 + c1 + 17)
            C[(t0 + c0) * nj + (t1 + c1 + 16)] = private_C[0][1];
          if (ni >= t0 + c0 + 17) {
            C[(t0 + c0 + 16) * nj + (t1 + c1)] = private_C[1][0];
            if (nj >= t1 + c1 + 17)
              C[(t0 + c0 + 16) * nj + (t1 + c1 + 16)] = private_C[1][1];
          }
        }
        __syncthreads();
      }
}
