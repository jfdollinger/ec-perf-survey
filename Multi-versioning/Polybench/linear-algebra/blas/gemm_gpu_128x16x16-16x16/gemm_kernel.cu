#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int nj, int nk, int ni)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_B[16][16];
    double private_C[8][1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 128 * b0; c0 < ni; c0 += 32768)
      for (int c1 = 16 * b1; c1 < nj; c1 += 4096) {
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          private_C[0][0] = C[(t0 + c0) * nj + (t1 + c1)];
          if (ni >= t0 + c0 + 17) {
            private_C[1][0] = C[(t0 + c0 + 16) * nj + (t1 + c1)];
            if (ni >= t0 + c0 + 33) {
              private_C[2][0] = C[(t0 + c0 + 32) * nj + (t1 + c1)];
              if (ni >= t0 + c0 + 49) {
                private_C[3][0] = C[(t0 + c0 + 48) * nj + (t1 + c1)];
                if (ni >= t0 + c0 + 65) {
                  private_C[4][0] = C[(t0 + c0 + 64) * nj + (t1 + c1)];
                  if (ni >= t0 + c0 + 81) {
                    private_C[5][0] = C[(t0 + c0 + 80) * nj + (t1 + c1)];
                    if (ni >= t0 + c0 + 97) {
                      private_C[6][0] = C[(t0 + c0 + 96) * nj + (t1 + c1)];
                      if (ni >= t0 + c0 + 113)
                        private_C[7][0] = C[(t0 + c0 + 112) * nj + (t1 + c1)];
                    }
                  }
                }
              }
            }
          }
        }
        for (int c2 = 0; c2 < nk; c2 += 16) {
          if (nj >= t1 + c1 + 1 && nk >= t0 + c2 + 1)
            shared_B[t0][t1] = B[(t0 + c2) * nj + (t1 + c1)];
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1 && c2 == 0) {
            private_C[0][0] *= beta;
            if (ni >= t0 + c0 + 17) {
              private_C[1][0] *= beta;
              if (ni >= t0 + c0 + 33) {
                private_C[2][0] *= beta;
                if (ni >= t0 + c0 + 49) {
                  private_C[3][0] *= beta;
                  if (ni >= t0 + c0 + 65) {
                    private_C[4][0] *= beta;
                    if (ni >= t0 + c0 + 81) {
                      private_C[5][0] *= beta;
                      if (ni >= t0 + c0 + 97) {
                        private_C[6][0] *= beta;
                        if (ni >= t0 + c0 + 113)
                          private_C[7][0] *= beta;
                      }
                    }
                  }
                }
              }
            }
          }
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1)
            for (int c3 = 0; c3 <= ppcg_min(15, nk - c2 - 1); c3 += 1) {
              private_C[0][0] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * shared_B[c3][t1]);
              if (ni >= t0 + c0 + 17) {
                private_C[1][0] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                if (ni >= t0 + c0 + 33) {
                  private_C[2][0] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                  if (ni >= t0 + c0 + 49) {
                    private_C[3][0] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                    if (ni >= t0 + c0 + 65) {
                      private_C[4][0] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                      if (ni >= t0 + c0 + 81) {
                        private_C[5][0] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                        if (ni >= t0 + c0 + 97) {
                          private_C[6][0] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                          if (ni >= t0 + c0 + 113)
                            private_C[7][0] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                        }
                      }
                    }
                  }
                }
              }
            }
          __syncthreads();
        }
        if (nk == 0) {
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
            private_C[0][0] *= beta;
            if (ni >= t0 + c0 + 17) {
              private_C[1][0] *= beta;
              if (ni >= t0 + c0 + 33) {
                private_C[2][0] *= beta;
                if (ni >= t0 + c0 + 49) {
                  private_C[3][0] *= beta;
                  if (ni >= t0 + c0 + 65) {
                    private_C[4][0] *= beta;
                    if (ni >= t0 + c0 + 81) {
                      private_C[5][0] *= beta;
                      if (ni >= t0 + c0 + 97) {
                        private_C[6][0] *= beta;
                        if (ni >= t0 + c0 + 113)
                          private_C[7][0] *= beta;
                      }
                    }
                  }
                }
              }
            }
          }
          __syncthreads();
        }
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          C[(t0 + c0) * nj + (t1 + c1)] = private_C[0][0];
          if (ni >= t0 + c0 + 17) {
            C[(t0 + c0 + 16) * nj + (t1 + c1)] = private_C[1][0];
            if (ni >= t0 + c0 + 33) {
              C[(t0 + c0 + 32) * nj + (t1 + c1)] = private_C[2][0];
              if (ni >= t0 + c0 + 49) {
                C[(t0 + c0 + 48) * nj + (t1 + c1)] = private_C[3][0];
                if (ni >= t0 + c0 + 65) {
                  C[(t0 + c0 + 64) * nj + (t1 + c1)] = private_C[4][0];
                  if (ni >= t0 + c0 + 81) {
                    C[(t0 + c0 + 80) * nj + (t1 + c1)] = private_C[5][0];
                    if (ni >= t0 + c0 + 97) {
                      C[(t0 + c0 + 96) * nj + (t1 + c1)] = private_C[6][0];
                      if (ni >= t0 + c0 + 113)
                        C[(t0 + c0 + 112) * nj + (t1 + c1)] = private_C[7][0];
                    }
                  }
                }
              }
            }
          }
        }
        __syncthreads();
      }
}
