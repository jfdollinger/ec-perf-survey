#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int nj, int nk, int ni)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    double private_C[16][16];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 128 * b0; c0 < ni; c0 += 32768)
      for (int c1 = 128 * b1; c1 < nj; c1 += 32768) {
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          private_C[0][0] = C[(t0 + c0) * nj + (t1 + c1)];
          if (nj >= t1 + c1 + 9) {
            private_C[0][1] = C[(t0 + c0) * nj + (t1 + c1 + 8)];
            if (nj >= t1 + c1 + 17) {
              private_C[0][2] = C[(t0 + c0) * nj + (t1 + c1 + 16)];
              if (nj >= t1 + c1 + 25) {
                private_C[0][3] = C[(t0 + c0) * nj + (t1 + c1 + 24)];
                if (nj >= t1 + c1 + 33) {
                  private_C[0][4] = C[(t0 + c0) * nj + (t1 + c1 + 32)];
                  if (nj >= t1 + c1 + 41) {
                    private_C[0][5] = C[(t0 + c0) * nj + (t1 + c1 + 40)];
                    if (nj >= t1 + c1 + 49) {
                      private_C[0][6] = C[(t0 + c0) * nj + (t1 + c1 + 48)];
                      if (nj >= t1 + c1 + 57) {
                        private_C[0][7] = C[(t0 + c0) * nj + (t1 + c1 + 56)];
                        if (nj >= t1 + c1 + 65) {
                          private_C[0][8] = C[(t0 + c0) * nj + (t1 + c1 + 64)];
                          if (nj >= t1 + c1 + 73) {
                            private_C[0][9] = C[(t0 + c0) * nj + (t1 + c1 + 72)];
                            if (nj >= t1 + c1 + 81) {
                              private_C[0][10] = C[(t0 + c0) * nj + (t1 + c1 + 80)];
                              if (nj >= t1 + c1 + 89) {
                                private_C[0][11] = C[(t0 + c0) * nj + (t1 + c1 + 88)];
                                if (nj >= t1 + c1 + 97) {
                                  private_C[0][12] = C[(t0 + c0) * nj + (t1 + c1 + 96)];
                                  if (nj >= t1 + c1 + 105) {
                                    private_C[0][13] = C[(t0 + c0) * nj + (t1 + c1 + 104)];
                                    if (nj >= t1 + c1 + 113) {
                                      private_C[0][14] = C[(t0 + c0) * nj + (t1 + c1 + 112)];
                                      if (nj >= t1 + c1 + 121)
                                        private_C[0][15] = C[(t0 + c0) * nj + (t1 + c1 + 120)];
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          if (ni >= t0 + c0 + 9) {
            private_C[1][0] = C[(t0 + c0 + 8) * nj + (t1 + c1)];
            if (nj >= t1 + c1 + 9) {
              private_C[1][1] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 8)];
              if (nj >= t1 + c1 + 17) {
                private_C[1][2] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 16)];
                if (nj >= t1 + c1 + 25) {
                  private_C[1][3] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 24)];
                  if (nj >= t1 + c1 + 33) {
                    private_C[1][4] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 32)];
                    if (nj >= t1 + c1 + 41) {
                      private_C[1][5] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 40)];
                      if (nj >= t1 + c1 + 49) {
                        private_C[1][6] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 48)];
                        if (nj >= t1 + c1 + 57) {
                          private_C[1][7] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 56)];
                          if (nj >= t1 + c1 + 65) {
                            private_C[1][8] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 64)];
                            if (nj >= t1 + c1 + 73) {
                              private_C[1][9] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 72)];
                              if (nj >= t1 + c1 + 81) {
                                private_C[1][10] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 80)];
                                if (nj >= t1 + c1 + 89) {
                                  private_C[1][11] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 88)];
                                  if (nj >= t1 + c1 + 97) {
                                    private_C[1][12] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 96)];
                                    if (nj >= t1 + c1 + 105) {
                                      private_C[1][13] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 104)];
                                      if (nj >= t1 + c1 + 113) {
                                        private_C[1][14] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 112)];
                                        if (nj >= t1 + c1 + 121)
                                          private_C[1][15] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 120)];
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
            if (ni >= t0 + c0 + 17) {
              private_C[2][0] = C[(t0 + c0 + 16) * nj + (t1 + c1)];
              if (nj >= t1 + c1 + 9) {
                private_C[2][1] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 8)];
                if (nj >= t1 + c1 + 17) {
                  private_C[2][2] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 16)];
                  if (nj >= t1 + c1 + 25) {
                    private_C[2][3] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 24)];
                    if (nj >= t1 + c1 + 33) {
                      private_C[2][4] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 32)];
                      if (nj >= t1 + c1 + 41) {
                        private_C[2][5] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 40)];
                        if (nj >= t1 + c1 + 49) {
                          private_C[2][6] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 48)];
                          if (nj >= t1 + c1 + 57) {
                            private_C[2][7] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 56)];
                            if (nj >= t1 + c1 + 65) {
                              private_C[2][8] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 64)];
                              if (nj >= t1 + c1 + 73) {
                                private_C[2][9] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 72)];
                                if (nj >= t1 + c1 + 81) {
                                  private_C[2][10] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 80)];
                                  if (nj >= t1 + c1 + 89) {
                                    private_C[2][11] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 88)];
                                    if (nj >= t1 + c1 + 97) {
                                      private_C[2][12] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 96)];
                                      if (nj >= t1 + c1 + 105) {
                                        private_C[2][13] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 104)];
                                        if (nj >= t1 + c1 + 113) {
                                          private_C[2][14] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 112)];
                                          if (nj >= t1 + c1 + 121)
                                            private_C[2][15] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 120)];
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
              if (ni >= t0 + c0 + 25) {
                private_C[3][0] = C[(t0 + c0 + 24) * nj + (t1 + c1)];
                if (nj >= t1 + c1 + 9) {
                  private_C[3][1] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 8)];
                  if (nj >= t1 + c1 + 17) {
                    private_C[3][2] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 16)];
                    if (nj >= t1 + c1 + 25) {
                      private_C[3][3] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 24)];
                      if (nj >= t1 + c1 + 33) {
                        private_C[3][4] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 32)];
                        if (nj >= t1 + c1 + 41) {
                          private_C[3][5] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 40)];
                          if (nj >= t1 + c1 + 49) {
                            private_C[3][6] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 48)];
                            if (nj >= t1 + c1 + 57) {
                              private_C[3][7] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 56)];
                              if (nj >= t1 + c1 + 65) {
                                private_C[3][8] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 64)];
                                if (nj >= t1 + c1 + 73) {
                                  private_C[3][9] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 72)];
                                  if (nj >= t1 + c1 + 81) {
                                    private_C[3][10] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 80)];
                                    if (nj >= t1 + c1 + 89) {
                                      private_C[3][11] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 88)];
                                      if (nj >= t1 + c1 + 97) {
                                        private_C[3][12] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 96)];
                                        if (nj >= t1 + c1 + 105) {
                                          private_C[3][13] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 104)];
                                          if (nj >= t1 + c1 + 113) {
                                            private_C[3][14] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 112)];
                                            if (nj >= t1 + c1 + 121)
                                              private_C[3][15] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 120)];
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
                if (ni >= t0 + c0 + 33) {
                  private_C[4][0] = C[(t0 + c0 + 32) * nj + (t1 + c1)];
                  if (nj >= t1 + c1 + 9) {
                    private_C[4][1] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 8)];
                    if (nj >= t1 + c1 + 17) {
                      private_C[4][2] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 16)];
                      if (nj >= t1 + c1 + 25) {
                        private_C[4][3] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 24)];
                        if (nj >= t1 + c1 + 33) {
                          private_C[4][4] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 32)];
                          if (nj >= t1 + c1 + 41) {
                            private_C[4][5] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 40)];
                            if (nj >= t1 + c1 + 49) {
                              private_C[4][6] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 48)];
                              if (nj >= t1 + c1 + 57) {
                                private_C[4][7] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 56)];
                                if (nj >= t1 + c1 + 65) {
                                  private_C[4][8] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 64)];
                                  if (nj >= t1 + c1 + 73) {
                                    private_C[4][9] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 72)];
                                    if (nj >= t1 + c1 + 81) {
                                      private_C[4][10] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 80)];
                                      if (nj >= t1 + c1 + 89) {
                                        private_C[4][11] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 88)];
                                        if (nj >= t1 + c1 + 97) {
                                          private_C[4][12] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 96)];
                                          if (nj >= t1 + c1 + 105) {
                                            private_C[4][13] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 104)];
                                            if (nj >= t1 + c1 + 113) {
                                              private_C[4][14] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 112)];
                                              if (nj >= t1 + c1 + 121)
                                                private_C[4][15] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 120)];
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                  if (ni >= t0 + c0 + 41) {
                    private_C[5][0] = C[(t0 + c0 + 40) * nj + (t1 + c1)];
                    if (nj >= t1 + c1 + 9) {
                      private_C[5][1] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 8)];
                      if (nj >= t1 + c1 + 17) {
                        private_C[5][2] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 16)];
                        if (nj >= t1 + c1 + 25) {
                          private_C[5][3] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 24)];
                          if (nj >= t1 + c1 + 33) {
                            private_C[5][4] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 32)];
                            if (nj >= t1 + c1 + 41) {
                              private_C[5][5] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 40)];
                              if (nj >= t1 + c1 + 49) {
                                private_C[5][6] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 48)];
                                if (nj >= t1 + c1 + 57) {
                                  private_C[5][7] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 56)];
                                  if (nj >= t1 + c1 + 65) {
                                    private_C[5][8] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 64)];
                                    if (nj >= t1 + c1 + 73) {
                                      private_C[5][9] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 72)];
                                      if (nj >= t1 + c1 + 81) {
                                        private_C[5][10] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 80)];
                                        if (nj >= t1 + c1 + 89) {
                                          private_C[5][11] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 88)];
                                          if (nj >= t1 + c1 + 97) {
                                            private_C[5][12] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 96)];
                                            if (nj >= t1 + c1 + 105) {
                                              private_C[5][13] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 104)];
                                              if (nj >= t1 + c1 + 113) {
                                                private_C[5][14] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 112)];
                                                if (nj >= t1 + c1 + 121)
                                                  private_C[5][15] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 120)];
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                    if (ni >= t0 + c0 + 49) {
                      private_C[6][0] = C[(t0 + c0 + 48) * nj + (t1 + c1)];
                      if (nj >= t1 + c1 + 9) {
                        private_C[6][1] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 8)];
                        if (nj >= t1 + c1 + 17) {
                          private_C[6][2] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 16)];
                          if (nj >= t1 + c1 + 25) {
                            private_C[6][3] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 24)];
                            if (nj >= t1 + c1 + 33) {
                              private_C[6][4] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 32)];
                              if (nj >= t1 + c1 + 41) {
                                private_C[6][5] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 40)];
                                if (nj >= t1 + c1 + 49) {
                                  private_C[6][6] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 48)];
                                  if (nj >= t1 + c1 + 57) {
                                    private_C[6][7] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 56)];
                                    if (nj >= t1 + c1 + 65) {
                                      private_C[6][8] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 64)];
                                      if (nj >= t1 + c1 + 73) {
                                        private_C[6][9] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 72)];
                                        if (nj >= t1 + c1 + 81) {
                                          private_C[6][10] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 80)];
                                          if (nj >= t1 + c1 + 89) {
                                            private_C[6][11] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 88)];
                                            if (nj >= t1 + c1 + 97) {
                                              private_C[6][12] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 96)];
                                              if (nj >= t1 + c1 + 105) {
                                                private_C[6][13] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 104)];
                                                if (nj >= t1 + c1 + 113) {
                                                  private_C[6][14] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 112)];
                                                  if (nj >= t1 + c1 + 121)
                                                    private_C[6][15] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 120)];
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                      if (ni >= t0 + c0 + 57) {
                        private_C[7][0] = C[(t0 + c0 + 56) * nj + (t1 + c1)];
                        if (nj >= t1 + c1 + 9) {
                          private_C[7][1] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 8)];
                          if (nj >= t1 + c1 + 17) {
                            private_C[7][2] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 16)];
                            if (nj >= t1 + c1 + 25) {
                              private_C[7][3] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 24)];
                              if (nj >= t1 + c1 + 33) {
                                private_C[7][4] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 32)];
                                if (nj >= t1 + c1 + 41) {
                                  private_C[7][5] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 40)];
                                  if (nj >= t1 + c1 + 49) {
                                    private_C[7][6] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 48)];
                                    if (nj >= t1 + c1 + 57) {
                                      private_C[7][7] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 56)];
                                      if (nj >= t1 + c1 + 65) {
                                        private_C[7][8] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 64)];
                                        if (nj >= t1 + c1 + 73) {
                                          private_C[7][9] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 72)];
                                          if (nj >= t1 + c1 + 81) {
                                            private_C[7][10] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 80)];
                                            if (nj >= t1 + c1 + 89) {
                                              private_C[7][11] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 88)];
                                              if (nj >= t1 + c1 + 97) {
                                                private_C[7][12] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 96)];
                                                if (nj >= t1 + c1 + 105) {
                                                  private_C[7][13] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 104)];
                                                  if (nj >= t1 + c1 + 113) {
                                                    private_C[7][14] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 112)];
                                                    if (nj >= t1 + c1 + 121)
                                                      private_C[7][15] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 120)];
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                        if (ni >= t0 + c0 + 65) {
                          private_C[8][0] = C[(t0 + c0 + 64) * nj + (t1 + c1)];
                          if (nj >= t1 + c1 + 9) {
                            private_C[8][1] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 8)];
                            if (nj >= t1 + c1 + 17) {
                              private_C[8][2] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 16)];
                              if (nj >= t1 + c1 + 25) {
                                private_C[8][3] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 24)];
                                if (nj >= t1 + c1 + 33) {
                                  private_C[8][4] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 32)];
                                  if (nj >= t1 + c1 + 41) {
                                    private_C[8][5] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 40)];
                                    if (nj >= t1 + c1 + 49) {
                                      private_C[8][6] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 48)];
                                      if (nj >= t1 + c1 + 57) {
                                        private_C[8][7] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 56)];
                                        if (nj >= t1 + c1 + 65) {
                                          private_C[8][8] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 64)];
                                          if (nj >= t1 + c1 + 73) {
                                            private_C[8][9] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 72)];
                                            if (nj >= t1 + c1 + 81) {
                                              private_C[8][10] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 80)];
                                              if (nj >= t1 + c1 + 89) {
                                                private_C[8][11] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 88)];
                                                if (nj >= t1 + c1 + 97) {
                                                  private_C[8][12] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 96)];
                                                  if (nj >= t1 + c1 + 105) {
                                                    private_C[8][13] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 104)];
                                                    if (nj >= t1 + c1 + 113) {
                                                      private_C[8][14] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 112)];
                                                      if (nj >= t1 + c1 + 121)
                                                        private_C[8][15] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 120)];
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                          if (ni >= t0 + c0 + 73) {
                            private_C[9][0] = C[(t0 + c0 + 72) * nj + (t1 + c1)];
                            if (nj >= t1 + c1 + 9) {
                              private_C[9][1] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 8)];
                              if (nj >= t1 + c1 + 17) {
                                private_C[9][2] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 16)];
                                if (nj >= t1 + c1 + 25) {
                                  private_C[9][3] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 24)];
                                  if (nj >= t1 + c1 + 33) {
                                    private_C[9][4] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 32)];
                                    if (nj >= t1 + c1 + 41) {
                                      private_C[9][5] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 40)];
                                      if (nj >= t1 + c1 + 49) {
                                        private_C[9][6] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 48)];
                                        if (nj >= t1 + c1 + 57) {
                                          private_C[9][7] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 56)];
                                          if (nj >= t1 + c1 + 65) {
                                            private_C[9][8] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 64)];
                                            if (nj >= t1 + c1 + 73) {
                                              private_C[9][9] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 72)];
                                              if (nj >= t1 + c1 + 81) {
                                                private_C[9][10] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 80)];
                                                if (nj >= t1 + c1 + 89) {
                                                  private_C[9][11] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 88)];
                                                  if (nj >= t1 + c1 + 97) {
                                                    private_C[9][12] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 96)];
                                                    if (nj >= t1 + c1 + 105) {
                                                      private_C[9][13] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 104)];
                                                      if (nj >= t1 + c1 + 113) {
                                                        private_C[9][14] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 112)];
                                                        if (nj >= t1 + c1 + 121)
                                                          private_C[9][15] = C[(t0 + c0 + 72) * nj + (t1 + c1 + 120)];
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                            if (ni >= t0 + c0 + 81) {
                              private_C[10][0] = C[(t0 + c0 + 80) * nj + (t1 + c1)];
                              if (nj >= t1 + c1 + 9) {
                                private_C[10][1] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 8)];
                                if (nj >= t1 + c1 + 17) {
                                  private_C[10][2] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 16)];
                                  if (nj >= t1 + c1 + 25) {
                                    private_C[10][3] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 24)];
                                    if (nj >= t1 + c1 + 33) {
                                      private_C[10][4] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 32)];
                                      if (nj >= t1 + c1 + 41) {
                                        private_C[10][5] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 40)];
                                        if (nj >= t1 + c1 + 49) {
                                          private_C[10][6] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 48)];
                                          if (nj >= t1 + c1 + 57) {
                                            private_C[10][7] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 56)];
                                            if (nj >= t1 + c1 + 65) {
                                              private_C[10][8] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 64)];
                                              if (nj >= t1 + c1 + 73) {
                                                private_C[10][9] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 72)];
                                                if (nj >= t1 + c1 + 81) {
                                                  private_C[10][10] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 80)];
                                                  if (nj >= t1 + c1 + 89) {
                                                    private_C[10][11] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 88)];
                                                    if (nj >= t1 + c1 + 97) {
                                                      private_C[10][12] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 96)];
                                                      if (nj >= t1 + c1 + 105) {
                                                        private_C[10][13] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 104)];
                                                        if (nj >= t1 + c1 + 113) {
                                                          private_C[10][14] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 112)];
                                                          if (nj >= t1 + c1 + 121)
                                                            private_C[10][15] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 120)];
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                              if (ni >= t0 + c0 + 89) {
                                private_C[11][0] = C[(t0 + c0 + 88) * nj + (t1 + c1)];
                                if (nj >= t1 + c1 + 9) {
                                  private_C[11][1] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 8)];
                                  if (nj >= t1 + c1 + 17) {
                                    private_C[11][2] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 16)];
                                    if (nj >= t1 + c1 + 25) {
                                      private_C[11][3] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 24)];
                                      if (nj >= t1 + c1 + 33) {
                                        private_C[11][4] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 32)];
                                        if (nj >= t1 + c1 + 41) {
                                          private_C[11][5] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 40)];
                                          if (nj >= t1 + c1 + 49) {
                                            private_C[11][6] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 48)];
                                            if (nj >= t1 + c1 + 57) {
                                              private_C[11][7] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 56)];
                                              if (nj >= t1 + c1 + 65) {
                                                private_C[11][8] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 64)];
                                                if (nj >= t1 + c1 + 73) {
                                                  private_C[11][9] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 72)];
                                                  if (nj >= t1 + c1 + 81) {
                                                    private_C[11][10] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 80)];
                                                    if (nj >= t1 + c1 + 89) {
                                                      private_C[11][11] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 88)];
                                                      if (nj >= t1 + c1 + 97) {
                                                        private_C[11][12] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 96)];
                                                        if (nj >= t1 + c1 + 105) {
                                                          private_C[11][13] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 104)];
                                                          if (nj >= t1 + c1 + 113) {
                                                            private_C[11][14] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 112)];
                                                            if (nj >= t1 + c1 + 121)
                                                              private_C[11][15] = C[(t0 + c0 + 88) * nj + (t1 + c1 + 120)];
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                                if (ni >= t0 + c0 + 97) {
                                  private_C[12][0] = C[(t0 + c0 + 96) * nj + (t1 + c1)];
                                  if (nj >= t1 + c1 + 9) {
                                    private_C[12][1] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 8)];
                                    if (nj >= t1 + c1 + 17) {
                                      private_C[12][2] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 16)];
                                      if (nj >= t1 + c1 + 25) {
                                        private_C[12][3] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 24)];
                                        if (nj >= t1 + c1 + 33) {
                                          private_C[12][4] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 32)];
                                          if (nj >= t1 + c1 + 41) {
                                            private_C[12][5] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 40)];
                                            if (nj >= t1 + c1 + 49) {
                                              private_C[12][6] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 48)];
                                              if (nj >= t1 + c1 + 57) {
                                                private_C[12][7] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 56)];
                                                if (nj >= t1 + c1 + 65) {
                                                  private_C[12][8] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 64)];
                                                  if (nj >= t1 + c1 + 73) {
                                                    private_C[12][9] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 72)];
                                                    if (nj >= t1 + c1 + 81) {
                                                      private_C[12][10] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 80)];
                                                      if (nj >= t1 + c1 + 89) {
                                                        private_C[12][11] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 88)];
                                                        if (nj >= t1 + c1 + 97) {
                                                          private_C[12][12] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 96)];
                                                          if (nj >= t1 + c1 + 105) {
                                                            private_C[12][13] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 104)];
                                                            if (nj >= t1 + c1 + 113) {
                                                              private_C[12][14] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 112)];
                                                              if (nj >= t1 + c1 + 121)
                                                                private_C[12][15] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 120)];
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                  if (ni >= t0 + c0 + 105) {
                                    private_C[13][0] = C[(t0 + c0 + 104) * nj + (t1 + c1)];
                                    if (nj >= t1 + c1 + 9) {
                                      private_C[13][1] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 8)];
                                      if (nj >= t1 + c1 + 17) {
                                        private_C[13][2] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 16)];
                                        if (nj >= t1 + c1 + 25) {
                                          private_C[13][3] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 24)];
                                          if (nj >= t1 + c1 + 33) {
                                            private_C[13][4] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 32)];
                                            if (nj >= t1 + c1 + 41) {
                                              private_C[13][5] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 40)];
                                              if (nj >= t1 + c1 + 49) {
                                                private_C[13][6] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 48)];
                                                if (nj >= t1 + c1 + 57) {
                                                  private_C[13][7] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 56)];
                                                  if (nj >= t1 + c1 + 65) {
                                                    private_C[13][8] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 64)];
                                                    if (nj >= t1 + c1 + 73) {
                                                      private_C[13][9] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 72)];
                                                      if (nj >= t1 + c1 + 81) {
                                                        private_C[13][10] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 80)];
                                                        if (nj >= t1 + c1 + 89) {
                                                          private_C[13][11] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 88)];
                                                          if (nj >= t1 + c1 + 97) {
                                                            private_C[13][12] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 96)];
                                                            if (nj >= t1 + c1 + 105) {
                                                              private_C[13][13] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 104)];
                                                              if (nj >= t1 + c1 + 113) {
                                                                private_C[13][14] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 112)];
                                                                if (nj >= t1 + c1 + 121)
                                                                  private_C[13][15] = C[(t0 + c0 + 104) * nj + (t1 + c1 + 120)];
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                    if (ni >= t0 + c0 + 113) {
                                      private_C[14][0] = C[(t0 + c0 + 112) * nj + (t1 + c1)];
                                      if (nj >= t1 + c1 + 9) {
                                        private_C[14][1] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 8)];
                                        if (nj >= t1 + c1 + 17) {
                                          private_C[14][2] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 16)];
                                          if (nj >= t1 + c1 + 25) {
                                            private_C[14][3] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 24)];
                                            if (nj >= t1 + c1 + 33) {
                                              private_C[14][4] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 32)];
                                              if (nj >= t1 + c1 + 41) {
                                                private_C[14][5] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 40)];
                                                if (nj >= t1 + c1 + 49) {
                                                  private_C[14][6] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 48)];
                                                  if (nj >= t1 + c1 + 57) {
                                                    private_C[14][7] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 56)];
                                                    if (nj >= t1 + c1 + 65) {
                                                      private_C[14][8] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 64)];
                                                      if (nj >= t1 + c1 + 73) {
                                                        private_C[14][9] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 72)];
                                                        if (nj >= t1 + c1 + 81) {
                                                          private_C[14][10] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 80)];
                                                          if (nj >= t1 + c1 + 89) {
                                                            private_C[14][11] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 88)];
                                                            if (nj >= t1 + c1 + 97) {
                                                              private_C[14][12] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 96)];
                                                              if (nj >= t1 + c1 + 105) {
                                                                private_C[14][13] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 104)];
                                                                if (nj >= t1 + c1 + 113) {
                                                                  private_C[14][14] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 112)];
                                                                  if (nj >= t1 + c1 + 121)
                                                                    private_C[14][15] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 120)];
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                      if (ni >= t0 + c0 + 121) {
                                        private_C[15][0] = C[(t0 + c0 + 120) * nj + (t1 + c1)];
                                        if (nj >= t1 + c1 + 9) {
                                          private_C[15][1] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 8)];
                                          if (nj >= t1 + c1 + 17) {
                                            private_C[15][2] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 16)];
                                            if (nj >= t1 + c1 + 25) {
                                              private_C[15][3] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 24)];
                                              if (nj >= t1 + c1 + 33) {
                                                private_C[15][4] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 32)];
                                                if (nj >= t1 + c1 + 41) {
                                                  private_C[15][5] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 40)];
                                                  if (nj >= t1 + c1 + 49) {
                                                    private_C[15][6] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 48)];
                                                    if (nj >= t1 + c1 + 57) {
                                                      private_C[15][7] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 56)];
                                                      if (nj >= t1 + c1 + 65) {
                                                        private_C[15][8] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 64)];
                                                        if (nj >= t1 + c1 + 73) {
                                                          private_C[15][9] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 72)];
                                                          if (nj >= t1 + c1 + 81) {
                                                            private_C[15][10] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 80)];
                                                            if (nj >= t1 + c1 + 89) {
                                                              private_C[15][11] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 88)];
                                                              if (nj >= t1 + c1 + 97) {
                                                                private_C[15][12] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 96)];
                                                                if (nj >= t1 + c1 + 105) {
                                                                  private_C[15][13] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 104)];
                                                                  if (nj >= t1 + c1 + 113) {
                                                                    private_C[15][14] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 112)];
                                                                    if (nj >= t1 + c1 + 121)
                                                                      private_C[15][15] = C[(t0 + c0 + 120) * nj + (t1 + c1 + 120)];
                                                                  }
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          private_C[0][0] *= beta;
          if (nj >= t1 + c1 + 9) {
            private_C[0][1] *= beta;
            if (nj >= t1 + c1 + 17) {
              private_C[0][2] *= beta;
              if (nj >= t1 + c1 + 25) {
                private_C[0][3] *= beta;
                if (nj >= t1 + c1 + 33) {
                  private_C[0][4] *= beta;
                  if (nj >= t1 + c1 + 41) {
                    private_C[0][5] *= beta;
                    if (nj >= t1 + c1 + 49) {
                      private_C[0][6] *= beta;
                      if (nj >= t1 + c1 + 57) {
                        private_C[0][7] *= beta;
                        if (nj >= t1 + c1 + 65) {
                          private_C[0][8] *= beta;
                          if (nj >= t1 + c1 + 73) {
                            private_C[0][9] *= beta;
                            if (nj >= t1 + c1 + 81) {
                              private_C[0][10] *= beta;
                              if (nj >= t1 + c1 + 89) {
                                private_C[0][11] *= beta;
                                if (nj >= t1 + c1 + 97) {
                                  private_C[0][12] *= beta;
                                  if (nj >= t1 + c1 + 105) {
                                    private_C[0][13] *= beta;
                                    if (nj >= t1 + c1 + 113) {
                                      private_C[0][14] *= beta;
                                      if (nj >= t1 + c1 + 121)
                                        private_C[0][15] *= beta;
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          if (ni >= t0 + c0 + 9) {
            private_C[1][0] *= beta;
            if (nj >= t1 + c1 + 9) {
              private_C[1][1] *= beta;
              if (nj >= t1 + c1 + 17) {
                private_C[1][2] *= beta;
                if (nj >= t1 + c1 + 25) {
                  private_C[1][3] *= beta;
                  if (nj >= t1 + c1 + 33) {
                    private_C[1][4] *= beta;
                    if (nj >= t1 + c1 + 41) {
                      private_C[1][5] *= beta;
                      if (nj >= t1 + c1 + 49) {
                        private_C[1][6] *= beta;
                        if (nj >= t1 + c1 + 57) {
                          private_C[1][7] *= beta;
                          if (nj >= t1 + c1 + 65) {
                            private_C[1][8] *= beta;
                            if (nj >= t1 + c1 + 73) {
                              private_C[1][9] *= beta;
                              if (nj >= t1 + c1 + 81) {
                                private_C[1][10] *= beta;
                                if (nj >= t1 + c1 + 89) {
                                  private_C[1][11] *= beta;
                                  if (nj >= t1 + c1 + 97) {
                                    private_C[1][12] *= beta;
                                    if (nj >= t1 + c1 + 105) {
                                      private_C[1][13] *= beta;
                                      if (nj >= t1 + c1 + 113) {
                                        private_C[1][14] *= beta;
                                        if (nj >= t1 + c1 + 121)
                                          private_C[1][15] *= beta;
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
            if (ni >= t0 + c0 + 17) {
              private_C[2][0] *= beta;
              if (nj >= t1 + c1 + 9) {
                private_C[2][1] *= beta;
                if (nj >= t1 + c1 + 17) {
                  private_C[2][2] *= beta;
                  if (nj >= t1 + c1 + 25) {
                    private_C[2][3] *= beta;
                    if (nj >= t1 + c1 + 33) {
                      private_C[2][4] *= beta;
                      if (nj >= t1 + c1 + 41) {
                        private_C[2][5] *= beta;
                        if (nj >= t1 + c1 + 49) {
                          private_C[2][6] *= beta;
                          if (nj >= t1 + c1 + 57) {
                            private_C[2][7] *= beta;
                            if (nj >= t1 + c1 + 65) {
                              private_C[2][8] *= beta;
                              if (nj >= t1 + c1 + 73) {
                                private_C[2][9] *= beta;
                                if (nj >= t1 + c1 + 81) {
                                  private_C[2][10] *= beta;
                                  if (nj >= t1 + c1 + 89) {
                                    private_C[2][11] *= beta;
                                    if (nj >= t1 + c1 + 97) {
                                      private_C[2][12] *= beta;
                                      if (nj >= t1 + c1 + 105) {
                                        private_C[2][13] *= beta;
                                        if (nj >= t1 + c1 + 113) {
                                          private_C[2][14] *= beta;
                                          if (nj >= t1 + c1 + 121)
                                            private_C[2][15] *= beta;
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
              if (ni >= t0 + c0 + 25) {
                private_C[3][0] *= beta;
                if (nj >= t1 + c1 + 9) {
                  private_C[3][1] *= beta;
                  if (nj >= t1 + c1 + 17) {
                    private_C[3][2] *= beta;
                    if (nj >= t1 + c1 + 25) {
                      private_C[3][3] *= beta;
                      if (nj >= t1 + c1 + 33) {
                        private_C[3][4] *= beta;
                        if (nj >= t1 + c1 + 41) {
                          private_C[3][5] *= beta;
                          if (nj >= t1 + c1 + 49) {
                            private_C[3][6] *= beta;
                            if (nj >= t1 + c1 + 57) {
                              private_C[3][7] *= beta;
                              if (nj >= t1 + c1 + 65) {
                                private_C[3][8] *= beta;
                                if (nj >= t1 + c1 + 73) {
                                  private_C[3][9] *= beta;
                                  if (nj >= t1 + c1 + 81) {
                                    private_C[3][10] *= beta;
                                    if (nj >= t1 + c1 + 89) {
                                      private_C[3][11] *= beta;
                                      if (nj >= t1 + c1 + 97) {
                                        private_C[3][12] *= beta;
                                        if (nj >= t1 + c1 + 105) {
                                          private_C[3][13] *= beta;
                                          if (nj >= t1 + c1 + 113) {
                                            private_C[3][14] *= beta;
                                            if (nj >= t1 + c1 + 121)
                                              private_C[3][15] *= beta;
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
                if (ni >= t0 + c0 + 33) {
                  private_C[4][0] *= beta;
                  if (nj >= t1 + c1 + 9) {
                    private_C[4][1] *= beta;
                    if (nj >= t1 + c1 + 17) {
                      private_C[4][2] *= beta;
                      if (nj >= t1 + c1 + 25) {
                        private_C[4][3] *= beta;
                        if (nj >= t1 + c1 + 33) {
                          private_C[4][4] *= beta;
                          if (nj >= t1 + c1 + 41) {
                            private_C[4][5] *= beta;
                            if (nj >= t1 + c1 + 49) {
                              private_C[4][6] *= beta;
                              if (nj >= t1 + c1 + 57) {
                                private_C[4][7] *= beta;
                                if (nj >= t1 + c1 + 65) {
                                  private_C[4][8] *= beta;
                                  if (nj >= t1 + c1 + 73) {
                                    private_C[4][9] *= beta;
                                    if (nj >= t1 + c1 + 81) {
                                      private_C[4][10] *= beta;
                                      if (nj >= t1 + c1 + 89) {
                                        private_C[4][11] *= beta;
                                        if (nj >= t1 + c1 + 97) {
                                          private_C[4][12] *= beta;
                                          if (nj >= t1 + c1 + 105) {
                                            private_C[4][13] *= beta;
                                            if (nj >= t1 + c1 + 113) {
                                              private_C[4][14] *= beta;
                                              if (nj >= t1 + c1 + 121)
                                                private_C[4][15] *= beta;
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                  if (ni >= t0 + c0 + 41) {
                    private_C[5][0] *= beta;
                    if (nj >= t1 + c1 + 9) {
                      private_C[5][1] *= beta;
                      if (nj >= t1 + c1 + 17) {
                        private_C[5][2] *= beta;
                        if (nj >= t1 + c1 + 25) {
                          private_C[5][3] *= beta;
                          if (nj >= t1 + c1 + 33) {
                            private_C[5][4] *= beta;
                            if (nj >= t1 + c1 + 41) {
                              private_C[5][5] *= beta;
                              if (nj >= t1 + c1 + 49) {
                                private_C[5][6] *= beta;
                                if (nj >= t1 + c1 + 57) {
                                  private_C[5][7] *= beta;
                                  if (nj >= t1 + c1 + 65) {
                                    private_C[5][8] *= beta;
                                    if (nj >= t1 + c1 + 73) {
                                      private_C[5][9] *= beta;
                                      if (nj >= t1 + c1 + 81) {
                                        private_C[5][10] *= beta;
                                        if (nj >= t1 + c1 + 89) {
                                          private_C[5][11] *= beta;
                                          if (nj >= t1 + c1 + 97) {
                                            private_C[5][12] *= beta;
                                            if (nj >= t1 + c1 + 105) {
                                              private_C[5][13] *= beta;
                                              if (nj >= t1 + c1 + 113) {
                                                private_C[5][14] *= beta;
                                                if (nj >= t1 + c1 + 121)
                                                  private_C[5][15] *= beta;
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                    if (ni >= t0 + c0 + 49) {
                      private_C[6][0] *= beta;
                      if (nj >= t1 + c1 + 9) {
                        private_C[6][1] *= beta;
                        if (nj >= t1 + c1 + 17) {
                          private_C[6][2] *= beta;
                          if (nj >= t1 + c1 + 25) {
                            private_C[6][3] *= beta;
                            if (nj >= t1 + c1 + 33) {
                              private_C[6][4] *= beta;
                              if (nj >= t1 + c1 + 41) {
                                private_C[6][5] *= beta;
                                if (nj >= t1 + c1 + 49) {
                                  private_C[6][6] *= beta;
                                  if (nj >= t1 + c1 + 57) {
                                    private_C[6][7] *= beta;
                                    if (nj >= t1 + c1 + 65) {
                                      private_C[6][8] *= beta;
                                      if (nj >= t1 + c1 + 73) {
                                        private_C[6][9] *= beta;
                                        if (nj >= t1 + c1 + 81) {
                                          private_C[6][10] *= beta;
                                          if (nj >= t1 + c1 + 89) {
                                            private_C[6][11] *= beta;
                                            if (nj >= t1 + c1 + 97) {
                                              private_C[6][12] *= beta;
                                              if (nj >= t1 + c1 + 105) {
                                                private_C[6][13] *= beta;
                                                if (nj >= t1 + c1 + 113) {
                                                  private_C[6][14] *= beta;
                                                  if (nj >= t1 + c1 + 121)
                                                    private_C[6][15] *= beta;
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                      if (ni >= t0 + c0 + 57) {
                        private_C[7][0] *= beta;
                        if (nj >= t1 + c1 + 9) {
                          private_C[7][1] *= beta;
                          if (nj >= t1 + c1 + 17) {
                            private_C[7][2] *= beta;
                            if (nj >= t1 + c1 + 25) {
                              private_C[7][3] *= beta;
                              if (nj >= t1 + c1 + 33) {
                                private_C[7][4] *= beta;
                                if (nj >= t1 + c1 + 41) {
                                  private_C[7][5] *= beta;
                                  if (nj >= t1 + c1 + 49) {
                                    private_C[7][6] *= beta;
                                    if (nj >= t1 + c1 + 57) {
                                      private_C[7][7] *= beta;
                                      if (nj >= t1 + c1 + 65) {
                                        private_C[7][8] *= beta;
                                        if (nj >= t1 + c1 + 73) {
                                          private_C[7][9] *= beta;
                                          if (nj >= t1 + c1 + 81) {
                                            private_C[7][10] *= beta;
                                            if (nj >= t1 + c1 + 89) {
                                              private_C[7][11] *= beta;
                                              if (nj >= t1 + c1 + 97) {
                                                private_C[7][12] *= beta;
                                                if (nj >= t1 + c1 + 105) {
                                                  private_C[7][13] *= beta;
                                                  if (nj >= t1 + c1 + 113) {
                                                    private_C[7][14] *= beta;
                                                    if (nj >= t1 + c1 + 121)
                                                      private_C[7][15] *= beta;
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                        if (ni >= t0 + c0 + 65) {
                          private_C[8][0] *= beta;
                          if (nj >= t1 + c1 + 9) {
                            private_C[8][1] *= beta;
                            if (nj >= t1 + c1 + 17) {
                              private_C[8][2] *= beta;
                              if (nj >= t1 + c1 + 25) {
                                private_C[8][3] *= beta;
                                if (nj >= t1 + c1 + 33) {
                                  private_C[8][4] *= beta;
                                  if (nj >= t1 + c1 + 41) {
                                    private_C[8][5] *= beta;
                                    if (nj >= t1 + c1 + 49) {
                                      private_C[8][6] *= beta;
                                      if (nj >= t1 + c1 + 57) {
                                        private_C[8][7] *= beta;
                                        if (nj >= t1 + c1 + 65) {
                                          private_C[8][8] *= beta;
                                          if (nj >= t1 + c1 + 73) {
                                            private_C[8][9] *= beta;
                                            if (nj >= t1 + c1 + 81) {
                                              private_C[8][10] *= beta;
                                              if (nj >= t1 + c1 + 89) {
                                                private_C[8][11] *= beta;
                                                if (nj >= t1 + c1 + 97) {
                                                  private_C[8][12] *= beta;
                                                  if (nj >= t1 + c1 + 105) {
                                                    private_C[8][13] *= beta;
                                                    if (nj >= t1 + c1 + 113) {
                                                      private_C[8][14] *= beta;
                                                      if (nj >= t1 + c1 + 121)
                                                        private_C[8][15] *= beta;
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                          if (ni >= t0 + c0 + 73) {
                            private_C[9][0] *= beta;
                            if (nj >= t1 + c1 + 9) {
                              private_C[9][1] *= beta;
                              if (nj >= t1 + c1 + 17) {
                                private_C[9][2] *= beta;
                                if (nj >= t1 + c1 + 25) {
                                  private_C[9][3] *= beta;
                                  if (nj >= t1 + c1 + 33) {
                                    private_C[9][4] *= beta;
                                    if (nj >= t1 + c1 + 41) {
                                      private_C[9][5] *= beta;
                                      if (nj >= t1 + c1 + 49) {
                                        private_C[9][6] *= beta;
                                        if (nj >= t1 + c1 + 57) {
                                          private_C[9][7] *= beta;
                                          if (nj >= t1 + c1 + 65) {
                                            private_C[9][8] *= beta;
                                            if (nj >= t1 + c1 + 73) {
                                              private_C[9][9] *= beta;
                                              if (nj >= t1 + c1 + 81) {
                                                private_C[9][10] *= beta;
                                                if (nj >= t1 + c1 + 89) {
                                                  private_C[9][11] *= beta;
                                                  if (nj >= t1 + c1 + 97) {
                                                    private_C[9][12] *= beta;
                                                    if (nj >= t1 + c1 + 105) {
                                                      private_C[9][13] *= beta;
                                                      if (nj >= t1 + c1 + 113) {
                                                        private_C[9][14] *= beta;
                                                        if (nj >= t1 + c1 + 121)
                                                          private_C[9][15] *= beta;
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                            if (ni >= t0 + c0 + 81) {
                              private_C[10][0] *= beta;
                              if (nj >= t1 + c1 + 9) {
                                private_C[10][1] *= beta;
                                if (nj >= t1 + c1 + 17) {
                                  private_C[10][2] *= beta;
                                  if (nj >= t1 + c1 + 25) {
                                    private_C[10][3] *= beta;
                                    if (nj >= t1 + c1 + 33) {
                                      private_C[10][4] *= beta;
                                      if (nj >= t1 + c1 + 41) {
                                        private_C[10][5] *= beta;
                                        if (nj >= t1 + c1 + 49) {
                                          private_C[10][6] *= beta;
                                          if (nj >= t1 + c1 + 57) {
                                            private_C[10][7] *= beta;
                                            if (nj >= t1 + c1 + 65) {
                                              private_C[10][8] *= beta;
                                              if (nj >= t1 + c1 + 73) {
                                                private_C[10][9] *= beta;
                                                if (nj >= t1 + c1 + 81) {
                                                  private_C[10][10] *= beta;
                                                  if (nj >= t1 + c1 + 89) {
                                                    private_C[10][11] *= beta;
                                                    if (nj >= t1 + c1 + 97) {
                                                      private_C[10][12] *= beta;
                                                      if (nj >= t1 + c1 + 105) {
                                                        private_C[10][13] *= beta;
                                                        if (nj >= t1 + c1 + 113) {
                                                          private_C[10][14] *= beta;
                                                          if (nj >= t1 + c1 + 121)
                                                            private_C[10][15] *= beta;
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                              if (ni >= t0 + c0 + 89) {
                                private_C[11][0] *= beta;
                                if (nj >= t1 + c1 + 9) {
                                  private_C[11][1] *= beta;
                                  if (nj >= t1 + c1 + 17) {
                                    private_C[11][2] *= beta;
                                    if (nj >= t1 + c1 + 25) {
                                      private_C[11][3] *= beta;
                                      if (nj >= t1 + c1 + 33) {
                                        private_C[11][4] *= beta;
                                        if (nj >= t1 + c1 + 41) {
                                          private_C[11][5] *= beta;
                                          if (nj >= t1 + c1 + 49) {
                                            private_C[11][6] *= beta;
                                            if (nj >= t1 + c1 + 57) {
                                              private_C[11][7] *= beta;
                                              if (nj >= t1 + c1 + 65) {
                                                private_C[11][8] *= beta;
                                                if (nj >= t1 + c1 + 73) {
                                                  private_C[11][9] *= beta;
                                                  if (nj >= t1 + c1 + 81) {
                                                    private_C[11][10] *= beta;
                                                    if (nj >= t1 + c1 + 89) {
                                                      private_C[11][11] *= beta;
                                                      if (nj >= t1 + c1 + 97) {
                                                        private_C[11][12] *= beta;
                                                        if (nj >= t1 + c1 + 105) {
                                                          private_C[11][13] *= beta;
                                                          if (nj >= t1 + c1 + 113) {
                                                            private_C[11][14] *= beta;
                                                            if (nj >= t1 + c1 + 121)
                                                              private_C[11][15] *= beta;
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                                if (ni >= t0 + c0 + 97) {
                                  private_C[12][0] *= beta;
                                  if (nj >= t1 + c1 + 9) {
                                    private_C[12][1] *= beta;
                                    if (nj >= t1 + c1 + 17) {
                                      private_C[12][2] *= beta;
                                      if (nj >= t1 + c1 + 25) {
                                        private_C[12][3] *= beta;
                                        if (nj >= t1 + c1 + 33) {
                                          private_C[12][4] *= beta;
                                          if (nj >= t1 + c1 + 41) {
                                            private_C[12][5] *= beta;
                                            if (nj >= t1 + c1 + 49) {
                                              private_C[12][6] *= beta;
                                              if (nj >= t1 + c1 + 57) {
                                                private_C[12][7] *= beta;
                                                if (nj >= t1 + c1 + 65) {
                                                  private_C[12][8] *= beta;
                                                  if (nj >= t1 + c1 + 73) {
                                                    private_C[12][9] *= beta;
                                                    if (nj >= t1 + c1 + 81) {
                                                      private_C[12][10] *= beta;
                                                      if (nj >= t1 + c1 + 89) {
                                                        private_C[12][11] *= beta;
                                                        if (nj >= t1 + c1 + 97) {
                                                          private_C[12][12] *= beta;
                                                          if (nj >= t1 + c1 + 105) {
                                                            private_C[12][13] *= beta;
                                                            if (nj >= t1 + c1 + 113) {
                                                              private_C[12][14] *= beta;
                                                              if (nj >= t1 + c1 + 121)
                                                                private_C[12][15] *= beta;
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                  if (ni >= t0 + c0 + 105) {
                                    private_C[13][0] *= beta;
                                    if (nj >= t1 + c1 + 9) {
                                      private_C[13][1] *= beta;
                                      if (nj >= t1 + c1 + 17) {
                                        private_C[13][2] *= beta;
                                        if (nj >= t1 + c1 + 25) {
                                          private_C[13][3] *= beta;
                                          if (nj >= t1 + c1 + 33) {
                                            private_C[13][4] *= beta;
                                            if (nj >= t1 + c1 + 41) {
                                              private_C[13][5] *= beta;
                                              if (nj >= t1 + c1 + 49) {
                                                private_C[13][6] *= beta;
                                                if (nj >= t1 + c1 + 57) {
                                                  private_C[13][7] *= beta;
                                                  if (nj >= t1 + c1 + 65) {
                                                    private_C[13][8] *= beta;
                                                    if (nj >= t1 + c1 + 73) {
                                                      private_C[13][9] *= beta;
                                                      if (nj >= t1 + c1 + 81) {
                                                        private_C[13][10] *= beta;
                                                        if (nj >= t1 + c1 + 89) {
                                                          private_C[13][11] *= beta;
                                                          if (nj >= t1 + c1 + 97) {
                                                            private_C[13][12] *= beta;
                                                            if (nj >= t1 + c1 + 105) {
                                                              private_C[13][13] *= beta;
                                                              if (nj >= t1 + c1 + 113) {
                                                                private_C[13][14] *= beta;
                                                                if (nj >= t1 + c1 + 121)
                                                                  private_C[13][15] *= beta;
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                    if (ni >= t0 + c0 + 113) {
                                      private_C[14][0] *= beta;
                                      if (nj >= t1 + c1 + 9) {
                                        private_C[14][1] *= beta;
                                        if (nj >= t1 + c1 + 17) {
                                          private_C[14][2] *= beta;
                                          if (nj >= t1 + c1 + 25) {
                                            private_C[14][3] *= beta;
                                            if (nj >= t1 + c1 + 33) {
                                              private_C[14][4] *= beta;
                                              if (nj >= t1 + c1 + 41) {
                                                private_C[14][5] *= beta;
                                                if (nj >= t1 + c1 + 49) {
                                                  private_C[14][6] *= beta;
                                                  if (nj >= t1 + c1 + 57) {
                                                    private_C[14][7] *= beta;
                                                    if (nj >= t1 + c1 + 65) {
                                                      private_C[14][8] *= beta;
                                                      if (nj >= t1 + c1 + 73) {
                                                        private_C[14][9] *= beta;
                                                        if (nj >= t1 + c1 + 81) {
                                                          private_C[14][10] *= beta;
                                                          if (nj >= t1 + c1 + 89) {
                                                            private_C[14][11] *= beta;
                                                            if (nj >= t1 + c1 + 97) {
                                                              private_C[14][12] *= beta;
                                                              if (nj >= t1 + c1 + 105) {
                                                                private_C[14][13] *= beta;
                                                                if (nj >= t1 + c1 + 113) {
                                                                  private_C[14][14] *= beta;
                                                                  if (nj >= t1 + c1 + 121)
                                                                    private_C[14][15] *= beta;
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                      if (ni >= t0 + c0 + 121) {
                                        private_C[15][0] *= beta;
                                        if (nj >= t1 + c1 + 9) {
                                          private_C[15][1] *= beta;
                                          if (nj >= t1 + c1 + 17) {
                                            private_C[15][2] *= beta;
                                            if (nj >= t1 + c1 + 25) {
                                              private_C[15][3] *= beta;
                                              if (nj >= t1 + c1 + 33) {
                                                private_C[15][4] *= beta;
                                                if (nj >= t1 + c1 + 41) {
                                                  private_C[15][5] *= beta;
                                                  if (nj >= t1 + c1 + 49) {
                                                    private_C[15][6] *= beta;
                                                    if (nj >= t1 + c1 + 57) {
                                                      private_C[15][7] *= beta;
                                                      if (nj >= t1 + c1 + 65) {
                                                        private_C[15][8] *= beta;
                                                        if (nj >= t1 + c1 + 73) {
                                                          private_C[15][9] *= beta;
                                                          if (nj >= t1 + c1 + 81) {
                                                            private_C[15][10] *= beta;
                                                            if (nj >= t1 + c1 + 89) {
                                                              private_C[15][11] *= beta;
                                                              if (nj >= t1 + c1 + 97) {
                                                                private_C[15][12] *= beta;
                                                                if (nj >= t1 + c1 + 105) {
                                                                  private_C[15][13] *= beta;
                                                                  if (nj >= t1 + c1 + 113) {
                                                                    private_C[15][14] *= beta;
                                                                    if (nj >= t1 + c1 + 121)
                                                                      private_C[15][15] *= beta;
                                                                  }
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          for (int c2 = 0; c2 < nk; c2 += 128)
            for (int c3 = 0; c3 <= ppcg_min(127, nk - c2 - 1); c3 += 1) {
              private_C[0][0] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
              if (nj >= t1 + c1 + 9) {
                private_C[0][1] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                if (nj >= t1 + c1 + 17) {
                  private_C[0][2] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                  if (nj >= t1 + c1 + 25) {
                    private_C[0][3] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                    if (nj >= t1 + c1 + 33) {
                      private_C[0][4] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                      if (nj >= t1 + c1 + 41) {
                        private_C[0][5] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                        if (nj >= t1 + c1 + 49) {
                          private_C[0][6] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                          if (nj >= t1 + c1 + 57) {
                            private_C[0][7] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                            if (nj >= t1 + c1 + 65) {
                              private_C[0][8] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                              if (nj >= t1 + c1 + 73) {
                                private_C[0][9] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                if (nj >= t1 + c1 + 81) {
                                  private_C[0][10] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                  if (nj >= t1 + c1 + 89) {
                                    private_C[0][11] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                    if (nj >= t1 + c1 + 97) {
                                      private_C[0][12] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                      if (nj >= t1 + c1 + 105) {
                                        private_C[0][13] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                        if (nj >= t1 + c1 + 113) {
                                          private_C[0][14] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                          if (nj >= t1 + c1 + 121)
                                            private_C[0][15] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
              if (ni >= t0 + c0 + 9) {
                private_C[1][0] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                if (nj >= t1 + c1 + 9) {
                  private_C[1][1] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                  if (nj >= t1 + c1 + 17) {
                    private_C[1][2] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                    if (nj >= t1 + c1 + 25) {
                      private_C[1][3] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                      if (nj >= t1 + c1 + 33) {
                        private_C[1][4] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                        if (nj >= t1 + c1 + 41) {
                          private_C[1][5] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                          if (nj >= t1 + c1 + 49) {
                            private_C[1][6] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                            if (nj >= t1 + c1 + 57) {
                              private_C[1][7] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                              if (nj >= t1 + c1 + 65) {
                                private_C[1][8] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                if (nj >= t1 + c1 + 73) {
                                  private_C[1][9] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                  if (nj >= t1 + c1 + 81) {
                                    private_C[1][10] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                    if (nj >= t1 + c1 + 89) {
                                      private_C[1][11] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                      if (nj >= t1 + c1 + 97) {
                                        private_C[1][12] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                        if (nj >= t1 + c1 + 105) {
                                          private_C[1][13] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                          if (nj >= t1 + c1 + 113) {
                                            private_C[1][14] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                            if (nj >= t1 + c1 + 121)
                                              private_C[1][15] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
                if (ni >= t0 + c0 + 17) {
                  private_C[2][0] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                  if (nj >= t1 + c1 + 9) {
                    private_C[2][1] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                    if (nj >= t1 + c1 + 17) {
                      private_C[2][2] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                      if (nj >= t1 + c1 + 25) {
                        private_C[2][3] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                        if (nj >= t1 + c1 + 33) {
                          private_C[2][4] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                          if (nj >= t1 + c1 + 41) {
                            private_C[2][5] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                            if (nj >= t1 + c1 + 49) {
                              private_C[2][6] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                              if (nj >= t1 + c1 + 57) {
                                private_C[2][7] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                if (nj >= t1 + c1 + 65) {
                                  private_C[2][8] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                  if (nj >= t1 + c1 + 73) {
                                    private_C[2][9] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                    if (nj >= t1 + c1 + 81) {
                                      private_C[2][10] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                      if (nj >= t1 + c1 + 89) {
                                        private_C[2][11] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                        if (nj >= t1 + c1 + 97) {
                                          private_C[2][12] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                          if (nj >= t1 + c1 + 105) {
                                            private_C[2][13] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                            if (nj >= t1 + c1 + 113) {
                                              private_C[2][14] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                              if (nj >= t1 + c1 + 121)
                                                private_C[2][15] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                  if (ni >= t0 + c0 + 25) {
                    private_C[3][0] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                    if (nj >= t1 + c1 + 9) {
                      private_C[3][1] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                      if (nj >= t1 + c1 + 17) {
                        private_C[3][2] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                        if (nj >= t1 + c1 + 25) {
                          private_C[3][3] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                          if (nj >= t1 + c1 + 33) {
                            private_C[3][4] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                            if (nj >= t1 + c1 + 41) {
                              private_C[3][5] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                              if (nj >= t1 + c1 + 49) {
                                private_C[3][6] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                if (nj >= t1 + c1 + 57) {
                                  private_C[3][7] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                  if (nj >= t1 + c1 + 65) {
                                    private_C[3][8] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                    if (nj >= t1 + c1 + 73) {
                                      private_C[3][9] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                      if (nj >= t1 + c1 + 81) {
                                        private_C[3][10] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                        if (nj >= t1 + c1 + 89) {
                                          private_C[3][11] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                          if (nj >= t1 + c1 + 97) {
                                            private_C[3][12] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                            if (nj >= t1 + c1 + 105) {
                                              private_C[3][13] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                              if (nj >= t1 + c1 + 113) {
                                                private_C[3][14] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                                if (nj >= t1 + c1 + 121)
                                                  private_C[3][15] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                    if (ni >= t0 + c0 + 33) {
                      private_C[4][0] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                      if (nj >= t1 + c1 + 9) {
                        private_C[4][1] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                        if (nj >= t1 + c1 + 17) {
                          private_C[4][2] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                          if (nj >= t1 + c1 + 25) {
                            private_C[4][3] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                            if (nj >= t1 + c1 + 33) {
                              private_C[4][4] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                              if (nj >= t1 + c1 + 41) {
                                private_C[4][5] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                if (nj >= t1 + c1 + 49) {
                                  private_C[4][6] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                  if (nj >= t1 + c1 + 57) {
                                    private_C[4][7] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                    if (nj >= t1 + c1 + 65) {
                                      private_C[4][8] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                      if (nj >= t1 + c1 + 73) {
                                        private_C[4][9] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                        if (nj >= t1 + c1 + 81) {
                                          private_C[4][10] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                          if (nj >= t1 + c1 + 89) {
                                            private_C[4][11] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                            if (nj >= t1 + c1 + 97) {
                                              private_C[4][12] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                              if (nj >= t1 + c1 + 105) {
                                                private_C[4][13] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                                if (nj >= t1 + c1 + 113) {
                                                  private_C[4][14] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                                  if (nj >= t1 + c1 + 121)
                                                    private_C[4][15] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                      if (ni >= t0 + c0 + 41) {
                        private_C[5][0] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                        if (nj >= t1 + c1 + 9) {
                          private_C[5][1] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                          if (nj >= t1 + c1 + 17) {
                            private_C[5][2] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                            if (nj >= t1 + c1 + 25) {
                              private_C[5][3] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                              if (nj >= t1 + c1 + 33) {
                                private_C[5][4] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                if (nj >= t1 + c1 + 41) {
                                  private_C[5][5] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                  if (nj >= t1 + c1 + 49) {
                                    private_C[5][6] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                    if (nj >= t1 + c1 + 57) {
                                      private_C[5][7] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                      if (nj >= t1 + c1 + 65) {
                                        private_C[5][8] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                        if (nj >= t1 + c1 + 73) {
                                          private_C[5][9] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                          if (nj >= t1 + c1 + 81) {
                                            private_C[5][10] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                            if (nj >= t1 + c1 + 89) {
                                              private_C[5][11] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                              if (nj >= t1 + c1 + 97) {
                                                private_C[5][12] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                                if (nj >= t1 + c1 + 105) {
                                                  private_C[5][13] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                                  if (nj >= t1 + c1 + 113) {
                                                    private_C[5][14] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                                    if (nj >= t1 + c1 + 121)
                                                      private_C[5][15] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                        if (ni >= t0 + c0 + 49) {
                          private_C[6][0] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                          if (nj >= t1 + c1 + 9) {
                            private_C[6][1] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                            if (nj >= t1 + c1 + 17) {
                              private_C[6][2] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                              if (nj >= t1 + c1 + 25) {
                                private_C[6][3] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                                if (nj >= t1 + c1 + 33) {
                                  private_C[6][4] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                  if (nj >= t1 + c1 + 41) {
                                    private_C[6][5] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                    if (nj >= t1 + c1 + 49) {
                                      private_C[6][6] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                      if (nj >= t1 + c1 + 57) {
                                        private_C[6][7] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                        if (nj >= t1 + c1 + 65) {
                                          private_C[6][8] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                          if (nj >= t1 + c1 + 73) {
                                            private_C[6][9] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                            if (nj >= t1 + c1 + 81) {
                                              private_C[6][10] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                              if (nj >= t1 + c1 + 89) {
                                                private_C[6][11] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                                if (nj >= t1 + c1 + 97) {
                                                  private_C[6][12] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                                  if (nj >= t1 + c1 + 105) {
                                                    private_C[6][13] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                                    if (nj >= t1 + c1 + 113) {
                                                      private_C[6][14] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                                      if (nj >= t1 + c1 + 121)
                                                        private_C[6][15] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                          if (ni >= t0 + c0 + 57) {
                            private_C[7][0] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                            if (nj >= t1 + c1 + 9) {
                              private_C[7][1] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                              if (nj >= t1 + c1 + 17) {
                                private_C[7][2] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                                if (nj >= t1 + c1 + 25) {
                                  private_C[7][3] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                                  if (nj >= t1 + c1 + 33) {
                                    private_C[7][4] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                    if (nj >= t1 + c1 + 41) {
                                      private_C[7][5] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                      if (nj >= t1 + c1 + 49) {
                                        private_C[7][6] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                        if (nj >= t1 + c1 + 57) {
                                          private_C[7][7] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                          if (nj >= t1 + c1 + 65) {
                                            private_C[7][8] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                            if (nj >= t1 + c1 + 73) {
                                              private_C[7][9] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                              if (nj >= t1 + c1 + 81) {
                                                private_C[7][10] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                                if (nj >= t1 + c1 + 89) {
                                                  private_C[7][11] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                                  if (nj >= t1 + c1 + 97) {
                                                    private_C[7][12] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                                    if (nj >= t1 + c1 + 105) {
                                                      private_C[7][13] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                                      if (nj >= t1 + c1 + 113) {
                                                        private_C[7][14] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                                        if (nj >= t1 + c1 + 121)
                                                          private_C[7][15] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                            if (ni >= t0 + c0 + 65) {
                              private_C[8][0] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                              if (nj >= t1 + c1 + 9) {
                                private_C[8][1] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                                if (nj >= t1 + c1 + 17) {
                                  private_C[8][2] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                                  if (nj >= t1 + c1 + 25) {
                                    private_C[8][3] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                                    if (nj >= t1 + c1 + 33) {
                                      private_C[8][4] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                      if (nj >= t1 + c1 + 41) {
                                        private_C[8][5] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                        if (nj >= t1 + c1 + 49) {
                                          private_C[8][6] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                          if (nj >= t1 + c1 + 57) {
                                            private_C[8][7] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                            if (nj >= t1 + c1 + 65) {
                                              private_C[8][8] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                              if (nj >= t1 + c1 + 73) {
                                                private_C[8][9] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                                if (nj >= t1 + c1 + 81) {
                                                  private_C[8][10] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                                  if (nj >= t1 + c1 + 89) {
                                                    private_C[8][11] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                                    if (nj >= t1 + c1 + 97) {
                                                      private_C[8][12] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                                      if (nj >= t1 + c1 + 105) {
                                                        private_C[8][13] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                                        if (nj >= t1 + c1 + 113) {
                                                          private_C[8][14] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                                          if (nj >= t1 + c1 + 121)
                                                            private_C[8][15] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                              if (ni >= t0 + c0 + 73) {
                                private_C[9][0] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                if (nj >= t1 + c1 + 9) {
                                  private_C[9][1] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                                  if (nj >= t1 + c1 + 17) {
                                    private_C[9][2] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                                    if (nj >= t1 + c1 + 25) {
                                      private_C[9][3] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                                      if (nj >= t1 + c1 + 33) {
                                        private_C[9][4] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                        if (nj >= t1 + c1 + 41) {
                                          private_C[9][5] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                          if (nj >= t1 + c1 + 49) {
                                            private_C[9][6] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                            if (nj >= t1 + c1 + 57) {
                                              private_C[9][7] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                              if (nj >= t1 + c1 + 65) {
                                                private_C[9][8] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                                if (nj >= t1 + c1 + 73) {
                                                  private_C[9][9] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                                  if (nj >= t1 + c1 + 81) {
                                                    private_C[9][10] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                                    if (nj >= t1 + c1 + 89) {
                                                      private_C[9][11] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                                      if (nj >= t1 + c1 + 97) {
                                                        private_C[9][12] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                                        if (nj >= t1 + c1 + 105) {
                                                          private_C[9][13] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                                          if (nj >= t1 + c1 + 113) {
                                                            private_C[9][14] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                                            if (nj >= t1 + c1 + 121)
                                                              private_C[9][15] += ((alpha * A[(t0 + c0 + 72) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                                if (ni >= t0 + c0 + 81) {
                                  private_C[10][0] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                  if (nj >= t1 + c1 + 9) {
                                    private_C[10][1] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                                    if (nj >= t1 + c1 + 17) {
                                      private_C[10][2] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                                      if (nj >= t1 + c1 + 25) {
                                        private_C[10][3] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                                        if (nj >= t1 + c1 + 33) {
                                          private_C[10][4] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                          if (nj >= t1 + c1 + 41) {
                                            private_C[10][5] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                            if (nj >= t1 + c1 + 49) {
                                              private_C[10][6] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                              if (nj >= t1 + c1 + 57) {
                                                private_C[10][7] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                                if (nj >= t1 + c1 + 65) {
                                                  private_C[10][8] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                                  if (nj >= t1 + c1 + 73) {
                                                    private_C[10][9] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                                    if (nj >= t1 + c1 + 81) {
                                                      private_C[10][10] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                                      if (nj >= t1 + c1 + 89) {
                                                        private_C[10][11] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                                        if (nj >= t1 + c1 + 97) {
                                                          private_C[10][12] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                                          if (nj >= t1 + c1 + 105) {
                                                            private_C[10][13] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                                            if (nj >= t1 + c1 + 113) {
                                                              private_C[10][14] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                                              if (nj >= t1 + c1 + 121)
                                                                private_C[10][15] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                  if (ni >= t0 + c0 + 89) {
                                    private_C[11][0] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                    if (nj >= t1 + c1 + 9) {
                                      private_C[11][1] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                                      if (nj >= t1 + c1 + 17) {
                                        private_C[11][2] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                                        if (nj >= t1 + c1 + 25) {
                                          private_C[11][3] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                                          if (nj >= t1 + c1 + 33) {
                                            private_C[11][4] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                            if (nj >= t1 + c1 + 41) {
                                              private_C[11][5] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                              if (nj >= t1 + c1 + 49) {
                                                private_C[11][6] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                                if (nj >= t1 + c1 + 57) {
                                                  private_C[11][7] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                                  if (nj >= t1 + c1 + 65) {
                                                    private_C[11][8] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                                    if (nj >= t1 + c1 + 73) {
                                                      private_C[11][9] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                                      if (nj >= t1 + c1 + 81) {
                                                        private_C[11][10] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                                        if (nj >= t1 + c1 + 89) {
                                                          private_C[11][11] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                                          if (nj >= t1 + c1 + 97) {
                                                            private_C[11][12] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                                            if (nj >= t1 + c1 + 105) {
                                                              private_C[11][13] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                                              if (nj >= t1 + c1 + 113) {
                                                                private_C[11][14] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                                                if (nj >= t1 + c1 + 121)
                                                                  private_C[11][15] += ((alpha * A[(t0 + c0 + 88) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                    if (ni >= t0 + c0 + 97) {
                                      private_C[12][0] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                      if (nj >= t1 + c1 + 9) {
                                        private_C[12][1] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                                        if (nj >= t1 + c1 + 17) {
                                          private_C[12][2] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                                          if (nj >= t1 + c1 + 25) {
                                            private_C[12][3] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                                            if (nj >= t1 + c1 + 33) {
                                              private_C[12][4] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                              if (nj >= t1 + c1 + 41) {
                                                private_C[12][5] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                                if (nj >= t1 + c1 + 49) {
                                                  private_C[12][6] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                                  if (nj >= t1 + c1 + 57) {
                                                    private_C[12][7] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                                    if (nj >= t1 + c1 + 65) {
                                                      private_C[12][8] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                                      if (nj >= t1 + c1 + 73) {
                                                        private_C[12][9] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                                        if (nj >= t1 + c1 + 81) {
                                                          private_C[12][10] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                                          if (nj >= t1 + c1 + 89) {
                                                            private_C[12][11] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                                            if (nj >= t1 + c1 + 97) {
                                                              private_C[12][12] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                                              if (nj >= t1 + c1 + 105) {
                                                                private_C[12][13] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                                                if (nj >= t1 + c1 + 113) {
                                                                  private_C[12][14] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                                                  if (nj >= t1 + c1 + 121)
                                                                    private_C[12][15] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                      if (ni >= t0 + c0 + 105) {
                                        private_C[13][0] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                        if (nj >= t1 + c1 + 9) {
                                          private_C[13][1] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                                          if (nj >= t1 + c1 + 17) {
                                            private_C[13][2] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                                            if (nj >= t1 + c1 + 25) {
                                              private_C[13][3] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                                              if (nj >= t1 + c1 + 33) {
                                                private_C[13][4] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                                if (nj >= t1 + c1 + 41) {
                                                  private_C[13][5] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                                  if (nj >= t1 + c1 + 49) {
                                                    private_C[13][6] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                                    if (nj >= t1 + c1 + 57) {
                                                      private_C[13][7] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                                      if (nj >= t1 + c1 + 65) {
                                                        private_C[13][8] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                                        if (nj >= t1 + c1 + 73) {
                                                          private_C[13][9] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                                          if (nj >= t1 + c1 + 81) {
                                                            private_C[13][10] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                                            if (nj >= t1 + c1 + 89) {
                                                              private_C[13][11] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                                              if (nj >= t1 + c1 + 97) {
                                                                private_C[13][12] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                                                if (nj >= t1 + c1 + 105) {
                                                                  private_C[13][13] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                                                  if (nj >= t1 + c1 + 113) {
                                                                    private_C[13][14] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                                                    if (nj >= t1 + c1 + 121)
                                                                      private_C[13][15] += ((alpha * A[(t0 + c0 + 104) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                                                  }
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                        if (ni >= t0 + c0 + 113) {
                                          private_C[14][0] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                          if (nj >= t1 + c1 + 9) {
                                            private_C[14][1] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                                            if (nj >= t1 + c1 + 17) {
                                              private_C[14][2] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                                              if (nj >= t1 + c1 + 25) {
                                                private_C[14][3] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                                                if (nj >= t1 + c1 + 33) {
                                                  private_C[14][4] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                                  if (nj >= t1 + c1 + 41) {
                                                    private_C[14][5] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                                    if (nj >= t1 + c1 + 49) {
                                                      private_C[14][6] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                                      if (nj >= t1 + c1 + 57) {
                                                        private_C[14][7] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                                        if (nj >= t1 + c1 + 65) {
                                                          private_C[14][8] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                                          if (nj >= t1 + c1 + 73) {
                                                            private_C[14][9] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                                            if (nj >= t1 + c1 + 81) {
                                                              private_C[14][10] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                                              if (nj >= t1 + c1 + 89) {
                                                                private_C[14][11] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                                                if (nj >= t1 + c1 + 97) {
                                                                  private_C[14][12] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                                                  if (nj >= t1 + c1 + 105) {
                                                                    private_C[14][13] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                                                    if (nj >= t1 + c1 + 113) {
                                                                      private_C[14][14] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                                                      if (nj >= t1 + c1 + 121)
                                                                        private_C[14][15] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                                                    }
                                                                  }
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                          if (ni >= t0 + c0 + 121) {
                                            private_C[15][0] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                            if (nj >= t1 + c1 + 9) {
                                              private_C[15][1] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                                              if (nj >= t1 + c1 + 17) {
                                                private_C[15][2] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                                                if (nj >= t1 + c1 + 25) {
                                                  private_C[15][3] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                                                  if (nj >= t1 + c1 + 33) {
                                                    private_C[15][4] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                                    if (nj >= t1 + c1 + 41) {
                                                      private_C[15][5] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                                      if (nj >= t1 + c1 + 49) {
                                                        private_C[15][6] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                                        if (nj >= t1 + c1 + 57) {
                                                          private_C[15][7] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                                          if (nj >= t1 + c1 + 65) {
                                                            private_C[15][8] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                                            if (nj >= t1 + c1 + 73) {
                                                              private_C[15][9] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 72)]);
                                                              if (nj >= t1 + c1 + 81) {
                                                                private_C[15][10] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 80)]);
                                                                if (nj >= t1 + c1 + 89) {
                                                                  private_C[15][11] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 88)]);
                                                                  if (nj >= t1 + c1 + 97) {
                                                                    private_C[15][12] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                                                    if (nj >= t1 + c1 + 105) {
                                                                      private_C[15][13] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 104)]);
                                                                      if (nj >= t1 + c1 + 113) {
                                                                        private_C[15][14] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 112)]);
                                                                        if (nj >= t1 + c1 + 121)
                                                                          private_C[15][15] += ((alpha * A[(t0 + c0 + 120) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 120)]);
                                                                      }
                                                                    }
                                                                  }
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          C[(t0 + c0) * nj + (t1 + c1)] = private_C[0][0];
          if (nj >= t1 + c1 + 9) {
            C[(t0 + c0) * nj + (t1 + c1 + 8)] = private_C[0][1];
            if (nj >= t1 + c1 + 17) {
              C[(t0 + c0) * nj + (t1 + c1 + 16)] = private_C[0][2];
              if (nj >= t1 + c1 + 25) {
                C[(t0 + c0) * nj + (t1 + c1 + 24)] = private_C[0][3];
                if (nj >= t1 + c1 + 33) {
                  C[(t0 + c0) * nj + (t1 + c1 + 32)] = private_C[0][4];
                  if (nj >= t1 + c1 + 41) {
                    C[(t0 + c0) * nj + (t1 + c1 + 40)] = private_C[0][5];
                    if (nj >= t1 + c1 + 49) {
                      C[(t0 + c0) * nj + (t1 + c1 + 48)] = private_C[0][6];
                      if (nj >= t1 + c1 + 57) {
                        C[(t0 + c0) * nj + (t1 + c1 + 56)] = private_C[0][7];
                        if (nj >= t1 + c1 + 65) {
                          C[(t0 + c0) * nj + (t1 + c1 + 64)] = private_C[0][8];
                          if (nj >= t1 + c1 + 73) {
                            C[(t0 + c0) * nj + (t1 + c1 + 72)] = private_C[0][9];
                            if (nj >= t1 + c1 + 81) {
                              C[(t0 + c0) * nj + (t1 + c1 + 80)] = private_C[0][10];
                              if (nj >= t1 + c1 + 89) {
                                C[(t0 + c0) * nj + (t1 + c1 + 88)] = private_C[0][11];
                                if (nj >= t1 + c1 + 97) {
                                  C[(t0 + c0) * nj + (t1 + c1 + 96)] = private_C[0][12];
                                  if (nj >= t1 + c1 + 105) {
                                    C[(t0 + c0) * nj + (t1 + c1 + 104)] = private_C[0][13];
                                    if (nj >= t1 + c1 + 113) {
                                      C[(t0 + c0) * nj + (t1 + c1 + 112)] = private_C[0][14];
                                      if (nj >= t1 + c1 + 121)
                                        C[(t0 + c0) * nj + (t1 + c1 + 120)] = private_C[0][15];
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          if (ni >= t0 + c0 + 9) {
            C[(t0 + c0 + 8) * nj + (t1 + c1)] = private_C[1][0];
            if (nj >= t1 + c1 + 9) {
              C[(t0 + c0 + 8) * nj + (t1 + c1 + 8)] = private_C[1][1];
              if (nj >= t1 + c1 + 17) {
                C[(t0 + c0 + 8) * nj + (t1 + c1 + 16)] = private_C[1][2];
                if (nj >= t1 + c1 + 25) {
                  C[(t0 + c0 + 8) * nj + (t1 + c1 + 24)] = private_C[1][3];
                  if (nj >= t1 + c1 + 33) {
                    C[(t0 + c0 + 8) * nj + (t1 + c1 + 32)] = private_C[1][4];
                    if (nj >= t1 + c1 + 41) {
                      C[(t0 + c0 + 8) * nj + (t1 + c1 + 40)] = private_C[1][5];
                      if (nj >= t1 + c1 + 49) {
                        C[(t0 + c0 + 8) * nj + (t1 + c1 + 48)] = private_C[1][6];
                        if (nj >= t1 + c1 + 57) {
                          C[(t0 + c0 + 8) * nj + (t1 + c1 + 56)] = private_C[1][7];
                          if (nj >= t1 + c1 + 65) {
                            C[(t0 + c0 + 8) * nj + (t1 + c1 + 64)] = private_C[1][8];
                            if (nj >= t1 + c1 + 73) {
                              C[(t0 + c0 + 8) * nj + (t1 + c1 + 72)] = private_C[1][9];
                              if (nj >= t1 + c1 + 81) {
                                C[(t0 + c0 + 8) * nj + (t1 + c1 + 80)] = private_C[1][10];
                                if (nj >= t1 + c1 + 89) {
                                  C[(t0 + c0 + 8) * nj + (t1 + c1 + 88)] = private_C[1][11];
                                  if (nj >= t1 + c1 + 97) {
                                    C[(t0 + c0 + 8) * nj + (t1 + c1 + 96)] = private_C[1][12];
                                    if (nj >= t1 + c1 + 105) {
                                      C[(t0 + c0 + 8) * nj + (t1 + c1 + 104)] = private_C[1][13];
                                      if (nj >= t1 + c1 + 113) {
                                        C[(t0 + c0 + 8) * nj + (t1 + c1 + 112)] = private_C[1][14];
                                        if (nj >= t1 + c1 + 121)
                                          C[(t0 + c0 + 8) * nj + (t1 + c1 + 120)] = private_C[1][15];
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
            if (ni >= t0 + c0 + 17) {
              C[(t0 + c0 + 16) * nj + (t1 + c1)] = private_C[2][0];
              if (nj >= t1 + c1 + 9) {
                C[(t0 + c0 + 16) * nj + (t1 + c1 + 8)] = private_C[2][1];
                if (nj >= t1 + c1 + 17) {
                  C[(t0 + c0 + 16) * nj + (t1 + c1 + 16)] = private_C[2][2];
                  if (nj >= t1 + c1 + 25) {
                    C[(t0 + c0 + 16) * nj + (t1 + c1 + 24)] = private_C[2][3];
                    if (nj >= t1 + c1 + 33) {
                      C[(t0 + c0 + 16) * nj + (t1 + c1 + 32)] = private_C[2][4];
                      if (nj >= t1 + c1 + 41) {
                        C[(t0 + c0 + 16) * nj + (t1 + c1 + 40)] = private_C[2][5];
                        if (nj >= t1 + c1 + 49) {
                          C[(t0 + c0 + 16) * nj + (t1 + c1 + 48)] = private_C[2][6];
                          if (nj >= t1 + c1 + 57) {
                            C[(t0 + c0 + 16) * nj + (t1 + c1 + 56)] = private_C[2][7];
                            if (nj >= t1 + c1 + 65) {
                              C[(t0 + c0 + 16) * nj + (t1 + c1 + 64)] = private_C[2][8];
                              if (nj >= t1 + c1 + 73) {
                                C[(t0 + c0 + 16) * nj + (t1 + c1 + 72)] = private_C[2][9];
                                if (nj >= t1 + c1 + 81) {
                                  C[(t0 + c0 + 16) * nj + (t1 + c1 + 80)] = private_C[2][10];
                                  if (nj >= t1 + c1 + 89) {
                                    C[(t0 + c0 + 16) * nj + (t1 + c1 + 88)] = private_C[2][11];
                                    if (nj >= t1 + c1 + 97) {
                                      C[(t0 + c0 + 16) * nj + (t1 + c1 + 96)] = private_C[2][12];
                                      if (nj >= t1 + c1 + 105) {
                                        C[(t0 + c0 + 16) * nj + (t1 + c1 + 104)] = private_C[2][13];
                                        if (nj >= t1 + c1 + 113) {
                                          C[(t0 + c0 + 16) * nj + (t1 + c1 + 112)] = private_C[2][14];
                                          if (nj >= t1 + c1 + 121)
                                            C[(t0 + c0 + 16) * nj + (t1 + c1 + 120)] = private_C[2][15];
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
              if (ni >= t0 + c0 + 25) {
                C[(t0 + c0 + 24) * nj + (t1 + c1)] = private_C[3][0];
                if (nj >= t1 + c1 + 9) {
                  C[(t0 + c0 + 24) * nj + (t1 + c1 + 8)] = private_C[3][1];
                  if (nj >= t1 + c1 + 17) {
                    C[(t0 + c0 + 24) * nj + (t1 + c1 + 16)] = private_C[3][2];
                    if (nj >= t1 + c1 + 25) {
                      C[(t0 + c0 + 24) * nj + (t1 + c1 + 24)] = private_C[3][3];
                      if (nj >= t1 + c1 + 33) {
                        C[(t0 + c0 + 24) * nj + (t1 + c1 + 32)] = private_C[3][4];
                        if (nj >= t1 + c1 + 41) {
                          C[(t0 + c0 + 24) * nj + (t1 + c1 + 40)] = private_C[3][5];
                          if (nj >= t1 + c1 + 49) {
                            C[(t0 + c0 + 24) * nj + (t1 + c1 + 48)] = private_C[3][6];
                            if (nj >= t1 + c1 + 57) {
                              C[(t0 + c0 + 24) * nj + (t1 + c1 + 56)] = private_C[3][7];
                              if (nj >= t1 + c1 + 65) {
                                C[(t0 + c0 + 24) * nj + (t1 + c1 + 64)] = private_C[3][8];
                                if (nj >= t1 + c1 + 73) {
                                  C[(t0 + c0 + 24) * nj + (t1 + c1 + 72)] = private_C[3][9];
                                  if (nj >= t1 + c1 + 81) {
                                    C[(t0 + c0 + 24) * nj + (t1 + c1 + 80)] = private_C[3][10];
                                    if (nj >= t1 + c1 + 89) {
                                      C[(t0 + c0 + 24) * nj + (t1 + c1 + 88)] = private_C[3][11];
                                      if (nj >= t1 + c1 + 97) {
                                        C[(t0 + c0 + 24) * nj + (t1 + c1 + 96)] = private_C[3][12];
                                        if (nj >= t1 + c1 + 105) {
                                          C[(t0 + c0 + 24) * nj + (t1 + c1 + 104)] = private_C[3][13];
                                          if (nj >= t1 + c1 + 113) {
                                            C[(t0 + c0 + 24) * nj + (t1 + c1 + 112)] = private_C[3][14];
                                            if (nj >= t1 + c1 + 121)
                                              C[(t0 + c0 + 24) * nj + (t1 + c1 + 120)] = private_C[3][15];
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
                if (ni >= t0 + c0 + 33) {
                  C[(t0 + c0 + 32) * nj + (t1 + c1)] = private_C[4][0];
                  if (nj >= t1 + c1 + 9) {
                    C[(t0 + c0 + 32) * nj + (t1 + c1 + 8)] = private_C[4][1];
                    if (nj >= t1 + c1 + 17) {
                      C[(t0 + c0 + 32) * nj + (t1 + c1 + 16)] = private_C[4][2];
                      if (nj >= t1 + c1 + 25) {
                        C[(t0 + c0 + 32) * nj + (t1 + c1 + 24)] = private_C[4][3];
                        if (nj >= t1 + c1 + 33) {
                          C[(t0 + c0 + 32) * nj + (t1 + c1 + 32)] = private_C[4][4];
                          if (nj >= t1 + c1 + 41) {
                            C[(t0 + c0 + 32) * nj + (t1 + c1 + 40)] = private_C[4][5];
                            if (nj >= t1 + c1 + 49) {
                              C[(t0 + c0 + 32) * nj + (t1 + c1 + 48)] = private_C[4][6];
                              if (nj >= t1 + c1 + 57) {
                                C[(t0 + c0 + 32) * nj + (t1 + c1 + 56)] = private_C[4][7];
                                if (nj >= t1 + c1 + 65) {
                                  C[(t0 + c0 + 32) * nj + (t1 + c1 + 64)] = private_C[4][8];
                                  if (nj >= t1 + c1 + 73) {
                                    C[(t0 + c0 + 32) * nj + (t1 + c1 + 72)] = private_C[4][9];
                                    if (nj >= t1 + c1 + 81) {
                                      C[(t0 + c0 + 32) * nj + (t1 + c1 + 80)] = private_C[4][10];
                                      if (nj >= t1 + c1 + 89) {
                                        C[(t0 + c0 + 32) * nj + (t1 + c1 + 88)] = private_C[4][11];
                                        if (nj >= t1 + c1 + 97) {
                                          C[(t0 + c0 + 32) * nj + (t1 + c1 + 96)] = private_C[4][12];
                                          if (nj >= t1 + c1 + 105) {
                                            C[(t0 + c0 + 32) * nj + (t1 + c1 + 104)] = private_C[4][13];
                                            if (nj >= t1 + c1 + 113) {
                                              C[(t0 + c0 + 32) * nj + (t1 + c1 + 112)] = private_C[4][14];
                                              if (nj >= t1 + c1 + 121)
                                                C[(t0 + c0 + 32) * nj + (t1 + c1 + 120)] = private_C[4][15];
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                  if (ni >= t0 + c0 + 41) {
                    C[(t0 + c0 + 40) * nj + (t1 + c1)] = private_C[5][0];
                    if (nj >= t1 + c1 + 9) {
                      C[(t0 + c0 + 40) * nj + (t1 + c1 + 8)] = private_C[5][1];
                      if (nj >= t1 + c1 + 17) {
                        C[(t0 + c0 + 40) * nj + (t1 + c1 + 16)] = private_C[5][2];
                        if (nj >= t1 + c1 + 25) {
                          C[(t0 + c0 + 40) * nj + (t1 + c1 + 24)] = private_C[5][3];
                          if (nj >= t1 + c1 + 33) {
                            C[(t0 + c0 + 40) * nj + (t1 + c1 + 32)] = private_C[5][4];
                            if (nj >= t1 + c1 + 41) {
                              C[(t0 + c0 + 40) * nj + (t1 + c1 + 40)] = private_C[5][5];
                              if (nj >= t1 + c1 + 49) {
                                C[(t0 + c0 + 40) * nj + (t1 + c1 + 48)] = private_C[5][6];
                                if (nj >= t1 + c1 + 57) {
                                  C[(t0 + c0 + 40) * nj + (t1 + c1 + 56)] = private_C[5][7];
                                  if (nj >= t1 + c1 + 65) {
                                    C[(t0 + c0 + 40) * nj + (t1 + c1 + 64)] = private_C[5][8];
                                    if (nj >= t1 + c1 + 73) {
                                      C[(t0 + c0 + 40) * nj + (t1 + c1 + 72)] = private_C[5][9];
                                      if (nj >= t1 + c1 + 81) {
                                        C[(t0 + c0 + 40) * nj + (t1 + c1 + 80)] = private_C[5][10];
                                        if (nj >= t1 + c1 + 89) {
                                          C[(t0 + c0 + 40) * nj + (t1 + c1 + 88)] = private_C[5][11];
                                          if (nj >= t1 + c1 + 97) {
                                            C[(t0 + c0 + 40) * nj + (t1 + c1 + 96)] = private_C[5][12];
                                            if (nj >= t1 + c1 + 105) {
                                              C[(t0 + c0 + 40) * nj + (t1 + c1 + 104)] = private_C[5][13];
                                              if (nj >= t1 + c1 + 113) {
                                                C[(t0 + c0 + 40) * nj + (t1 + c1 + 112)] = private_C[5][14];
                                                if (nj >= t1 + c1 + 121)
                                                  C[(t0 + c0 + 40) * nj + (t1 + c1 + 120)] = private_C[5][15];
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                    if (ni >= t0 + c0 + 49) {
                      C[(t0 + c0 + 48) * nj + (t1 + c1)] = private_C[6][0];
                      if (nj >= t1 + c1 + 9) {
                        C[(t0 + c0 + 48) * nj + (t1 + c1 + 8)] = private_C[6][1];
                        if (nj >= t1 + c1 + 17) {
                          C[(t0 + c0 + 48) * nj + (t1 + c1 + 16)] = private_C[6][2];
                          if (nj >= t1 + c1 + 25) {
                            C[(t0 + c0 + 48) * nj + (t1 + c1 + 24)] = private_C[6][3];
                            if (nj >= t1 + c1 + 33) {
                              C[(t0 + c0 + 48) * nj + (t1 + c1 + 32)] = private_C[6][4];
                              if (nj >= t1 + c1 + 41) {
                                C[(t0 + c0 + 48) * nj + (t1 + c1 + 40)] = private_C[6][5];
                                if (nj >= t1 + c1 + 49) {
                                  C[(t0 + c0 + 48) * nj + (t1 + c1 + 48)] = private_C[6][6];
                                  if (nj >= t1 + c1 + 57) {
                                    C[(t0 + c0 + 48) * nj + (t1 + c1 + 56)] = private_C[6][7];
                                    if (nj >= t1 + c1 + 65) {
                                      C[(t0 + c0 + 48) * nj + (t1 + c1 + 64)] = private_C[6][8];
                                      if (nj >= t1 + c1 + 73) {
                                        C[(t0 + c0 + 48) * nj + (t1 + c1 + 72)] = private_C[6][9];
                                        if (nj >= t1 + c1 + 81) {
                                          C[(t0 + c0 + 48) * nj + (t1 + c1 + 80)] = private_C[6][10];
                                          if (nj >= t1 + c1 + 89) {
                                            C[(t0 + c0 + 48) * nj + (t1 + c1 + 88)] = private_C[6][11];
                                            if (nj >= t1 + c1 + 97) {
                                              C[(t0 + c0 + 48) * nj + (t1 + c1 + 96)] = private_C[6][12];
                                              if (nj >= t1 + c1 + 105) {
                                                C[(t0 + c0 + 48) * nj + (t1 + c1 + 104)] = private_C[6][13];
                                                if (nj >= t1 + c1 + 113) {
                                                  C[(t0 + c0 + 48) * nj + (t1 + c1 + 112)] = private_C[6][14];
                                                  if (nj >= t1 + c1 + 121)
                                                    C[(t0 + c0 + 48) * nj + (t1 + c1 + 120)] = private_C[6][15];
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                      if (ni >= t0 + c0 + 57) {
                        C[(t0 + c0 + 56) * nj + (t1 + c1)] = private_C[7][0];
                        if (nj >= t1 + c1 + 9) {
                          C[(t0 + c0 + 56) * nj + (t1 + c1 + 8)] = private_C[7][1];
                          if (nj >= t1 + c1 + 17) {
                            C[(t0 + c0 + 56) * nj + (t1 + c1 + 16)] = private_C[7][2];
                            if (nj >= t1 + c1 + 25) {
                              C[(t0 + c0 + 56) * nj + (t1 + c1 + 24)] = private_C[7][3];
                              if (nj >= t1 + c1 + 33) {
                                C[(t0 + c0 + 56) * nj + (t1 + c1 + 32)] = private_C[7][4];
                                if (nj >= t1 + c1 + 41) {
                                  C[(t0 + c0 + 56) * nj + (t1 + c1 + 40)] = private_C[7][5];
                                  if (nj >= t1 + c1 + 49) {
                                    C[(t0 + c0 + 56) * nj + (t1 + c1 + 48)] = private_C[7][6];
                                    if (nj >= t1 + c1 + 57) {
                                      C[(t0 + c0 + 56) * nj + (t1 + c1 + 56)] = private_C[7][7];
                                      if (nj >= t1 + c1 + 65) {
                                        C[(t0 + c0 + 56) * nj + (t1 + c1 + 64)] = private_C[7][8];
                                        if (nj >= t1 + c1 + 73) {
                                          C[(t0 + c0 + 56) * nj + (t1 + c1 + 72)] = private_C[7][9];
                                          if (nj >= t1 + c1 + 81) {
                                            C[(t0 + c0 + 56) * nj + (t1 + c1 + 80)] = private_C[7][10];
                                            if (nj >= t1 + c1 + 89) {
                                              C[(t0 + c0 + 56) * nj + (t1 + c1 + 88)] = private_C[7][11];
                                              if (nj >= t1 + c1 + 97) {
                                                C[(t0 + c0 + 56) * nj + (t1 + c1 + 96)] = private_C[7][12];
                                                if (nj >= t1 + c1 + 105) {
                                                  C[(t0 + c0 + 56) * nj + (t1 + c1 + 104)] = private_C[7][13];
                                                  if (nj >= t1 + c1 + 113) {
                                                    C[(t0 + c0 + 56) * nj + (t1 + c1 + 112)] = private_C[7][14];
                                                    if (nj >= t1 + c1 + 121)
                                                      C[(t0 + c0 + 56) * nj + (t1 + c1 + 120)] = private_C[7][15];
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                        if (ni >= t0 + c0 + 65) {
                          C[(t0 + c0 + 64) * nj + (t1 + c1)] = private_C[8][0];
                          if (nj >= t1 + c1 + 9) {
                            C[(t0 + c0 + 64) * nj + (t1 + c1 + 8)] = private_C[8][1];
                            if (nj >= t1 + c1 + 17) {
                              C[(t0 + c0 + 64) * nj + (t1 + c1 + 16)] = private_C[8][2];
                              if (nj >= t1 + c1 + 25) {
                                C[(t0 + c0 + 64) * nj + (t1 + c1 + 24)] = private_C[8][3];
                                if (nj >= t1 + c1 + 33) {
                                  C[(t0 + c0 + 64) * nj + (t1 + c1 + 32)] = private_C[8][4];
                                  if (nj >= t1 + c1 + 41) {
                                    C[(t0 + c0 + 64) * nj + (t1 + c1 + 40)] = private_C[8][5];
                                    if (nj >= t1 + c1 + 49) {
                                      C[(t0 + c0 + 64) * nj + (t1 + c1 + 48)] = private_C[8][6];
                                      if (nj >= t1 + c1 + 57) {
                                        C[(t0 + c0 + 64) * nj + (t1 + c1 + 56)] = private_C[8][7];
                                        if (nj >= t1 + c1 + 65) {
                                          C[(t0 + c0 + 64) * nj + (t1 + c1 + 64)] = private_C[8][8];
                                          if (nj >= t1 + c1 + 73) {
                                            C[(t0 + c0 + 64) * nj + (t1 + c1 + 72)] = private_C[8][9];
                                            if (nj >= t1 + c1 + 81) {
                                              C[(t0 + c0 + 64) * nj + (t1 + c1 + 80)] = private_C[8][10];
                                              if (nj >= t1 + c1 + 89) {
                                                C[(t0 + c0 + 64) * nj + (t1 + c1 + 88)] = private_C[8][11];
                                                if (nj >= t1 + c1 + 97) {
                                                  C[(t0 + c0 + 64) * nj + (t1 + c1 + 96)] = private_C[8][12];
                                                  if (nj >= t1 + c1 + 105) {
                                                    C[(t0 + c0 + 64) * nj + (t1 + c1 + 104)] = private_C[8][13];
                                                    if (nj >= t1 + c1 + 113) {
                                                      C[(t0 + c0 + 64) * nj + (t1 + c1 + 112)] = private_C[8][14];
                                                      if (nj >= t1 + c1 + 121)
                                                        C[(t0 + c0 + 64) * nj + (t1 + c1 + 120)] = private_C[8][15];
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                          if (ni >= t0 + c0 + 73) {
                            C[(t0 + c0 + 72) * nj + (t1 + c1)] = private_C[9][0];
                            if (nj >= t1 + c1 + 9) {
                              C[(t0 + c0 + 72) * nj + (t1 + c1 + 8)] = private_C[9][1];
                              if (nj >= t1 + c1 + 17) {
                                C[(t0 + c0 + 72) * nj + (t1 + c1 + 16)] = private_C[9][2];
                                if (nj >= t1 + c1 + 25) {
                                  C[(t0 + c0 + 72) * nj + (t1 + c1 + 24)] = private_C[9][3];
                                  if (nj >= t1 + c1 + 33) {
                                    C[(t0 + c0 + 72) * nj + (t1 + c1 + 32)] = private_C[9][4];
                                    if (nj >= t1 + c1 + 41) {
                                      C[(t0 + c0 + 72) * nj + (t1 + c1 + 40)] = private_C[9][5];
                                      if (nj >= t1 + c1 + 49) {
                                        C[(t0 + c0 + 72) * nj + (t1 + c1 + 48)] = private_C[9][6];
                                        if (nj >= t1 + c1 + 57) {
                                          C[(t0 + c0 + 72) * nj + (t1 + c1 + 56)] = private_C[9][7];
                                          if (nj >= t1 + c1 + 65) {
                                            C[(t0 + c0 + 72) * nj + (t1 + c1 + 64)] = private_C[9][8];
                                            if (nj >= t1 + c1 + 73) {
                                              C[(t0 + c0 + 72) * nj + (t1 + c1 + 72)] = private_C[9][9];
                                              if (nj >= t1 + c1 + 81) {
                                                C[(t0 + c0 + 72) * nj + (t1 + c1 + 80)] = private_C[9][10];
                                                if (nj >= t1 + c1 + 89) {
                                                  C[(t0 + c0 + 72) * nj + (t1 + c1 + 88)] = private_C[9][11];
                                                  if (nj >= t1 + c1 + 97) {
                                                    C[(t0 + c0 + 72) * nj + (t1 + c1 + 96)] = private_C[9][12];
                                                    if (nj >= t1 + c1 + 105) {
                                                      C[(t0 + c0 + 72) * nj + (t1 + c1 + 104)] = private_C[9][13];
                                                      if (nj >= t1 + c1 + 113) {
                                                        C[(t0 + c0 + 72) * nj + (t1 + c1 + 112)] = private_C[9][14];
                                                        if (nj >= t1 + c1 + 121)
                                                          C[(t0 + c0 + 72) * nj + (t1 + c1 + 120)] = private_C[9][15];
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                            if (ni >= t0 + c0 + 81) {
                              C[(t0 + c0 + 80) * nj + (t1 + c1)] = private_C[10][0];
                              if (nj >= t1 + c1 + 9) {
                                C[(t0 + c0 + 80) * nj + (t1 + c1 + 8)] = private_C[10][1];
                                if (nj >= t1 + c1 + 17) {
                                  C[(t0 + c0 + 80) * nj + (t1 + c1 + 16)] = private_C[10][2];
                                  if (nj >= t1 + c1 + 25) {
                                    C[(t0 + c0 + 80) * nj + (t1 + c1 + 24)] = private_C[10][3];
                                    if (nj >= t1 + c1 + 33) {
                                      C[(t0 + c0 + 80) * nj + (t1 + c1 + 32)] = private_C[10][4];
                                      if (nj >= t1 + c1 + 41) {
                                        C[(t0 + c0 + 80) * nj + (t1 + c1 + 40)] = private_C[10][5];
                                        if (nj >= t1 + c1 + 49) {
                                          C[(t0 + c0 + 80) * nj + (t1 + c1 + 48)] = private_C[10][6];
                                          if (nj >= t1 + c1 + 57) {
                                            C[(t0 + c0 + 80) * nj + (t1 + c1 + 56)] = private_C[10][7];
                                            if (nj >= t1 + c1 + 65) {
                                              C[(t0 + c0 + 80) * nj + (t1 + c1 + 64)] = private_C[10][8];
                                              if (nj >= t1 + c1 + 73) {
                                                C[(t0 + c0 + 80) * nj + (t1 + c1 + 72)] = private_C[10][9];
                                                if (nj >= t1 + c1 + 81) {
                                                  C[(t0 + c0 + 80) * nj + (t1 + c1 + 80)] = private_C[10][10];
                                                  if (nj >= t1 + c1 + 89) {
                                                    C[(t0 + c0 + 80) * nj + (t1 + c1 + 88)] = private_C[10][11];
                                                    if (nj >= t1 + c1 + 97) {
                                                      C[(t0 + c0 + 80) * nj + (t1 + c1 + 96)] = private_C[10][12];
                                                      if (nj >= t1 + c1 + 105) {
                                                        C[(t0 + c0 + 80) * nj + (t1 + c1 + 104)] = private_C[10][13];
                                                        if (nj >= t1 + c1 + 113) {
                                                          C[(t0 + c0 + 80) * nj + (t1 + c1 + 112)] = private_C[10][14];
                                                          if (nj >= t1 + c1 + 121)
                                                            C[(t0 + c0 + 80) * nj + (t1 + c1 + 120)] = private_C[10][15];
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                              if (ni >= t0 + c0 + 89) {
                                C[(t0 + c0 + 88) * nj + (t1 + c1)] = private_C[11][0];
                                if (nj >= t1 + c1 + 9) {
                                  C[(t0 + c0 + 88) * nj + (t1 + c1 + 8)] = private_C[11][1];
                                  if (nj >= t1 + c1 + 17) {
                                    C[(t0 + c0 + 88) * nj + (t1 + c1 + 16)] = private_C[11][2];
                                    if (nj >= t1 + c1 + 25) {
                                      C[(t0 + c0 + 88) * nj + (t1 + c1 + 24)] = private_C[11][3];
                                      if (nj >= t1 + c1 + 33) {
                                        C[(t0 + c0 + 88) * nj + (t1 + c1 + 32)] = private_C[11][4];
                                        if (nj >= t1 + c1 + 41) {
                                          C[(t0 + c0 + 88) * nj + (t1 + c1 + 40)] = private_C[11][5];
                                          if (nj >= t1 + c1 + 49) {
                                            C[(t0 + c0 + 88) * nj + (t1 + c1 + 48)] = private_C[11][6];
                                            if (nj >= t1 + c1 + 57) {
                                              C[(t0 + c0 + 88) * nj + (t1 + c1 + 56)] = private_C[11][7];
                                              if (nj >= t1 + c1 + 65) {
                                                C[(t0 + c0 + 88) * nj + (t1 + c1 + 64)] = private_C[11][8];
                                                if (nj >= t1 + c1 + 73) {
                                                  C[(t0 + c0 + 88) * nj + (t1 + c1 + 72)] = private_C[11][9];
                                                  if (nj >= t1 + c1 + 81) {
                                                    C[(t0 + c0 + 88) * nj + (t1 + c1 + 80)] = private_C[11][10];
                                                    if (nj >= t1 + c1 + 89) {
                                                      C[(t0 + c0 + 88) * nj + (t1 + c1 + 88)] = private_C[11][11];
                                                      if (nj >= t1 + c1 + 97) {
                                                        C[(t0 + c0 + 88) * nj + (t1 + c1 + 96)] = private_C[11][12];
                                                        if (nj >= t1 + c1 + 105) {
                                                          C[(t0 + c0 + 88) * nj + (t1 + c1 + 104)] = private_C[11][13];
                                                          if (nj >= t1 + c1 + 113) {
                                                            C[(t0 + c0 + 88) * nj + (t1 + c1 + 112)] = private_C[11][14];
                                                            if (nj >= t1 + c1 + 121)
                                                              C[(t0 + c0 + 88) * nj + (t1 + c1 + 120)] = private_C[11][15];
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                                if (ni >= t0 + c0 + 97) {
                                  C[(t0 + c0 + 96) * nj + (t1 + c1)] = private_C[12][0];
                                  if (nj >= t1 + c1 + 9) {
                                    C[(t0 + c0 + 96) * nj + (t1 + c1 + 8)] = private_C[12][1];
                                    if (nj >= t1 + c1 + 17) {
                                      C[(t0 + c0 + 96) * nj + (t1 + c1 + 16)] = private_C[12][2];
                                      if (nj >= t1 + c1 + 25) {
                                        C[(t0 + c0 + 96) * nj + (t1 + c1 + 24)] = private_C[12][3];
                                        if (nj >= t1 + c1 + 33) {
                                          C[(t0 + c0 + 96) * nj + (t1 + c1 + 32)] = private_C[12][4];
                                          if (nj >= t1 + c1 + 41) {
                                            C[(t0 + c0 + 96) * nj + (t1 + c1 + 40)] = private_C[12][5];
                                            if (nj >= t1 + c1 + 49) {
                                              C[(t0 + c0 + 96) * nj + (t1 + c1 + 48)] = private_C[12][6];
                                              if (nj >= t1 + c1 + 57) {
                                                C[(t0 + c0 + 96) * nj + (t1 + c1 + 56)] = private_C[12][7];
                                                if (nj >= t1 + c1 + 65) {
                                                  C[(t0 + c0 + 96) * nj + (t1 + c1 + 64)] = private_C[12][8];
                                                  if (nj >= t1 + c1 + 73) {
                                                    C[(t0 + c0 + 96) * nj + (t1 + c1 + 72)] = private_C[12][9];
                                                    if (nj >= t1 + c1 + 81) {
                                                      C[(t0 + c0 + 96) * nj + (t1 + c1 + 80)] = private_C[12][10];
                                                      if (nj >= t1 + c1 + 89) {
                                                        C[(t0 + c0 + 96) * nj + (t1 + c1 + 88)] = private_C[12][11];
                                                        if (nj >= t1 + c1 + 97) {
                                                          C[(t0 + c0 + 96) * nj + (t1 + c1 + 96)] = private_C[12][12];
                                                          if (nj >= t1 + c1 + 105) {
                                                            C[(t0 + c0 + 96) * nj + (t1 + c1 + 104)] = private_C[12][13];
                                                            if (nj >= t1 + c1 + 113) {
                                                              C[(t0 + c0 + 96) * nj + (t1 + c1 + 112)] = private_C[12][14];
                                                              if (nj >= t1 + c1 + 121)
                                                                C[(t0 + c0 + 96) * nj + (t1 + c1 + 120)] = private_C[12][15];
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                  if (ni >= t0 + c0 + 105) {
                                    C[(t0 + c0 + 104) * nj + (t1 + c1)] = private_C[13][0];
                                    if (nj >= t1 + c1 + 9) {
                                      C[(t0 + c0 + 104) * nj + (t1 + c1 + 8)] = private_C[13][1];
                                      if (nj >= t1 + c1 + 17) {
                                        C[(t0 + c0 + 104) * nj + (t1 + c1 + 16)] = private_C[13][2];
                                        if (nj >= t1 + c1 + 25) {
                                          C[(t0 + c0 + 104) * nj + (t1 + c1 + 24)] = private_C[13][3];
                                          if (nj >= t1 + c1 + 33) {
                                            C[(t0 + c0 + 104) * nj + (t1 + c1 + 32)] = private_C[13][4];
                                            if (nj >= t1 + c1 + 41) {
                                              C[(t0 + c0 + 104) * nj + (t1 + c1 + 40)] = private_C[13][5];
                                              if (nj >= t1 + c1 + 49) {
                                                C[(t0 + c0 + 104) * nj + (t1 + c1 + 48)] = private_C[13][6];
                                                if (nj >= t1 + c1 + 57) {
                                                  C[(t0 + c0 + 104) * nj + (t1 + c1 + 56)] = private_C[13][7];
                                                  if (nj >= t1 + c1 + 65) {
                                                    C[(t0 + c0 + 104) * nj + (t1 + c1 + 64)] = private_C[13][8];
                                                    if (nj >= t1 + c1 + 73) {
                                                      C[(t0 + c0 + 104) * nj + (t1 + c1 + 72)] = private_C[13][9];
                                                      if (nj >= t1 + c1 + 81) {
                                                        C[(t0 + c0 + 104) * nj + (t1 + c1 + 80)] = private_C[13][10];
                                                        if (nj >= t1 + c1 + 89) {
                                                          C[(t0 + c0 + 104) * nj + (t1 + c1 + 88)] = private_C[13][11];
                                                          if (nj >= t1 + c1 + 97) {
                                                            C[(t0 + c0 + 104) * nj + (t1 + c1 + 96)] = private_C[13][12];
                                                            if (nj >= t1 + c1 + 105) {
                                                              C[(t0 + c0 + 104) * nj + (t1 + c1 + 104)] = private_C[13][13];
                                                              if (nj >= t1 + c1 + 113) {
                                                                C[(t0 + c0 + 104) * nj + (t1 + c1 + 112)] = private_C[13][14];
                                                                if (nj >= t1 + c1 + 121)
                                                                  C[(t0 + c0 + 104) * nj + (t1 + c1 + 120)] = private_C[13][15];
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                    if (ni >= t0 + c0 + 113) {
                                      C[(t0 + c0 + 112) * nj + (t1 + c1)] = private_C[14][0];
                                      if (nj >= t1 + c1 + 9) {
                                        C[(t0 + c0 + 112) * nj + (t1 + c1 + 8)] = private_C[14][1];
                                        if (nj >= t1 + c1 + 17) {
                                          C[(t0 + c0 + 112) * nj + (t1 + c1 + 16)] = private_C[14][2];
                                          if (nj >= t1 + c1 + 25) {
                                            C[(t0 + c0 + 112) * nj + (t1 + c1 + 24)] = private_C[14][3];
                                            if (nj >= t1 + c1 + 33) {
                                              C[(t0 + c0 + 112) * nj + (t1 + c1 + 32)] = private_C[14][4];
                                              if (nj >= t1 + c1 + 41) {
                                                C[(t0 + c0 + 112) * nj + (t1 + c1 + 40)] = private_C[14][5];
                                                if (nj >= t1 + c1 + 49) {
                                                  C[(t0 + c0 + 112) * nj + (t1 + c1 + 48)] = private_C[14][6];
                                                  if (nj >= t1 + c1 + 57) {
                                                    C[(t0 + c0 + 112) * nj + (t1 + c1 + 56)] = private_C[14][7];
                                                    if (nj >= t1 + c1 + 65) {
                                                      C[(t0 + c0 + 112) * nj + (t1 + c1 + 64)] = private_C[14][8];
                                                      if (nj >= t1 + c1 + 73) {
                                                        C[(t0 + c0 + 112) * nj + (t1 + c1 + 72)] = private_C[14][9];
                                                        if (nj >= t1 + c1 + 81) {
                                                          C[(t0 + c0 + 112) * nj + (t1 + c1 + 80)] = private_C[14][10];
                                                          if (nj >= t1 + c1 + 89) {
                                                            C[(t0 + c0 + 112) * nj + (t1 + c1 + 88)] = private_C[14][11];
                                                            if (nj >= t1 + c1 + 97) {
                                                              C[(t0 + c0 + 112) * nj + (t1 + c1 + 96)] = private_C[14][12];
                                                              if (nj >= t1 + c1 + 105) {
                                                                C[(t0 + c0 + 112) * nj + (t1 + c1 + 104)] = private_C[14][13];
                                                                if (nj >= t1 + c1 + 113) {
                                                                  C[(t0 + c0 + 112) * nj + (t1 + c1 + 112)] = private_C[14][14];
                                                                  if (nj >= t1 + c1 + 121)
                                                                    C[(t0 + c0 + 112) * nj + (t1 + c1 + 120)] = private_C[14][15];
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                      if (ni >= t0 + c0 + 121) {
                                        C[(t0 + c0 + 120) * nj + (t1 + c1)] = private_C[15][0];
                                        if (nj >= t1 + c1 + 9) {
                                          C[(t0 + c0 + 120) * nj + (t1 + c1 + 8)] = private_C[15][1];
                                          if (nj >= t1 + c1 + 17) {
                                            C[(t0 + c0 + 120) * nj + (t1 + c1 + 16)] = private_C[15][2];
                                            if (nj >= t1 + c1 + 25) {
                                              C[(t0 + c0 + 120) * nj + (t1 + c1 + 24)] = private_C[15][3];
                                              if (nj >= t1 + c1 + 33) {
                                                C[(t0 + c0 + 120) * nj + (t1 + c1 + 32)] = private_C[15][4];
                                                if (nj >= t1 + c1 + 41) {
                                                  C[(t0 + c0 + 120) * nj + (t1 + c1 + 40)] = private_C[15][5];
                                                  if (nj >= t1 + c1 + 49) {
                                                    C[(t0 + c0 + 120) * nj + (t1 + c1 + 48)] = private_C[15][6];
                                                    if (nj >= t1 + c1 + 57) {
                                                      C[(t0 + c0 + 120) * nj + (t1 + c1 + 56)] = private_C[15][7];
                                                      if (nj >= t1 + c1 + 65) {
                                                        C[(t0 + c0 + 120) * nj + (t1 + c1 + 64)] = private_C[15][8];
                                                        if (nj >= t1 + c1 + 73) {
                                                          C[(t0 + c0 + 120) * nj + (t1 + c1 + 72)] = private_C[15][9];
                                                          if (nj >= t1 + c1 + 81) {
                                                            C[(t0 + c0 + 120) * nj + (t1 + c1 + 80)] = private_C[15][10];
                                                            if (nj >= t1 + c1 + 89) {
                                                              C[(t0 + c0 + 120) * nj + (t1 + c1 + 88)] = private_C[15][11];
                                                              if (nj >= t1 + c1 + 97) {
                                                                C[(t0 + c0 + 120) * nj + (t1 + c1 + 96)] = private_C[15][12];
                                                                if (nj >= t1 + c1 + 105) {
                                                                  C[(t0 + c0 + 120) * nj + (t1 + c1 + 104)] = private_C[15][13];
                                                                  if (nj >= t1 + c1 + 113) {
                                                                    C[(t0 + c0 + 120) * nj + (t1 + c1 + 112)] = private_C[15][14];
                                                                    if (nj >= t1 + c1 + 121)
                                                                      C[(t0 + c0 + 120) * nj + (t1 + c1 + 120)] = private_C[15][15];
                                                                  }
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
        }
        __syncthreads();
      }
}
