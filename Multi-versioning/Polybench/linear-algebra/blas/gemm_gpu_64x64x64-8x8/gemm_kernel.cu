#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int nj, int nk, int ni)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    double private_C[8][8];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 64 * b0; c0 < ni; c0 += 16384)
      for (int c1 = 64 * b1; c1 < nj; c1 += 16384) {
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          private_C[0][0] = C[(t0 + c0) * nj + (t1 + c1)];
          if (nj >= t1 + c1 + 9) {
            private_C[0][1] = C[(t0 + c0) * nj + (t1 + c1 + 8)];
            if (nj >= t1 + c1 + 17) {
              private_C[0][2] = C[(t0 + c0) * nj + (t1 + c1 + 16)];
              if (nj >= t1 + c1 + 25) {
                private_C[0][3] = C[(t0 + c0) * nj + (t1 + c1 + 24)];
                if (nj >= t1 + c1 + 33) {
                  private_C[0][4] = C[(t0 + c0) * nj + (t1 + c1 + 32)];
                  if (nj >= t1 + c1 + 41) {
                    private_C[0][5] = C[(t0 + c0) * nj + (t1 + c1 + 40)];
                    if (nj >= t1 + c1 + 49) {
                      private_C[0][6] = C[(t0 + c0) * nj + (t1 + c1 + 48)];
                      if (nj >= t1 + c1 + 57)
                        private_C[0][7] = C[(t0 + c0) * nj + (t1 + c1 + 56)];
                    }
                  }
                }
              }
            }
          }
          if (ni >= t0 + c0 + 9) {
            private_C[1][0] = C[(t0 + c0 + 8) * nj + (t1 + c1)];
            if (nj >= t1 + c1 + 9) {
              private_C[1][1] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 8)];
              if (nj >= t1 + c1 + 17) {
                private_C[1][2] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 16)];
                if (nj >= t1 + c1 + 25) {
                  private_C[1][3] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 24)];
                  if (nj >= t1 + c1 + 33) {
                    private_C[1][4] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 32)];
                    if (nj >= t1 + c1 + 41) {
                      private_C[1][5] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 40)];
                      if (nj >= t1 + c1 + 49) {
                        private_C[1][6] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 48)];
                        if (nj >= t1 + c1 + 57)
                          private_C[1][7] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 56)];
                      }
                    }
                  }
                }
              }
            }
            if (ni >= t0 + c0 + 17) {
              private_C[2][0] = C[(t0 + c0 + 16) * nj + (t1 + c1)];
              if (nj >= t1 + c1 + 9) {
                private_C[2][1] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 8)];
                if (nj >= t1 + c1 + 17) {
                  private_C[2][2] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 16)];
                  if (nj >= t1 + c1 + 25) {
                    private_C[2][3] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 24)];
                    if (nj >= t1 + c1 + 33) {
                      private_C[2][4] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 32)];
                      if (nj >= t1 + c1 + 41) {
                        private_C[2][5] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 40)];
                        if (nj >= t1 + c1 + 49) {
                          private_C[2][6] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 48)];
                          if (nj >= t1 + c1 + 57)
                            private_C[2][7] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 56)];
                        }
                      }
                    }
                  }
                }
              }
              if (ni >= t0 + c0 + 25) {
                private_C[3][0] = C[(t0 + c0 + 24) * nj + (t1 + c1)];
                if (nj >= t1 + c1 + 9) {
                  private_C[3][1] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 8)];
                  if (nj >= t1 + c1 + 17) {
                    private_C[3][2] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 16)];
                    if (nj >= t1 + c1 + 25) {
                      private_C[3][3] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 24)];
                      if (nj >= t1 + c1 + 33) {
                        private_C[3][4] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 32)];
                        if (nj >= t1 + c1 + 41) {
                          private_C[3][5] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 40)];
                          if (nj >= t1 + c1 + 49) {
                            private_C[3][6] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 48)];
                            if (nj >= t1 + c1 + 57)
                              private_C[3][7] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 56)];
                          }
                        }
                      }
                    }
                  }
                }
                if (ni >= t0 + c0 + 33) {
                  private_C[4][0] = C[(t0 + c0 + 32) * nj + (t1 + c1)];
                  if (nj >= t1 + c1 + 9) {
                    private_C[4][1] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 8)];
                    if (nj >= t1 + c1 + 17) {
                      private_C[4][2] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 16)];
                      if (nj >= t1 + c1 + 25) {
                        private_C[4][3] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 24)];
                        if (nj >= t1 + c1 + 33) {
                          private_C[4][4] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 32)];
                          if (nj >= t1 + c1 + 41) {
                            private_C[4][5] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 40)];
                            if (nj >= t1 + c1 + 49) {
                              private_C[4][6] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 48)];
                              if (nj >= t1 + c1 + 57)
                                private_C[4][7] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 56)];
                            }
                          }
                        }
                      }
                    }
                  }
                  if (ni >= t0 + c0 + 41) {
                    private_C[5][0] = C[(t0 + c0 + 40) * nj + (t1 + c1)];
                    if (nj >= t1 + c1 + 9) {
                      private_C[5][1] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 8)];
                      if (nj >= t1 + c1 + 17) {
                        private_C[5][2] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 16)];
                        if (nj >= t1 + c1 + 25) {
                          private_C[5][3] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 24)];
                          if (nj >= t1 + c1 + 33) {
                            private_C[5][4] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 32)];
                            if (nj >= t1 + c1 + 41) {
                              private_C[5][5] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 40)];
                              if (nj >= t1 + c1 + 49) {
                                private_C[5][6] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 48)];
                                if (nj >= t1 + c1 + 57)
                                  private_C[5][7] = C[(t0 + c0 + 40) * nj + (t1 + c1 + 56)];
                              }
                            }
                          }
                        }
                      }
                    }
                    if (ni >= t0 + c0 + 49) {
                      private_C[6][0] = C[(t0 + c0 + 48) * nj + (t1 + c1)];
                      if (nj >= t1 + c1 + 9) {
                        private_C[6][1] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 8)];
                        if (nj >= t1 + c1 + 17) {
                          private_C[6][2] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 16)];
                          if (nj >= t1 + c1 + 25) {
                            private_C[6][3] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 24)];
                            if (nj >= t1 + c1 + 33) {
                              private_C[6][4] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 32)];
                              if (nj >= t1 + c1 + 41) {
                                private_C[6][5] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 40)];
                                if (nj >= t1 + c1 + 49) {
                                  private_C[6][6] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 48)];
                                  if (nj >= t1 + c1 + 57)
                                    private_C[6][7] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 56)];
                                }
                              }
                            }
                          }
                        }
                      }
                      if (ni >= t0 + c0 + 57) {
                        private_C[7][0] = C[(t0 + c0 + 56) * nj + (t1 + c1)];
                        if (nj >= t1 + c1 + 9) {
                          private_C[7][1] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 8)];
                          if (nj >= t1 + c1 + 17) {
                            private_C[7][2] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 16)];
                            if (nj >= t1 + c1 + 25) {
                              private_C[7][3] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 24)];
                              if (nj >= t1 + c1 + 33) {
                                private_C[7][4] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 32)];
                                if (nj >= t1 + c1 + 41) {
                                  private_C[7][5] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 40)];
                                  if (nj >= t1 + c1 + 49) {
                                    private_C[7][6] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 48)];
                                    if (nj >= t1 + c1 + 57)
                                      private_C[7][7] = C[(t0 + c0 + 56) * nj + (t1 + c1 + 56)];
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          private_C[0][0] *= beta;
          if (nj >= t1 + c1 + 9) {
            private_C[0][1] *= beta;
            if (nj >= t1 + c1 + 17) {
              private_C[0][2] *= beta;
              if (nj >= t1 + c1 + 25) {
                private_C[0][3] *= beta;
                if (nj >= t1 + c1 + 33) {
                  private_C[0][4] *= beta;
                  if (nj >= t1 + c1 + 41) {
                    private_C[0][5] *= beta;
                    if (nj >= t1 + c1 + 49) {
                      private_C[0][6] *= beta;
                      if (nj >= t1 + c1 + 57)
                        private_C[0][7] *= beta;
                    }
                  }
                }
              }
            }
          }
          if (ni >= t0 + c0 + 9) {
            private_C[1][0] *= beta;
            if (nj >= t1 + c1 + 9) {
              private_C[1][1] *= beta;
              if (nj >= t1 + c1 + 17) {
                private_C[1][2] *= beta;
                if (nj >= t1 + c1 + 25) {
                  private_C[1][3] *= beta;
                  if (nj >= t1 + c1 + 33) {
                    private_C[1][4] *= beta;
                    if (nj >= t1 + c1 + 41) {
                      private_C[1][5] *= beta;
                      if (nj >= t1 + c1 + 49) {
                        private_C[1][6] *= beta;
                        if (nj >= t1 + c1 + 57)
                          private_C[1][7] *= beta;
                      }
                    }
                  }
                }
              }
            }
            if (ni >= t0 + c0 + 17) {
              private_C[2][0] *= beta;
              if (nj >= t1 + c1 + 9) {
                private_C[2][1] *= beta;
                if (nj >= t1 + c1 + 17) {
                  private_C[2][2] *= beta;
                  if (nj >= t1 + c1 + 25) {
                    private_C[2][3] *= beta;
                    if (nj >= t1 + c1 + 33) {
                      private_C[2][4] *= beta;
                      if (nj >= t1 + c1 + 41) {
                        private_C[2][5] *= beta;
                        if (nj >= t1 + c1 + 49) {
                          private_C[2][6] *= beta;
                          if (nj >= t1 + c1 + 57)
                            private_C[2][7] *= beta;
                        }
                      }
                    }
                  }
                }
              }
              if (ni >= t0 + c0 + 25) {
                private_C[3][0] *= beta;
                if (nj >= t1 + c1 + 9) {
                  private_C[3][1] *= beta;
                  if (nj >= t1 + c1 + 17) {
                    private_C[3][2] *= beta;
                    if (nj >= t1 + c1 + 25) {
                      private_C[3][3] *= beta;
                      if (nj >= t1 + c1 + 33) {
                        private_C[3][4] *= beta;
                        if (nj >= t1 + c1 + 41) {
                          private_C[3][5] *= beta;
                          if (nj >= t1 + c1 + 49) {
                            private_C[3][6] *= beta;
                            if (nj >= t1 + c1 + 57)
                              private_C[3][7] *= beta;
                          }
                        }
                      }
                    }
                  }
                }
                if (ni >= t0 + c0 + 33) {
                  private_C[4][0] *= beta;
                  if (nj >= t1 + c1 + 9) {
                    private_C[4][1] *= beta;
                    if (nj >= t1 + c1 + 17) {
                      private_C[4][2] *= beta;
                      if (nj >= t1 + c1 + 25) {
                        private_C[4][3] *= beta;
                        if (nj >= t1 + c1 + 33) {
                          private_C[4][4] *= beta;
                          if (nj >= t1 + c1 + 41) {
                            private_C[4][5] *= beta;
                            if (nj >= t1 + c1 + 49) {
                              private_C[4][6] *= beta;
                              if (nj >= t1 + c1 + 57)
                                private_C[4][7] *= beta;
                            }
                          }
                        }
                      }
                    }
                  }
                  if (ni >= t0 + c0 + 41) {
                    private_C[5][0] *= beta;
                    if (nj >= t1 + c1 + 9) {
                      private_C[5][1] *= beta;
                      if (nj >= t1 + c1 + 17) {
                        private_C[5][2] *= beta;
                        if (nj >= t1 + c1 + 25) {
                          private_C[5][3] *= beta;
                          if (nj >= t1 + c1 + 33) {
                            private_C[5][4] *= beta;
                            if (nj >= t1 + c1 + 41) {
                              private_C[5][5] *= beta;
                              if (nj >= t1 + c1 + 49) {
                                private_C[5][6] *= beta;
                                if (nj >= t1 + c1 + 57)
                                  private_C[5][7] *= beta;
                              }
                            }
                          }
                        }
                      }
                    }
                    if (ni >= t0 + c0 + 49) {
                      private_C[6][0] *= beta;
                      if (nj >= t1 + c1 + 9) {
                        private_C[6][1] *= beta;
                        if (nj >= t1 + c1 + 17) {
                          private_C[6][2] *= beta;
                          if (nj >= t1 + c1 + 25) {
                            private_C[6][3] *= beta;
                            if (nj >= t1 + c1 + 33) {
                              private_C[6][4] *= beta;
                              if (nj >= t1 + c1 + 41) {
                                private_C[6][5] *= beta;
                                if (nj >= t1 + c1 + 49) {
                                  private_C[6][6] *= beta;
                                  if (nj >= t1 + c1 + 57)
                                    private_C[6][7] *= beta;
                                }
                              }
                            }
                          }
                        }
                      }
                      if (ni >= t0 + c0 + 57) {
                        private_C[7][0] *= beta;
                        if (nj >= t1 + c1 + 9) {
                          private_C[7][1] *= beta;
                          if (nj >= t1 + c1 + 17) {
                            private_C[7][2] *= beta;
                            if (nj >= t1 + c1 + 25) {
                              private_C[7][3] *= beta;
                              if (nj >= t1 + c1 + 33) {
                                private_C[7][4] *= beta;
                                if (nj >= t1 + c1 + 41) {
                                  private_C[7][5] *= beta;
                                  if (nj >= t1 + c1 + 49) {
                                    private_C[7][6] *= beta;
                                    if (nj >= t1 + c1 + 57)
                                      private_C[7][7] *= beta;
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          for (int c2 = 0; c2 < nk; c2 += 64)
            for (int c3 = 0; c3 <= ppcg_min(63, nk - c2 - 1); c3 += 1) {
              private_C[0][0] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
              if (nj >= t1 + c1 + 9) {
                private_C[0][1] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                if (nj >= t1 + c1 + 17) {
                  private_C[0][2] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                  if (nj >= t1 + c1 + 25) {
                    private_C[0][3] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                    if (nj >= t1 + c1 + 33) {
                      private_C[0][4] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                      if (nj >= t1 + c1 + 41) {
                        private_C[0][5] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                        if (nj >= t1 + c1 + 49) {
                          private_C[0][6] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                          if (nj >= t1 + c1 + 57)
                            private_C[0][7] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                        }
                      }
                    }
                  }
                }
              }
              if (ni >= t0 + c0 + 9) {
                private_C[1][0] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                if (nj >= t1 + c1 + 9) {
                  private_C[1][1] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                  if (nj >= t1 + c1 + 17) {
                    private_C[1][2] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                    if (nj >= t1 + c1 + 25) {
                      private_C[1][3] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                      if (nj >= t1 + c1 + 33) {
                        private_C[1][4] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                        if (nj >= t1 + c1 + 41) {
                          private_C[1][5] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                          if (nj >= t1 + c1 + 49) {
                            private_C[1][6] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                            if (nj >= t1 + c1 + 57)
                              private_C[1][7] += ((alpha * A[(t0 + c0 + 8) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                          }
                        }
                      }
                    }
                  }
                }
                if (ni >= t0 + c0 + 17) {
                  private_C[2][0] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                  if (nj >= t1 + c1 + 9) {
                    private_C[2][1] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                    if (nj >= t1 + c1 + 17) {
                      private_C[2][2] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                      if (nj >= t1 + c1 + 25) {
                        private_C[2][3] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                        if (nj >= t1 + c1 + 33) {
                          private_C[2][4] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                          if (nj >= t1 + c1 + 41) {
                            private_C[2][5] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                            if (nj >= t1 + c1 + 49) {
                              private_C[2][6] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                              if (nj >= t1 + c1 + 57)
                                private_C[2][7] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                            }
                          }
                        }
                      }
                    }
                  }
                  if (ni >= t0 + c0 + 25) {
                    private_C[3][0] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                    if (nj >= t1 + c1 + 9) {
                      private_C[3][1] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                      if (nj >= t1 + c1 + 17) {
                        private_C[3][2] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                        if (nj >= t1 + c1 + 25) {
                          private_C[3][3] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                          if (nj >= t1 + c1 + 33) {
                            private_C[3][4] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                            if (nj >= t1 + c1 + 41) {
                              private_C[3][5] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                              if (nj >= t1 + c1 + 49) {
                                private_C[3][6] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                if (nj >= t1 + c1 + 57)
                                  private_C[3][7] += ((alpha * A[(t0 + c0 + 24) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                              }
                            }
                          }
                        }
                      }
                    }
                    if (ni >= t0 + c0 + 33) {
                      private_C[4][0] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                      if (nj >= t1 + c1 + 9) {
                        private_C[4][1] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                        if (nj >= t1 + c1 + 17) {
                          private_C[4][2] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                          if (nj >= t1 + c1 + 25) {
                            private_C[4][3] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                            if (nj >= t1 + c1 + 33) {
                              private_C[4][4] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                              if (nj >= t1 + c1 + 41) {
                                private_C[4][5] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                if (nj >= t1 + c1 + 49) {
                                  private_C[4][6] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                  if (nj >= t1 + c1 + 57)
                                    private_C[4][7] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                }
                              }
                            }
                          }
                        }
                      }
                      if (ni >= t0 + c0 + 41) {
                        private_C[5][0] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                        if (nj >= t1 + c1 + 9) {
                          private_C[5][1] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                          if (nj >= t1 + c1 + 17) {
                            private_C[5][2] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                            if (nj >= t1 + c1 + 25) {
                              private_C[5][3] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                              if (nj >= t1 + c1 + 33) {
                                private_C[5][4] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                if (nj >= t1 + c1 + 41) {
                                  private_C[5][5] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                  if (nj >= t1 + c1 + 49) {
                                    private_C[5][6] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                    if (nj >= t1 + c1 + 57)
                                      private_C[5][7] += ((alpha * A[(t0 + c0 + 40) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                  }
                                }
                              }
                            }
                          }
                        }
                        if (ni >= t0 + c0 + 49) {
                          private_C[6][0] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                          if (nj >= t1 + c1 + 9) {
                            private_C[6][1] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                            if (nj >= t1 + c1 + 17) {
                              private_C[6][2] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                              if (nj >= t1 + c1 + 25) {
                                private_C[6][3] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                                if (nj >= t1 + c1 + 33) {
                                  private_C[6][4] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                  if (nj >= t1 + c1 + 41) {
                                    private_C[6][5] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                    if (nj >= t1 + c1 + 49) {
                                      private_C[6][6] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                      if (nj >= t1 + c1 + 57)
                                        private_C[6][7] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                    }
                                  }
                                }
                              }
                            }
                          }
                          if (ni >= t0 + c0 + 57) {
                            private_C[7][0] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                            if (nj >= t1 + c1 + 9) {
                              private_C[7][1] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                              if (nj >= t1 + c1 + 17) {
                                private_C[7][2] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                                if (nj >= t1 + c1 + 25) {
                                  private_C[7][3] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                                  if (nj >= t1 + c1 + 33) {
                                    private_C[7][4] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                    if (nj >= t1 + c1 + 41) {
                                      private_C[7][5] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 40)]);
                                      if (nj >= t1 + c1 + 49) {
                                        private_C[7][6] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                                        if (nj >= t1 + c1 + 57)
                                          private_C[7][7] += ((alpha * A[(t0 + c0 + 56) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 56)]);
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          C[(t0 + c0) * nj + (t1 + c1)] = private_C[0][0];
          if (nj >= t1 + c1 + 9) {
            C[(t0 + c0) * nj + (t1 + c1 + 8)] = private_C[0][1];
            if (nj >= t1 + c1 + 17) {
              C[(t0 + c0) * nj + (t1 + c1 + 16)] = private_C[0][2];
              if (nj >= t1 + c1 + 25) {
                C[(t0 + c0) * nj + (t1 + c1 + 24)] = private_C[0][3];
                if (nj >= t1 + c1 + 33) {
                  C[(t0 + c0) * nj + (t1 + c1 + 32)] = private_C[0][4];
                  if (nj >= t1 + c1 + 41) {
                    C[(t0 + c0) * nj + (t1 + c1 + 40)] = private_C[0][5];
                    if (nj >= t1 + c1 + 49) {
                      C[(t0 + c0) * nj + (t1 + c1 + 48)] = private_C[0][6];
                      if (nj >= t1 + c1 + 57)
                        C[(t0 + c0) * nj + (t1 + c1 + 56)] = private_C[0][7];
                    }
                  }
                }
              }
            }
          }
          if (ni >= t0 + c0 + 9) {
            C[(t0 + c0 + 8) * nj + (t1 + c1)] = private_C[1][0];
            if (nj >= t1 + c1 + 9) {
              C[(t0 + c0 + 8) * nj + (t1 + c1 + 8)] = private_C[1][1];
              if (nj >= t1 + c1 + 17) {
                C[(t0 + c0 + 8) * nj + (t1 + c1 + 16)] = private_C[1][2];
                if (nj >= t1 + c1 + 25) {
                  C[(t0 + c0 + 8) * nj + (t1 + c1 + 24)] = private_C[1][3];
                  if (nj >= t1 + c1 + 33) {
                    C[(t0 + c0 + 8) * nj + (t1 + c1 + 32)] = private_C[1][4];
                    if (nj >= t1 + c1 + 41) {
                      C[(t0 + c0 + 8) * nj + (t1 + c1 + 40)] = private_C[1][5];
                      if (nj >= t1 + c1 + 49) {
                        C[(t0 + c0 + 8) * nj + (t1 + c1 + 48)] = private_C[1][6];
                        if (nj >= t1 + c1 + 57)
                          C[(t0 + c0 + 8) * nj + (t1 + c1 + 56)] = private_C[1][7];
                      }
                    }
                  }
                }
              }
            }
            if (ni >= t0 + c0 + 17) {
              C[(t0 + c0 + 16) * nj + (t1 + c1)] = private_C[2][0];
              if (nj >= t1 + c1 + 9) {
                C[(t0 + c0 + 16) * nj + (t1 + c1 + 8)] = private_C[2][1];
                if (nj >= t1 + c1 + 17) {
                  C[(t0 + c0 + 16) * nj + (t1 + c1 + 16)] = private_C[2][2];
                  if (nj >= t1 + c1 + 25) {
                    C[(t0 + c0 + 16) * nj + (t1 + c1 + 24)] = private_C[2][3];
                    if (nj >= t1 + c1 + 33) {
                      C[(t0 + c0 + 16) * nj + (t1 + c1 + 32)] = private_C[2][4];
                      if (nj >= t1 + c1 + 41) {
                        C[(t0 + c0 + 16) * nj + (t1 + c1 + 40)] = private_C[2][5];
                        if (nj >= t1 + c1 + 49) {
                          C[(t0 + c0 + 16) * nj + (t1 + c1 + 48)] = private_C[2][6];
                          if (nj >= t1 + c1 + 57)
                            C[(t0 + c0 + 16) * nj + (t1 + c1 + 56)] = private_C[2][7];
                        }
                      }
                    }
                  }
                }
              }
              if (ni >= t0 + c0 + 25) {
                C[(t0 + c0 + 24) * nj + (t1 + c1)] = private_C[3][0];
                if (nj >= t1 + c1 + 9) {
                  C[(t0 + c0 + 24) * nj + (t1 + c1 + 8)] = private_C[3][1];
                  if (nj >= t1 + c1 + 17) {
                    C[(t0 + c0 + 24) * nj + (t1 + c1 + 16)] = private_C[3][2];
                    if (nj >= t1 + c1 + 25) {
                      C[(t0 + c0 + 24) * nj + (t1 + c1 + 24)] = private_C[3][3];
                      if (nj >= t1 + c1 + 33) {
                        C[(t0 + c0 + 24) * nj + (t1 + c1 + 32)] = private_C[3][4];
                        if (nj >= t1 + c1 + 41) {
                          C[(t0 + c0 + 24) * nj + (t1 + c1 + 40)] = private_C[3][5];
                          if (nj >= t1 + c1 + 49) {
                            C[(t0 + c0 + 24) * nj + (t1 + c1 + 48)] = private_C[3][6];
                            if (nj >= t1 + c1 + 57)
                              C[(t0 + c0 + 24) * nj + (t1 + c1 + 56)] = private_C[3][7];
                          }
                        }
                      }
                    }
                  }
                }
                if (ni >= t0 + c0 + 33) {
                  C[(t0 + c0 + 32) * nj + (t1 + c1)] = private_C[4][0];
                  if (nj >= t1 + c1 + 9) {
                    C[(t0 + c0 + 32) * nj + (t1 + c1 + 8)] = private_C[4][1];
                    if (nj >= t1 + c1 + 17) {
                      C[(t0 + c0 + 32) * nj + (t1 + c1 + 16)] = private_C[4][2];
                      if (nj >= t1 + c1 + 25) {
                        C[(t0 + c0 + 32) * nj + (t1 + c1 + 24)] = private_C[4][3];
                        if (nj >= t1 + c1 + 33) {
                          C[(t0 + c0 + 32) * nj + (t1 + c1 + 32)] = private_C[4][4];
                          if (nj >= t1 + c1 + 41) {
                            C[(t0 + c0 + 32) * nj + (t1 + c1 + 40)] = private_C[4][5];
                            if (nj >= t1 + c1 + 49) {
                              C[(t0 + c0 + 32) * nj + (t1 + c1 + 48)] = private_C[4][6];
                              if (nj >= t1 + c1 + 57)
                                C[(t0 + c0 + 32) * nj + (t1 + c1 + 56)] = private_C[4][7];
                            }
                          }
                        }
                      }
                    }
                  }
                  if (ni >= t0 + c0 + 41) {
                    C[(t0 + c0 + 40) * nj + (t1 + c1)] = private_C[5][0];
                    if (nj >= t1 + c1 + 9) {
                      C[(t0 + c0 + 40) * nj + (t1 + c1 + 8)] = private_C[5][1];
                      if (nj >= t1 + c1 + 17) {
                        C[(t0 + c0 + 40) * nj + (t1 + c1 + 16)] = private_C[5][2];
                        if (nj >= t1 + c1 + 25) {
                          C[(t0 + c0 + 40) * nj + (t1 + c1 + 24)] = private_C[5][3];
                          if (nj >= t1 + c1 + 33) {
                            C[(t0 + c0 + 40) * nj + (t1 + c1 + 32)] = private_C[5][4];
                            if (nj >= t1 + c1 + 41) {
                              C[(t0 + c0 + 40) * nj + (t1 + c1 + 40)] = private_C[5][5];
                              if (nj >= t1 + c1 + 49) {
                                C[(t0 + c0 + 40) * nj + (t1 + c1 + 48)] = private_C[5][6];
                                if (nj >= t1 + c1 + 57)
                                  C[(t0 + c0 + 40) * nj + (t1 + c1 + 56)] = private_C[5][7];
                              }
                            }
                          }
                        }
                      }
                    }
                    if (ni >= t0 + c0 + 49) {
                      C[(t0 + c0 + 48) * nj + (t1 + c1)] = private_C[6][0];
                      if (nj >= t1 + c1 + 9) {
                        C[(t0 + c0 + 48) * nj + (t1 + c1 + 8)] = private_C[6][1];
                        if (nj >= t1 + c1 + 17) {
                          C[(t0 + c0 + 48) * nj + (t1 + c1 + 16)] = private_C[6][2];
                          if (nj >= t1 + c1 + 25) {
                            C[(t0 + c0 + 48) * nj + (t1 + c1 + 24)] = private_C[6][3];
                            if (nj >= t1 + c1 + 33) {
                              C[(t0 + c0 + 48) * nj + (t1 + c1 + 32)] = private_C[6][4];
                              if (nj >= t1 + c1 + 41) {
                                C[(t0 + c0 + 48) * nj + (t1 + c1 + 40)] = private_C[6][5];
                                if (nj >= t1 + c1 + 49) {
                                  C[(t0 + c0 + 48) * nj + (t1 + c1 + 48)] = private_C[6][6];
                                  if (nj >= t1 + c1 + 57)
                                    C[(t0 + c0 + 48) * nj + (t1 + c1 + 56)] = private_C[6][7];
                                }
                              }
                            }
                          }
                        }
                      }
                      if (ni >= t0 + c0 + 57) {
                        C[(t0 + c0 + 56) * nj + (t1 + c1)] = private_C[7][0];
                        if (nj >= t1 + c1 + 9) {
                          C[(t0 + c0 + 56) * nj + (t1 + c1 + 8)] = private_C[7][1];
                          if (nj >= t1 + c1 + 17) {
                            C[(t0 + c0 + 56) * nj + (t1 + c1 + 16)] = private_C[7][2];
                            if (nj >= t1 + c1 + 25) {
                              C[(t0 + c0 + 56) * nj + (t1 + c1 + 24)] = private_C[7][3];
                              if (nj >= t1 + c1 + 33) {
                                C[(t0 + c0 + 56) * nj + (t1 + c1 + 32)] = private_C[7][4];
                                if (nj >= t1 + c1 + 41) {
                                  C[(t0 + c0 + 56) * nj + (t1 + c1 + 40)] = private_C[7][5];
                                  if (nj >= t1 + c1 + 49) {
                                    C[(t0 + c0 + 56) * nj + (t1 + c1 + 48)] = private_C[7][6];
                                    if (nj >= t1 + c1 + 57)
                                      C[(t0 + c0 + 56) * nj + (t1 + c1 + 56)] = private_C[7][7];
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
        }
        __syncthreads();
      }
}
