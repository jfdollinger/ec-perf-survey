#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int nj, int nk, int ni)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    double private_C[1][1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < ni; c0 += 8192)
      for (int c1 = 32 * b1; c1 < nj; c1 += 8192) {
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          private_C[0][0] = C[(t0 + c0) * nj + (t1 + c1)];
          private_C[0][0] *= beta;
          for (int c2 = 0; c2 < nk; c2 += 64)
            for (int c5 = 0; c5 <= ppcg_min(63, nk - c2 - 1); c5 += 1)
              private_C[0][0] += ((alpha * A[(t0 + c0) * nk + (c2 + c5)]) * B[(c2 + c5) * nj + (t1 + c1)]);
          C[(t0 + c0) * nj + (t1 + c1)] = private_C[0][0];
        }
        __syncthreads();
      }
}
