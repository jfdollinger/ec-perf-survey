#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int nj, int nk, int ni)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_B[32][32];
    double private_C[8][1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 256 * b0; c0 < ni; c0 += 65536)
      for (int c1 = 32 * b1; c1 < nj; c1 += 8192) {
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          private_C[0][0] = C[(t0 + c0) * nj + (t1 + c1)];
          if (ni >= t0 + c0 + 33) {
            private_C[1][0] = C[(t0 + c0 + 32) * nj + (t1 + c1)];
            if (ni >= t0 + c0 + 65) {
              private_C[2][0] = C[(t0 + c0 + 64) * nj + (t1 + c1)];
              if (ni >= t0 + c0 + 97) {
                private_C[3][0] = C[(t0 + c0 + 96) * nj + (t1 + c1)];
                if (ni >= t0 + c0 + 129) {
                  private_C[4][0] = C[(t0 + c0 + 128) * nj + (t1 + c1)];
                  if (ni >= t0 + c0 + 161) {
                    private_C[5][0] = C[(t0 + c0 + 160) * nj + (t1 + c1)];
                    if (ni >= t0 + c0 + 193) {
                      private_C[6][0] = C[(t0 + c0 + 192) * nj + (t1 + c1)];
                      if (ni >= t0 + c0 + 225)
                        private_C[7][0] = C[(t0 + c0 + 224) * nj + (t1 + c1)];
                    }
                  }
                }
              }
            }
          }
        }
        for (int c2 = 0; c2 < nk; c2 += 32) {
          if (nj >= t1 + c1 + 1 && nk >= t0 + c2 + 1)
            shared_B[t0][t1] = B[(t0 + c2) * nj + (t1 + c1)];
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1 && c2 == 0) {
            private_C[0][0] *= beta;
            if (ni >= t0 + c0 + 33) {
              private_C[1][0] *= beta;
              if (ni >= t0 + c0 + 65) {
                private_C[2][0] *= beta;
                if (ni >= t0 + c0 + 97) {
                  private_C[3][0] *= beta;
                  if (ni >= t0 + c0 + 129) {
                    private_C[4][0] *= beta;
                    if (ni >= t0 + c0 + 161) {
                      private_C[5][0] *= beta;
                      if (ni >= t0 + c0 + 193) {
                        private_C[6][0] *= beta;
                        if (ni >= t0 + c0 + 225)
                          private_C[7][0] *= beta;
                      }
                    }
                  }
                }
              }
            }
          }
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1)
            for (int c3 = 0; c3 <= ppcg_min(31, nk - c2 - 1); c3 += 1) {
              private_C[0][0] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * shared_B[c3][t1]);
              if (ni >= t0 + c0 + 33) {
                private_C[1][0] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                if (ni >= t0 + c0 + 65) {
                  private_C[2][0] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                  if (ni >= t0 + c0 + 97) {
                    private_C[3][0] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                    if (ni >= t0 + c0 + 129) {
                      private_C[4][0] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                      if (ni >= t0 + c0 + 161) {
                        private_C[5][0] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                        if (ni >= t0 + c0 + 193) {
                          private_C[6][0] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                          if (ni >= t0 + c0 + 225)
                            private_C[7][0] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                        }
                      }
                    }
                  }
                }
              }
            }
          __syncthreads();
        }
        if (nk == 0) {
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
            private_C[0][0] *= beta;
            if (ni >= t0 + c0 + 33) {
              private_C[1][0] *= beta;
              if (ni >= t0 + c0 + 65) {
                private_C[2][0] *= beta;
                if (ni >= t0 + c0 + 97) {
                  private_C[3][0] *= beta;
                  if (ni >= t0 + c0 + 129) {
                    private_C[4][0] *= beta;
                    if (ni >= t0 + c0 + 161) {
                      private_C[5][0] *= beta;
                      if (ni >= t0 + c0 + 193) {
                        private_C[6][0] *= beta;
                        if (ni >= t0 + c0 + 225)
                          private_C[7][0] *= beta;
                      }
                    }
                  }
                }
              }
            }
          }
          __syncthreads();
        }
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          C[(t0 + c0) * nj + (t1 + c1)] = private_C[0][0];
          if (ni >= t0 + c0 + 33) {
            C[(t0 + c0 + 32) * nj + (t1 + c1)] = private_C[1][0];
            if (ni >= t0 + c0 + 65) {
              C[(t0 + c0 + 64) * nj + (t1 + c1)] = private_C[2][0];
              if (ni >= t0 + c0 + 97) {
                C[(t0 + c0 + 96) * nj + (t1 + c1)] = private_C[3][0];
                if (ni >= t0 + c0 + 129) {
                  C[(t0 + c0 + 128) * nj + (t1 + c1)] = private_C[4][0];
                  if (ni >= t0 + c0 + 161) {
                    C[(t0 + c0 + 160) * nj + (t1 + c1)] = private_C[5][0];
                    if (ni >= t0 + c0 + 193) {
                      C[(t0 + c0 + 192) * nj + (t1 + c1)] = private_C[6][0];
                      if (ni >= t0 + c0 + 225)
                        C[(t0 + c0 + 224) * nj + (t1 + c1)] = private_C[7][0];
                    }
                  }
                }
              }
            }
          }
        }
        __syncthreads();
      }
}
