#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int nj, int nk, int ni)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_B[16][64];
    double private_C[8][4];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 128 * b0; c0 < ni; c0 += 32768)
      for (int c1 = 64 * b1; c1 < nj; c1 += 16384) {
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          private_C[0][0] = C[(t0 + c0) * nj + (t1 + c1)];
          if (nj >= t1 + c1 + 17) {
            private_C[0][1] = C[(t0 + c0) * nj + (t1 + c1 + 16)];
            if (nj >= t1 + c1 + 33) {
              private_C[0][2] = C[(t0 + c0) * nj + (t1 + c1 + 32)];
              if (nj >= t1 + c1 + 49)
                private_C[0][3] = C[(t0 + c0) * nj + (t1 + c1 + 48)];
            }
          }
          if (ni >= t0 + c0 + 17) {
            private_C[1][0] = C[(t0 + c0 + 16) * nj + (t1 + c1)];
            if (nj >= t1 + c1 + 17) {
              private_C[1][1] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 16)];
              if (nj >= t1 + c1 + 33) {
                private_C[1][2] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 32)];
                if (nj >= t1 + c1 + 49)
                  private_C[1][3] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 48)];
              }
            }
            if (ni >= t0 + c0 + 33) {
              private_C[2][0] = C[(t0 + c0 + 32) * nj + (t1 + c1)];
              if (nj >= t1 + c1 + 17) {
                private_C[2][1] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 16)];
                if (nj >= t1 + c1 + 33) {
                  private_C[2][2] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 32)];
                  if (nj >= t1 + c1 + 49)
                    private_C[2][3] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 48)];
                }
              }
              if (ni >= t0 + c0 + 49) {
                private_C[3][0] = C[(t0 + c0 + 48) * nj + (t1 + c1)];
                if (nj >= t1 + c1 + 17) {
                  private_C[3][1] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 16)];
                  if (nj >= t1 + c1 + 33) {
                    private_C[3][2] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 32)];
                    if (nj >= t1 + c1 + 49)
                      private_C[3][3] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 48)];
                  }
                }
                if (ni >= t0 + c0 + 65) {
                  private_C[4][0] = C[(t0 + c0 + 64) * nj + (t1 + c1)];
                  if (nj >= t1 + c1 + 17) {
                    private_C[4][1] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 16)];
                    if (nj >= t1 + c1 + 33) {
                      private_C[4][2] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 32)];
                      if (nj >= t1 + c1 + 49)
                        private_C[4][3] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 48)];
                    }
                  }
                  if (ni >= t0 + c0 + 81) {
                    private_C[5][0] = C[(t0 + c0 + 80) * nj + (t1 + c1)];
                    if (nj >= t1 + c1 + 17) {
                      private_C[5][1] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 16)];
                      if (nj >= t1 + c1 + 33) {
                        private_C[5][2] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 32)];
                        if (nj >= t1 + c1 + 49)
                          private_C[5][3] = C[(t0 + c0 + 80) * nj + (t1 + c1 + 48)];
                      }
                    }
                    if (ni >= t0 + c0 + 97) {
                      private_C[6][0] = C[(t0 + c0 + 96) * nj + (t1 + c1)];
                      if (nj >= t1 + c1 + 17) {
                        private_C[6][1] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 16)];
                        if (nj >= t1 + c1 + 33) {
                          private_C[6][2] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 32)];
                          if (nj >= t1 + c1 + 49)
                            private_C[6][3] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 48)];
                        }
                      }
                      if (ni >= t0 + c0 + 113) {
                        private_C[7][0] = C[(t0 + c0 + 112) * nj + (t1 + c1)];
                        if (nj >= t1 + c1 + 17) {
                          private_C[7][1] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 16)];
                          if (nj >= t1 + c1 + 33) {
                            private_C[7][2] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 32)];
                            if (nj >= t1 + c1 + 49)
                              private_C[7][3] = C[(t0 + c0 + 112) * nj + (t1 + c1 + 48)];
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
        }
        for (int c2 = 0; c2 < nk; c2 += 16) {
          if (nk >= t0 + c2 + 1)
            for (int c4 = t1; c4 <= ppcg_min(63, nj - c1 - 1); c4 += 16)
              shared_B[t0][c4] = B[(t0 + c2) * nj + (c1 + c4)];
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1 && c2 == 0) {
            private_C[0][0] *= beta;
            if (nj >= t1 + c1 + 17) {
              private_C[0][1] *= beta;
              if (nj >= t1 + c1 + 33) {
                private_C[0][2] *= beta;
                if (nj >= t1 + c1 + 49)
                  private_C[0][3] *= beta;
              }
            }
            if (ni >= t0 + c0 + 17) {
              private_C[1][0] *= beta;
              if (nj >= t1 + c1 + 17) {
                private_C[1][1] *= beta;
                if (nj >= t1 + c1 + 33) {
                  private_C[1][2] *= beta;
                  if (nj >= t1 + c1 + 49)
                    private_C[1][3] *= beta;
                }
              }
              if (ni >= t0 + c0 + 33) {
                private_C[2][0] *= beta;
                if (nj >= t1 + c1 + 17) {
                  private_C[2][1] *= beta;
                  if (nj >= t1 + c1 + 33) {
                    private_C[2][2] *= beta;
                    if (nj >= t1 + c1 + 49)
                      private_C[2][3] *= beta;
                  }
                }
                if (ni >= t0 + c0 + 49) {
                  private_C[3][0] *= beta;
                  if (nj >= t1 + c1 + 17) {
                    private_C[3][1] *= beta;
                    if (nj >= t1 + c1 + 33) {
                      private_C[3][2] *= beta;
                      if (nj >= t1 + c1 + 49)
                        private_C[3][3] *= beta;
                    }
                  }
                  if (ni >= t0 + c0 + 65) {
                    private_C[4][0] *= beta;
                    if (nj >= t1 + c1 + 17) {
                      private_C[4][1] *= beta;
                      if (nj >= t1 + c1 + 33) {
                        private_C[4][2] *= beta;
                        if (nj >= t1 + c1 + 49)
                          private_C[4][3] *= beta;
                      }
                    }
                    if (ni >= t0 + c0 + 81) {
                      private_C[5][0] *= beta;
                      if (nj >= t1 + c1 + 17) {
                        private_C[5][1] *= beta;
                        if (nj >= t1 + c1 + 33) {
                          private_C[5][2] *= beta;
                          if (nj >= t1 + c1 + 49)
                            private_C[5][3] *= beta;
                        }
                      }
                      if (ni >= t0 + c0 + 97) {
                        private_C[6][0] *= beta;
                        if (nj >= t1 + c1 + 17) {
                          private_C[6][1] *= beta;
                          if (nj >= t1 + c1 + 33) {
                            private_C[6][2] *= beta;
                            if (nj >= t1 + c1 + 49)
                              private_C[6][3] *= beta;
                          }
                        }
                        if (ni >= t0 + c0 + 113) {
                          private_C[7][0] *= beta;
                          if (nj >= t1 + c1 + 17) {
                            private_C[7][1] *= beta;
                            if (nj >= t1 + c1 + 33) {
                              private_C[7][2] *= beta;
                              if (nj >= t1 + c1 + 49)
                                private_C[7][3] *= beta;
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1)
            for (int c3 = 0; c3 <= ppcg_min(15, nk - c2 - 1); c3 += 1) {
              private_C[0][0] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * shared_B[c3][t1]);
              if (nj >= t1 + c1 + 17) {
                private_C[0][1] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * shared_B[c3][t1 + 16]);
                if (nj >= t1 + c1 + 33) {
                  private_C[0][2] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * shared_B[c3][t1 + 32]);
                  if (nj >= t1 + c1 + 49)
                    private_C[0][3] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * shared_B[c3][t1 + 48]);
                }
              }
              if (ni >= t0 + c0 + 17) {
                private_C[1][0] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                if (nj >= t1 + c1 + 17) {
                  private_C[1][1] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * shared_B[c3][t1 + 16]);
                  if (nj >= t1 + c1 + 33) {
                    private_C[1][2] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * shared_B[c3][t1 + 32]);
                    if (nj >= t1 + c1 + 49)
                      private_C[1][3] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * shared_B[c3][t1 + 48]);
                  }
                }
                if (ni >= t0 + c0 + 33) {
                  private_C[2][0] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                  if (nj >= t1 + c1 + 17) {
                    private_C[2][1] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * shared_B[c3][t1 + 16]);
                    if (nj >= t1 + c1 + 33) {
                      private_C[2][2] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * shared_B[c3][t1 + 32]);
                      if (nj >= t1 + c1 + 49)
                        private_C[2][3] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * shared_B[c3][t1 + 48]);
                    }
                  }
                  if (ni >= t0 + c0 + 49) {
                    private_C[3][0] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                    if (nj >= t1 + c1 + 17) {
                      private_C[3][1] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * shared_B[c3][t1 + 16]);
                      if (nj >= t1 + c1 + 33) {
                        private_C[3][2] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * shared_B[c3][t1 + 32]);
                        if (nj >= t1 + c1 + 49)
                          private_C[3][3] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * shared_B[c3][t1 + 48]);
                      }
                    }
                    if (ni >= t0 + c0 + 65) {
                      private_C[4][0] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                      if (nj >= t1 + c1 + 17) {
                        private_C[4][1] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * shared_B[c3][t1 + 16]);
                        if (nj >= t1 + c1 + 33) {
                          private_C[4][2] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * shared_B[c3][t1 + 32]);
                          if (nj >= t1 + c1 + 49)
                            private_C[4][3] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * shared_B[c3][t1 + 48]);
                        }
                      }
                      if (ni >= t0 + c0 + 81) {
                        private_C[5][0] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                        if (nj >= t1 + c1 + 17) {
                          private_C[5][1] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * shared_B[c3][t1 + 16]);
                          if (nj >= t1 + c1 + 33) {
                            private_C[5][2] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * shared_B[c3][t1 + 32]);
                            if (nj >= t1 + c1 + 49)
                              private_C[5][3] += ((alpha * A[(t0 + c0 + 80) * nk + (c2 + c3)]) * shared_B[c3][t1 + 48]);
                          }
                        }
                        if (ni >= t0 + c0 + 97) {
                          private_C[6][0] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                          if (nj >= t1 + c1 + 17) {
                            private_C[6][1] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * shared_B[c3][t1 + 16]);
                            if (nj >= t1 + c1 + 33) {
                              private_C[6][2] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * shared_B[c3][t1 + 32]);
                              if (nj >= t1 + c1 + 49)
                                private_C[6][3] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * shared_B[c3][t1 + 48]);
                            }
                          }
                          if (ni >= t0 + c0 + 113) {
                            private_C[7][0] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * shared_B[c3][t1]);
                            if (nj >= t1 + c1 + 17) {
                              private_C[7][1] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * shared_B[c3][t1 + 16]);
                              if (nj >= t1 + c1 + 33) {
                                private_C[7][2] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * shared_B[c3][t1 + 32]);
                                if (nj >= t1 + c1 + 49)
                                  private_C[7][3] += ((alpha * A[(t0 + c0 + 112) * nk + (c2 + c3)]) * shared_B[c3][t1 + 48]);
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          __syncthreads();
        }
        if (nk == 0) {
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
            private_C[0][0] *= beta;
            if (nj >= t1 + c1 + 17) {
              private_C[0][1] *= beta;
              if (nj >= t1 + c1 + 33) {
                private_C[0][2] *= beta;
                if (nj >= t1 + c1 + 49)
                  private_C[0][3] *= beta;
              }
            }
            if (ni >= t0 + c0 + 17) {
              private_C[1][0] *= beta;
              if (nj >= t1 + c1 + 17) {
                private_C[1][1] *= beta;
                if (nj >= t1 + c1 + 33) {
                  private_C[1][2] *= beta;
                  if (nj >= t1 + c1 + 49)
                    private_C[1][3] *= beta;
                }
              }
              if (ni >= t0 + c0 + 33) {
                private_C[2][0] *= beta;
                if (nj >= t1 + c1 + 17) {
                  private_C[2][1] *= beta;
                  if (nj >= t1 + c1 + 33) {
                    private_C[2][2] *= beta;
                    if (nj >= t1 + c1 + 49)
                      private_C[2][3] *= beta;
                  }
                }
                if (ni >= t0 + c0 + 49) {
                  private_C[3][0] *= beta;
                  if (nj >= t1 + c1 + 17) {
                    private_C[3][1] *= beta;
                    if (nj >= t1 + c1 + 33) {
                      private_C[3][2] *= beta;
                      if (nj >= t1 + c1 + 49)
                        private_C[3][3] *= beta;
                    }
                  }
                  if (ni >= t0 + c0 + 65) {
                    private_C[4][0] *= beta;
                    if (nj >= t1 + c1 + 17) {
                      private_C[4][1] *= beta;
                      if (nj >= t1 + c1 + 33) {
                        private_C[4][2] *= beta;
                        if (nj >= t1 + c1 + 49)
                          private_C[4][3] *= beta;
                      }
                    }
                    if (ni >= t0 + c0 + 81) {
                      private_C[5][0] *= beta;
                      if (nj >= t1 + c1 + 17) {
                        private_C[5][1] *= beta;
                        if (nj >= t1 + c1 + 33) {
                          private_C[5][2] *= beta;
                          if (nj >= t1 + c1 + 49)
                            private_C[5][3] *= beta;
                        }
                      }
                      if (ni >= t0 + c0 + 97) {
                        private_C[6][0] *= beta;
                        if (nj >= t1 + c1 + 17) {
                          private_C[6][1] *= beta;
                          if (nj >= t1 + c1 + 33) {
                            private_C[6][2] *= beta;
                            if (nj >= t1 + c1 + 49)
                              private_C[6][3] *= beta;
                          }
                        }
                        if (ni >= t0 + c0 + 113) {
                          private_C[7][0] *= beta;
                          if (nj >= t1 + c1 + 17) {
                            private_C[7][1] *= beta;
                            if (nj >= t1 + c1 + 33) {
                              private_C[7][2] *= beta;
                              if (nj >= t1 + c1 + 49)
                                private_C[7][3] *= beta;
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          __syncthreads();
        }
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          C[(t0 + c0) * nj + (t1 + c1)] = private_C[0][0];
          if (nj >= t1 + c1 + 17) {
            C[(t0 + c0) * nj + (t1 + c1 + 16)] = private_C[0][1];
            if (nj >= t1 + c1 + 33) {
              C[(t0 + c0) * nj + (t1 + c1 + 32)] = private_C[0][2];
              if (nj >= t1 + c1 + 49)
                C[(t0 + c0) * nj + (t1 + c1 + 48)] = private_C[0][3];
            }
          }
          if (ni >= t0 + c0 + 17) {
            C[(t0 + c0 + 16) * nj + (t1 + c1)] = private_C[1][0];
            if (nj >= t1 + c1 + 17) {
              C[(t0 + c0 + 16) * nj + (t1 + c1 + 16)] = private_C[1][1];
              if (nj >= t1 + c1 + 33) {
                C[(t0 + c0 + 16) * nj + (t1 + c1 + 32)] = private_C[1][2];
                if (nj >= t1 + c1 + 49)
                  C[(t0 + c0 + 16) * nj + (t1 + c1 + 48)] = private_C[1][3];
              }
            }
            if (ni >= t0 + c0 + 33) {
              C[(t0 + c0 + 32) * nj + (t1 + c1)] = private_C[2][0];
              if (nj >= t1 + c1 + 17) {
                C[(t0 + c0 + 32) * nj + (t1 + c1 + 16)] = private_C[2][1];
                if (nj >= t1 + c1 + 33) {
                  C[(t0 + c0 + 32) * nj + (t1 + c1 + 32)] = private_C[2][2];
                  if (nj >= t1 + c1 + 49)
                    C[(t0 + c0 + 32) * nj + (t1 + c1 + 48)] = private_C[2][3];
                }
              }
              if (ni >= t0 + c0 + 49) {
                C[(t0 + c0 + 48) * nj + (t1 + c1)] = private_C[3][0];
                if (nj >= t1 + c1 + 17) {
                  C[(t0 + c0 + 48) * nj + (t1 + c1 + 16)] = private_C[3][1];
                  if (nj >= t1 + c1 + 33) {
                    C[(t0 + c0 + 48) * nj + (t1 + c1 + 32)] = private_C[3][2];
                    if (nj >= t1 + c1 + 49)
                      C[(t0 + c0 + 48) * nj + (t1 + c1 + 48)] = private_C[3][3];
                  }
                }
                if (ni >= t0 + c0 + 65) {
                  C[(t0 + c0 + 64) * nj + (t1 + c1)] = private_C[4][0];
                  if (nj >= t1 + c1 + 17) {
                    C[(t0 + c0 + 64) * nj + (t1 + c1 + 16)] = private_C[4][1];
                    if (nj >= t1 + c1 + 33) {
                      C[(t0 + c0 + 64) * nj + (t1 + c1 + 32)] = private_C[4][2];
                      if (nj >= t1 + c1 + 49)
                        C[(t0 + c0 + 64) * nj + (t1 + c1 + 48)] = private_C[4][3];
                    }
                  }
                  if (ni >= t0 + c0 + 81) {
                    C[(t0 + c0 + 80) * nj + (t1 + c1)] = private_C[5][0];
                    if (nj >= t1 + c1 + 17) {
                      C[(t0 + c0 + 80) * nj + (t1 + c1 + 16)] = private_C[5][1];
                      if (nj >= t1 + c1 + 33) {
                        C[(t0 + c0 + 80) * nj + (t1 + c1 + 32)] = private_C[5][2];
                        if (nj >= t1 + c1 + 49)
                          C[(t0 + c0 + 80) * nj + (t1 + c1 + 48)] = private_C[5][3];
                      }
                    }
                    if (ni >= t0 + c0 + 97) {
                      C[(t0 + c0 + 96) * nj + (t1 + c1)] = private_C[6][0];
                      if (nj >= t1 + c1 + 17) {
                        C[(t0 + c0 + 96) * nj + (t1 + c1 + 16)] = private_C[6][1];
                        if (nj >= t1 + c1 + 33) {
                          C[(t0 + c0 + 96) * nj + (t1 + c1 + 32)] = private_C[6][2];
                          if (nj >= t1 + c1 + 49)
                            C[(t0 + c0 + 96) * nj + (t1 + c1 + 48)] = private_C[6][3];
                        }
                      }
                      if (ni >= t0 + c0 + 113) {
                        C[(t0 + c0 + 112) * nj + (t1 + c1)] = private_C[7][0];
                        if (nj >= t1 + c1 + 17) {
                          C[(t0 + c0 + 112) * nj + (t1 + c1 + 16)] = private_C[7][1];
                          if (nj >= t1 + c1 + 33) {
                            C[(t0 + c0 + 112) * nj + (t1 + c1 + 32)] = private_C[7][2];
                            if (nj >= t1 + c1 + 49)
                              C[(t0 + c0 + 112) * nj + (t1 + c1 + 48)] = private_C[7][3];
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
        }
        __syncthreads();
      }
}
