#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int nj, int nk, int ni)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_A[32][32];
    double private_C[4][4];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < ni; c0 += 8192)
      for (int c1 = 32 * b1; c1 < nj; c1 += 8192) {
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          private_C[0][0] = C[(t0 + c0) * nj + (t1 + c1)];
          if (nj >= t1 + c1 + 9) {
            private_C[0][1] = C[(t0 + c0) * nj + (t1 + c1 + 8)];
            if (nj >= t1 + c1 + 17) {
              private_C[0][2] = C[(t0 + c0) * nj + (t1 + c1 + 16)];
              if (nj >= t1 + c1 + 25)
                private_C[0][3] = C[(t0 + c0) * nj + (t1 + c1 + 24)];
            }
          }
          if (ni >= t0 + c0 + 9) {
            private_C[1][0] = C[(t0 + c0 + 8) * nj + (t1 + c1)];
            if (nj >= t1 + c1 + 9) {
              private_C[1][1] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 8)];
              if (nj >= t1 + c1 + 17) {
                private_C[1][2] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 16)];
                if (nj >= t1 + c1 + 25)
                  private_C[1][3] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 24)];
              }
            }
            if (ni >= t0 + c0 + 17) {
              private_C[2][0] = C[(t0 + c0 + 16) * nj + (t1 + c1)];
              if (nj >= t1 + c1 + 9) {
                private_C[2][1] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 8)];
                if (nj >= t1 + c1 + 17) {
                  private_C[2][2] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 16)];
                  if (nj >= t1 + c1 + 25)
                    private_C[2][3] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 24)];
                }
              }
              if (ni >= t0 + c0 + 25) {
                private_C[3][0] = C[(t0 + c0 + 24) * nj + (t1 + c1)];
                if (nj >= t1 + c1 + 9) {
                  private_C[3][1] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 8)];
                  if (nj >= t1 + c1 + 17) {
                    private_C[3][2] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 16)];
                    if (nj >= t1 + c1 + 25)
                      private_C[3][3] = C[(t0 + c0 + 24) * nj + (t1 + c1 + 24)];
                  }
                }
              }
            }
          }
        }
        for (int c2 = 0; c2 < nk; c2 += 32) {
          for (int c3 = t0; c3 <= ppcg_min(31, ni - c0 - 1); c3 += 8)
            for (int c4 = t1; c4 <= ppcg_min(31, nk - c2 - 1); c4 += 8)
              shared_A[c3][c4] = A[(c0 + c3) * nk + (c2 + c4)];
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1 && c2 == 0) {
            private_C[0][0] *= beta;
            if (nj >= t1 + c1 + 9) {
              private_C[0][1] *= beta;
              if (nj >= t1 + c1 + 17) {
                private_C[0][2] *= beta;
                if (nj >= t1 + c1 + 25)
                  private_C[0][3] *= beta;
              }
            }
            if (ni >= t0 + c0 + 9) {
              private_C[1][0] *= beta;
              if (nj >= t1 + c1 + 9) {
                private_C[1][1] *= beta;
                if (nj >= t1 + c1 + 17) {
                  private_C[1][2] *= beta;
                  if (nj >= t1 + c1 + 25)
                    private_C[1][3] *= beta;
                }
              }
              if (ni >= t0 + c0 + 17) {
                private_C[2][0] *= beta;
                if (nj >= t1 + c1 + 9) {
                  private_C[2][1] *= beta;
                  if (nj >= t1 + c1 + 17) {
                    private_C[2][2] *= beta;
                    if (nj >= t1 + c1 + 25)
                      private_C[2][3] *= beta;
                  }
                }
                if (ni >= t0 + c0 + 25) {
                  private_C[3][0] *= beta;
                  if (nj >= t1 + c1 + 9) {
                    private_C[3][1] *= beta;
                    if (nj >= t1 + c1 + 17) {
                      private_C[3][2] *= beta;
                      if (nj >= t1 + c1 + 25)
                        private_C[3][3] *= beta;
                    }
                  }
                }
              }
            }
          }
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1)
            for (int c3 = 0; c3 <= ppcg_min(31, nk - c2 - 1); c3 += 1) {
              private_C[0][0] += ((alpha * shared_A[t0][c3]) * B[(c2 + c3) * nj + (t1 + c1)]);
              if (nj >= t1 + c1 + 9) {
                private_C[0][1] += ((alpha * shared_A[t0][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                if (nj >= t1 + c1 + 17) {
                  private_C[0][2] += ((alpha * shared_A[t0][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                  if (nj >= t1 + c1 + 25)
                    private_C[0][3] += ((alpha * shared_A[t0][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                }
              }
              if (ni >= t0 + c0 + 9) {
                private_C[1][0] += ((alpha * shared_A[t0 + 8][c3]) * B[(c2 + c3) * nj + (t1 + c1)]);
                if (nj >= t1 + c1 + 9) {
                  private_C[1][1] += ((alpha * shared_A[t0 + 8][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                  if (nj >= t1 + c1 + 17) {
                    private_C[1][2] += ((alpha * shared_A[t0 + 8][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                    if (nj >= t1 + c1 + 25)
                      private_C[1][3] += ((alpha * shared_A[t0 + 8][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                  }
                }
                if (ni >= t0 + c0 + 17) {
                  private_C[2][0] += ((alpha * shared_A[t0 + 16][c3]) * B[(c2 + c3) * nj + (t1 + c1)]);
                  if (nj >= t1 + c1 + 9) {
                    private_C[2][1] += ((alpha * shared_A[t0 + 16][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                    if (nj >= t1 + c1 + 17) {
                      private_C[2][2] += ((alpha * shared_A[t0 + 16][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                      if (nj >= t1 + c1 + 25)
                        private_C[2][3] += ((alpha * shared_A[t0 + 16][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                    }
                  }
                  if (ni >= t0 + c0 + 25) {
                    private_C[3][0] += ((alpha * shared_A[t0 + 24][c3]) * B[(c2 + c3) * nj + (t1 + c1)]);
                    if (nj >= t1 + c1 + 9) {
                      private_C[3][1] += ((alpha * shared_A[t0 + 24][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 8)]);
                      if (nj >= t1 + c1 + 17) {
                        private_C[3][2] += ((alpha * shared_A[t0 + 24][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                        if (nj >= t1 + c1 + 25)
                          private_C[3][3] += ((alpha * shared_A[t0 + 24][c3]) * B[(c2 + c3) * nj + (t1 + c1 + 24)]);
                      }
                    }
                  }
                }
              }
            }
          __syncthreads();
        }
        if (nk == 0) {
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
            private_C[0][0] *= beta;
            if (nj >= t1 + c1 + 9) {
              private_C[0][1] *= beta;
              if (nj >= t1 + c1 + 17) {
                private_C[0][2] *= beta;
                if (nj >= t1 + c1 + 25)
                  private_C[0][3] *= beta;
              }
            }
            if (ni >= t0 + c0 + 9) {
              private_C[1][0] *= beta;
              if (nj >= t1 + c1 + 9) {
                private_C[1][1] *= beta;
                if (nj >= t1 + c1 + 17) {
                  private_C[1][2] *= beta;
                  if (nj >= t1 + c1 + 25)
                    private_C[1][3] *= beta;
                }
              }
              if (ni >= t0 + c0 + 17) {
                private_C[2][0] *= beta;
                if (nj >= t1 + c1 + 9) {
                  private_C[2][1] *= beta;
                  if (nj >= t1 + c1 + 17) {
                    private_C[2][2] *= beta;
                    if (nj >= t1 + c1 + 25)
                      private_C[2][3] *= beta;
                  }
                }
                if (ni >= t0 + c0 + 25) {
                  private_C[3][0] *= beta;
                  if (nj >= t1 + c1 + 9) {
                    private_C[3][1] *= beta;
                    if (nj >= t1 + c1 + 17) {
                      private_C[3][2] *= beta;
                      if (nj >= t1 + c1 + 25)
                        private_C[3][3] *= beta;
                    }
                  }
                }
              }
            }
          }
          __syncthreads();
        }
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          C[(t0 + c0) * nj + (t1 + c1)] = private_C[0][0];
          if (nj >= t1 + c1 + 9) {
            C[(t0 + c0) * nj + (t1 + c1 + 8)] = private_C[0][1];
            if (nj >= t1 + c1 + 17) {
              C[(t0 + c0) * nj + (t1 + c1 + 16)] = private_C[0][2];
              if (nj >= t1 + c1 + 25)
                C[(t0 + c0) * nj + (t1 + c1 + 24)] = private_C[0][3];
            }
          }
          if (ni >= t0 + c0 + 9) {
            C[(t0 + c0 + 8) * nj + (t1 + c1)] = private_C[1][0];
            if (nj >= t1 + c1 + 9) {
              C[(t0 + c0 + 8) * nj + (t1 + c1 + 8)] = private_C[1][1];
              if (nj >= t1 + c1 + 17) {
                C[(t0 + c0 + 8) * nj + (t1 + c1 + 16)] = private_C[1][2];
                if (nj >= t1 + c1 + 25)
                  C[(t0 + c0 + 8) * nj + (t1 + c1 + 24)] = private_C[1][3];
              }
            }
            if (ni >= t0 + c0 + 17) {
              C[(t0 + c0 + 16) * nj + (t1 + c1)] = private_C[2][0];
              if (nj >= t1 + c1 + 9) {
                C[(t0 + c0 + 16) * nj + (t1 + c1 + 8)] = private_C[2][1];
                if (nj >= t1 + c1 + 17) {
                  C[(t0 + c0 + 16) * nj + (t1 + c1 + 16)] = private_C[2][2];
                  if (nj >= t1 + c1 + 25)
                    C[(t0 + c0 + 16) * nj + (t1 + c1 + 24)] = private_C[2][3];
                }
              }
              if (ni >= t0 + c0 + 25) {
                C[(t0 + c0 + 24) * nj + (t1 + c1)] = private_C[3][0];
                if (nj >= t1 + c1 + 9) {
                  C[(t0 + c0 + 24) * nj + (t1 + c1 + 8)] = private_C[3][1];
                  if (nj >= t1 + c1 + 17) {
                    C[(t0 + c0 + 24) * nj + (t1 + c1 + 16)] = private_C[3][2];
                    if (nj >= t1 + c1 + 25)
                      C[(t0 + c0 + 24) * nj + (t1 + c1 + 24)] = private_C[3][3];
                  }
                }
              }
            }
          }
        }
        __syncthreads();
      }
}
