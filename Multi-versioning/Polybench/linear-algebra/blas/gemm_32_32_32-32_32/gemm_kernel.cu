#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int nj, int nk, int ni)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_A[32][32];
    double private_C[1][1];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < ni; c0 += 8192)
      for (int c1 = 32 * b1; c1 < nj; c1 += 8192) {
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1)
          private_C[0][0] = C[(t0 + c0) * nj + (t1 + c1)];
        for (int c2 = 0; c2 < nk; c2 += 32) {
          if (ni >= t0 + c0 + 1 && nk >= t1 + c2 + 1)
            shared_A[t0][t1] = A[(t0 + c0) * nk + (t1 + c2)];
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1 && c2 == 0)
            private_C[0][0] *= beta;
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1)
            for (int c5 = 0; c5 <= ppcg_min(31, nk - c2 - 1); c5 += 1)
              private_C[0][0] += ((alpha * shared_A[t0][c5]) * B[(c2 + c5) * nj + (t1 + c1)]);
          __syncthreads();
        }
        if (nk == 0) {
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1)
            private_C[0][0] *= beta;
          __syncthreads();
        }
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1)
          C[(t0 + c0) * nj + (t1 + c1)] = private_C[0][0];
        __syncthreads();
      }
}
