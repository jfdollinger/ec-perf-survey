#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int nj, int nk, int ni)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    double private_C[4][4];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 64 * b0; c0 < ni; c0 += 16384)
      for (int c1 = 64 * b1; c1 < nj; c1 += 16384) {
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          private_C[0][0] = C[(t0 + c0) * nj + (t1 + c1)];
          if (nj >= t1 + c1 + 17) {
            private_C[0][1] = C[(t0 + c0) * nj + (t1 + c1 + 16)];
            if (nj >= t1 + c1 + 33) {
              private_C[0][2] = C[(t0 + c0) * nj + (t1 + c1 + 32)];
              if (nj >= t1 + c1 + 49)
                private_C[0][3] = C[(t0 + c0) * nj + (t1 + c1 + 48)];
            }
          }
          if (ni >= t0 + c0 + 17) {
            private_C[1][0] = C[(t0 + c0 + 16) * nj + (t1 + c1)];
            if (nj >= t1 + c1 + 17) {
              private_C[1][1] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 16)];
              if (nj >= t1 + c1 + 33) {
                private_C[1][2] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 32)];
                if (nj >= t1 + c1 + 49)
                  private_C[1][3] = C[(t0 + c0 + 16) * nj + (t1 + c1 + 48)];
              }
            }
            if (ni >= t0 + c0 + 33) {
              private_C[2][0] = C[(t0 + c0 + 32) * nj + (t1 + c1)];
              if (nj >= t1 + c1 + 17) {
                private_C[2][1] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 16)];
                if (nj >= t1 + c1 + 33) {
                  private_C[2][2] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 32)];
                  if (nj >= t1 + c1 + 49)
                    private_C[2][3] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 48)];
                }
              }
              if (ni >= t0 + c0 + 49) {
                private_C[3][0] = C[(t0 + c0 + 48) * nj + (t1 + c1)];
                if (nj >= t1 + c1 + 17) {
                  private_C[3][1] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 16)];
                  if (nj >= t1 + c1 + 33) {
                    private_C[3][2] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 32)];
                    if (nj >= t1 + c1 + 49)
                      private_C[3][3] = C[(t0 + c0 + 48) * nj + (t1 + c1 + 48)];
                  }
                }
              }
            }
          }
          private_C[0][0] *= beta;
          if (nj >= t1 + c1 + 17) {
            private_C[0][1] *= beta;
            if (nj >= t1 + c1 + 33) {
              private_C[0][2] *= beta;
              if (nj >= t1 + c1 + 49)
                private_C[0][3] *= beta;
            }
          }
          if (ni >= t0 + c0 + 17) {
            private_C[1][0] *= beta;
            if (nj >= t1 + c1 + 17) {
              private_C[1][1] *= beta;
              if (nj >= t1 + c1 + 33) {
                private_C[1][2] *= beta;
                if (nj >= t1 + c1 + 49)
                  private_C[1][3] *= beta;
              }
            }
            if (ni >= t0 + c0 + 33) {
              private_C[2][0] *= beta;
              if (nj >= t1 + c1 + 17) {
                private_C[2][1] *= beta;
                if (nj >= t1 + c1 + 33) {
                  private_C[2][2] *= beta;
                  if (nj >= t1 + c1 + 49)
                    private_C[2][3] *= beta;
                }
              }
              if (ni >= t0 + c0 + 49) {
                private_C[3][0] *= beta;
                if (nj >= t1 + c1 + 17) {
                  private_C[3][1] *= beta;
                  if (nj >= t1 + c1 + 33) {
                    private_C[3][2] *= beta;
                    if (nj >= t1 + c1 + 49)
                      private_C[3][3] *= beta;
                  }
                }
              }
            }
          }
          for (int c2 = 0; c2 < nk; c2 += 64)
            for (int c3 = 0; c3 <= ppcg_min(63, nk - c2 - 1); c3 += 1) {
              private_C[0][0] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
              if (nj >= t1 + c1 + 17) {
                private_C[0][1] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                if (nj >= t1 + c1 + 33) {
                  private_C[0][2] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                  if (nj >= t1 + c1 + 49)
                    private_C[0][3] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                }
              }
              if (ni >= t0 + c0 + 17) {
                private_C[1][0] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                if (nj >= t1 + c1 + 17) {
                  private_C[1][1] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                  if (nj >= t1 + c1 + 33) {
                    private_C[1][2] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                    if (nj >= t1 + c1 + 49)
                      private_C[1][3] += ((alpha * A[(t0 + c0 + 16) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                  }
                }
                if (ni >= t0 + c0 + 33) {
                  private_C[2][0] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                  if (nj >= t1 + c1 + 17) {
                    private_C[2][1] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                    if (nj >= t1 + c1 + 33) {
                      private_C[2][2] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                      if (nj >= t1 + c1 + 49)
                        private_C[2][3] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                    }
                  }
                  if (ni >= t0 + c0 + 49) {
                    private_C[3][0] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                    if (nj >= t1 + c1 + 17) {
                      private_C[3][1] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 16)]);
                      if (nj >= t1 + c1 + 33) {
                        private_C[3][2] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                        if (nj >= t1 + c1 + 49)
                          private_C[3][3] += ((alpha * A[(t0 + c0 + 48) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 48)]);
                      }
                    }
                  }
                }
              }
            }
          C[(t0 + c0) * nj + (t1 + c1)] = private_C[0][0];
          if (nj >= t1 + c1 + 17) {
            C[(t0 + c0) * nj + (t1 + c1 + 16)] = private_C[0][1];
            if (nj >= t1 + c1 + 33) {
              C[(t0 + c0) * nj + (t1 + c1 + 32)] = private_C[0][2];
              if (nj >= t1 + c1 + 49)
                C[(t0 + c0) * nj + (t1 + c1 + 48)] = private_C[0][3];
            }
          }
          if (ni >= t0 + c0 + 17) {
            C[(t0 + c0 + 16) * nj + (t1 + c1)] = private_C[1][0];
            if (nj >= t1 + c1 + 17) {
              C[(t0 + c0 + 16) * nj + (t1 + c1 + 16)] = private_C[1][1];
              if (nj >= t1 + c1 + 33) {
                C[(t0 + c0 + 16) * nj + (t1 + c1 + 32)] = private_C[1][2];
                if (nj >= t1 + c1 + 49)
                  C[(t0 + c0 + 16) * nj + (t1 + c1 + 48)] = private_C[1][3];
              }
            }
            if (ni >= t0 + c0 + 33) {
              C[(t0 + c0 + 32) * nj + (t1 + c1)] = private_C[2][0];
              if (nj >= t1 + c1 + 17) {
                C[(t0 + c0 + 32) * nj + (t1 + c1 + 16)] = private_C[2][1];
                if (nj >= t1 + c1 + 33) {
                  C[(t0 + c0 + 32) * nj + (t1 + c1 + 32)] = private_C[2][2];
                  if (nj >= t1 + c1 + 49)
                    C[(t0 + c0 + 32) * nj + (t1 + c1 + 48)] = private_C[2][3];
                }
              }
              if (ni >= t0 + c0 + 49) {
                C[(t0 + c0 + 48) * nj + (t1 + c1)] = private_C[3][0];
                if (nj >= t1 + c1 + 17) {
                  C[(t0 + c0 + 48) * nj + (t1 + c1 + 16)] = private_C[3][1];
                  if (nj >= t1 + c1 + 33) {
                    C[(t0 + c0 + 48) * nj + (t1 + c1 + 32)] = private_C[3][2];
                    if (nj >= t1 + c1 + 49)
                      C[(t0 + c0 + 48) * nj + (t1 + c1 + 48)] = private_C[3][3];
                  }
                }
              }
            }
          }
        }
        __syncthreads();
      }
}
