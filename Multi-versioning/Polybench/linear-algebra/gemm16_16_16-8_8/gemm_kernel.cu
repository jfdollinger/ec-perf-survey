#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int nj, int nk, int ni)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    __shared__ double shared_A[16][16];
    __shared__ double shared_B[16][16];
    double private_C[2][2];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 16 * b0; c0 < ni; c0 += 4096)
      for (int c1 = 16 * b1; c1 < nj; c1 += 4096) {
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          private_C[0][0] = C[(t0 + c0) * nj + (t1 + c1)];
          if (nj >= t1 + c1 + 9)
            private_C[0][1] = C[(t0 + c0) * nj + (t1 + c1 + 8)];
          if (ni >= t0 + c0 + 9) {
            private_C[1][0] = C[(t0 + c0 + 8) * nj + (t1 + c1)];
            if (nj >= t1 + c1 + 9)
              private_C[1][1] = C[(t0 + c0 + 8) * nj + (t1 + c1 + 8)];
          }
        }
        for (int c2 = 0; c2 < nk; c2 += 16) {
          for (int c3 = t0; c3 <= ppcg_min(15, ni - c0 - 1); c3 += 8)
            for (int c4 = t1; c4 <= ppcg_min(15, nk - c2 - 1); c4 += 8)
              shared_A[c3][c4] = A[(c0 + c3) * nk + (c2 + c4)];
          for (int c3 = t0; c3 <= ppcg_min(15, nk - c2 - 1); c3 += 8)
            for (int c4 = t1; c4 <= ppcg_min(15, nj - c1 - 1); c4 += 8)
              shared_B[c3][c4] = B[(c2 + c3) * nj + (c1 + c4)];
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1 && c2 == 0) {
            private_C[0][0] *= beta;
            if (nj >= t1 + c1 + 9)
              private_C[0][1] *= beta;
            if (ni >= t0 + c0 + 9) {
              private_C[1][0] *= beta;
              if (nj >= t1 + c1 + 9)
                private_C[1][1] *= beta;
            }
          }
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1)
            for (int c3 = 0; c3 <= ppcg_min(15, nk - c2 - 1); c3 += 1) {
              private_C[0][0] += ((alpha * shared_A[t0][c3]) * shared_B[c3][t1]);
              if (nj >= t1 + c1 + 9)
                private_C[0][1] += ((alpha * shared_A[t0][c3]) * shared_B[c3][t1 + 8]);
              if (ni >= t0 + c0 + 9) {
                private_C[1][0] += ((alpha * shared_A[t0 + 8][c3]) * shared_B[c3][t1]);
                if (nj >= t1 + c1 + 9)
                  private_C[1][1] += ((alpha * shared_A[t0 + 8][c3]) * shared_B[c3][t1 + 8]);
              }
            }
          __syncthreads();
        }
        if (nk == 0) {
          __syncthreads();
          if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
            private_C[0][0] *= beta;
            if (nj >= t1 + c1 + 9)
              private_C[0][1] *= beta;
            if (ni >= t0 + c0 + 9) {
              private_C[1][0] *= beta;
              if (nj >= t1 + c1 + 9)
                private_C[1][1] *= beta;
            }
          }
          __syncthreads();
        }
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          C[(t0 + c0) * nj + (t1 + c1)] = private_C[0][0];
          if (nj >= t1 + c1 + 9)
            C[(t0 + c0) * nj + (t1 + c1 + 8)] = private_C[0][1];
          if (ni >= t0 + c0 + 9) {
            C[(t0 + c0 + 8) * nj + (t1 + c1)] = private_C[1][0];
            if (nj >= t1 + c1 + 9)
              C[(t0 + c0 + 8) * nj + (t1 + c1 + 8)] = private_C[1][1];
          }
        }
        __syncthreads();
      }
}
