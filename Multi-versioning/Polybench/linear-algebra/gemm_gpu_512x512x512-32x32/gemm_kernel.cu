#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int nj, int nk, int ni)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    double private_C[16][16];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 512 * b0; c0 < ni; c0 += 131072)
      for (int c1 = 512 * b1; c1 < nj; c1 += 131072) {
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          private_C[0][0] = C[(t0 + c0) * nj + (t1 + c1)];
          if (nj >= t1 + c1 + 33) {
            private_C[0][1] = C[(t0 + c0) * nj + (t1 + c1 + 32)];
            if (nj >= t1 + c1 + 65) {
              private_C[0][2] = C[(t0 + c0) * nj + (t1 + c1 + 64)];
              if (nj >= t1 + c1 + 97) {
                private_C[0][3] = C[(t0 + c0) * nj + (t1 + c1 + 96)];
                if (nj >= t1 + c1 + 129) {
                  private_C[0][4] = C[(t0 + c0) * nj + (t1 + c1 + 128)];
                  if (nj >= t1 + c1 + 161) {
                    private_C[0][5] = C[(t0 + c0) * nj + (t1 + c1 + 160)];
                    if (nj >= t1 + c1 + 193) {
                      private_C[0][6] = C[(t0 + c0) * nj + (t1 + c1 + 192)];
                      if (nj >= t1 + c1 + 225) {
                        private_C[0][7] = C[(t0 + c0) * nj + (t1 + c1 + 224)];
                        if (nj >= t1 + c1 + 257) {
                          private_C[0][8] = C[(t0 + c0) * nj + (t1 + c1 + 256)];
                          if (nj >= t1 + c1 + 289) {
                            private_C[0][9] = C[(t0 + c0) * nj + (t1 + c1 + 288)];
                            if (nj >= t1 + c1 + 321) {
                              private_C[0][10] = C[(t0 + c0) * nj + (t1 + c1 + 320)];
                              if (nj >= t1 + c1 + 353) {
                                private_C[0][11] = C[(t0 + c0) * nj + (t1 + c1 + 352)];
                                if (nj >= t1 + c1 + 385) {
                                  private_C[0][12] = C[(t0 + c0) * nj + (t1 + c1 + 384)];
                                  if (nj >= t1 + c1 + 417) {
                                    private_C[0][13] = C[(t0 + c0) * nj + (t1 + c1 + 416)];
                                    if (nj >= t1 + c1 + 449) {
                                      private_C[0][14] = C[(t0 + c0) * nj + (t1 + c1 + 448)];
                                      if (nj >= t1 + c1 + 481)
                                        private_C[0][15] = C[(t0 + c0) * nj + (t1 + c1 + 480)];
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          if (ni >= t0 + c0 + 33) {
            private_C[1][0] = C[(t0 + c0 + 32) * nj + (t1 + c1)];
            if (nj >= t1 + c1 + 33) {
              private_C[1][1] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 32)];
              if (nj >= t1 + c1 + 65) {
                private_C[1][2] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 64)];
                if (nj >= t1 + c1 + 97) {
                  private_C[1][3] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 96)];
                  if (nj >= t1 + c1 + 129) {
                    private_C[1][4] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 128)];
                    if (nj >= t1 + c1 + 161) {
                      private_C[1][5] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 160)];
                      if (nj >= t1 + c1 + 193) {
                        private_C[1][6] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 192)];
                        if (nj >= t1 + c1 + 225) {
                          private_C[1][7] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 224)];
                          if (nj >= t1 + c1 + 257) {
                            private_C[1][8] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 256)];
                            if (nj >= t1 + c1 + 289) {
                              private_C[1][9] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 288)];
                              if (nj >= t1 + c1 + 321) {
                                private_C[1][10] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 320)];
                                if (nj >= t1 + c1 + 353) {
                                  private_C[1][11] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 352)];
                                  if (nj >= t1 + c1 + 385) {
                                    private_C[1][12] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 384)];
                                    if (nj >= t1 + c1 + 417) {
                                      private_C[1][13] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 416)];
                                      if (nj >= t1 + c1 + 449) {
                                        private_C[1][14] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 448)];
                                        if (nj >= t1 + c1 + 481)
                                          private_C[1][15] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 480)];
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
            if (ni >= t0 + c0 + 65) {
              private_C[2][0] = C[(t0 + c0 + 64) * nj + (t1 + c1)];
              if (nj >= t1 + c1 + 33) {
                private_C[2][1] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 32)];
                if (nj >= t1 + c1 + 65) {
                  private_C[2][2] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 64)];
                  if (nj >= t1 + c1 + 97) {
                    private_C[2][3] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 96)];
                    if (nj >= t1 + c1 + 129) {
                      private_C[2][4] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 128)];
                      if (nj >= t1 + c1 + 161) {
                        private_C[2][5] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 160)];
                        if (nj >= t1 + c1 + 193) {
                          private_C[2][6] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 192)];
                          if (nj >= t1 + c1 + 225) {
                            private_C[2][7] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 224)];
                            if (nj >= t1 + c1 + 257) {
                              private_C[2][8] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 256)];
                              if (nj >= t1 + c1 + 289) {
                                private_C[2][9] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 288)];
                                if (nj >= t1 + c1 + 321) {
                                  private_C[2][10] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 320)];
                                  if (nj >= t1 + c1 + 353) {
                                    private_C[2][11] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 352)];
                                    if (nj >= t1 + c1 + 385) {
                                      private_C[2][12] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 384)];
                                      if (nj >= t1 + c1 + 417) {
                                        private_C[2][13] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 416)];
                                        if (nj >= t1 + c1 + 449) {
                                          private_C[2][14] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 448)];
                                          if (nj >= t1 + c1 + 481)
                                            private_C[2][15] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 480)];
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
              if (ni >= t0 + c0 + 97) {
                private_C[3][0] = C[(t0 + c0 + 96) * nj + (t1 + c1)];
                if (nj >= t1 + c1 + 33) {
                  private_C[3][1] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 32)];
                  if (nj >= t1 + c1 + 65) {
                    private_C[3][2] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 64)];
                    if (nj >= t1 + c1 + 97) {
                      private_C[3][3] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 96)];
                      if (nj >= t1 + c1 + 129) {
                        private_C[3][4] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 128)];
                        if (nj >= t1 + c1 + 161) {
                          private_C[3][5] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 160)];
                          if (nj >= t1 + c1 + 193) {
                            private_C[3][6] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 192)];
                            if (nj >= t1 + c1 + 225) {
                              private_C[3][7] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 224)];
                              if (nj >= t1 + c1 + 257) {
                                private_C[3][8] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 256)];
                                if (nj >= t1 + c1 + 289) {
                                  private_C[3][9] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 288)];
                                  if (nj >= t1 + c1 + 321) {
                                    private_C[3][10] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 320)];
                                    if (nj >= t1 + c1 + 353) {
                                      private_C[3][11] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 352)];
                                      if (nj >= t1 + c1 + 385) {
                                        private_C[3][12] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 384)];
                                        if (nj >= t1 + c1 + 417) {
                                          private_C[3][13] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 416)];
                                          if (nj >= t1 + c1 + 449) {
                                            private_C[3][14] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 448)];
                                            if (nj >= t1 + c1 + 481)
                                              private_C[3][15] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 480)];
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
                if (ni >= t0 + c0 + 129) {
                  private_C[4][0] = C[(t0 + c0 + 128) * nj + (t1 + c1)];
                  if (nj >= t1 + c1 + 33) {
                    private_C[4][1] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 32)];
                    if (nj >= t1 + c1 + 65) {
                      private_C[4][2] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 64)];
                      if (nj >= t1 + c1 + 97) {
                        private_C[4][3] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 96)];
                        if (nj >= t1 + c1 + 129) {
                          private_C[4][4] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 128)];
                          if (nj >= t1 + c1 + 161) {
                            private_C[4][5] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 160)];
                            if (nj >= t1 + c1 + 193) {
                              private_C[4][6] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 192)];
                              if (nj >= t1 + c1 + 225) {
                                private_C[4][7] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 224)];
                                if (nj >= t1 + c1 + 257) {
                                  private_C[4][8] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 256)];
                                  if (nj >= t1 + c1 + 289) {
                                    private_C[4][9] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 288)];
                                    if (nj >= t1 + c1 + 321) {
                                      private_C[4][10] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 320)];
                                      if (nj >= t1 + c1 + 353) {
                                        private_C[4][11] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 352)];
                                        if (nj >= t1 + c1 + 385) {
                                          private_C[4][12] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 384)];
                                          if (nj >= t1 + c1 + 417) {
                                            private_C[4][13] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 416)];
                                            if (nj >= t1 + c1 + 449) {
                                              private_C[4][14] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 448)];
                                              if (nj >= t1 + c1 + 481)
                                                private_C[4][15] = C[(t0 + c0 + 128) * nj + (t1 + c1 + 480)];
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                  if (ni >= t0 + c0 + 161) {
                    private_C[5][0] = C[(t0 + c0 + 160) * nj + (t1 + c1)];
                    if (nj >= t1 + c1 + 33) {
                      private_C[5][1] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 32)];
                      if (nj >= t1 + c1 + 65) {
                        private_C[5][2] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 64)];
                        if (nj >= t1 + c1 + 97) {
                          private_C[5][3] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 96)];
                          if (nj >= t1 + c1 + 129) {
                            private_C[5][4] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 128)];
                            if (nj >= t1 + c1 + 161) {
                              private_C[5][5] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 160)];
                              if (nj >= t1 + c1 + 193) {
                                private_C[5][6] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 192)];
                                if (nj >= t1 + c1 + 225) {
                                  private_C[5][7] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 224)];
                                  if (nj >= t1 + c1 + 257) {
                                    private_C[5][8] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 256)];
                                    if (nj >= t1 + c1 + 289) {
                                      private_C[5][9] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 288)];
                                      if (nj >= t1 + c1 + 321) {
                                        private_C[5][10] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 320)];
                                        if (nj >= t1 + c1 + 353) {
                                          private_C[5][11] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 352)];
                                          if (nj >= t1 + c1 + 385) {
                                            private_C[5][12] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 384)];
                                            if (nj >= t1 + c1 + 417) {
                                              private_C[5][13] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 416)];
                                              if (nj >= t1 + c1 + 449) {
                                                private_C[5][14] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 448)];
                                                if (nj >= t1 + c1 + 481)
                                                  private_C[5][15] = C[(t0 + c0 + 160) * nj + (t1 + c1 + 480)];
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                    if (ni >= t0 + c0 + 193) {
                      private_C[6][0] = C[(t0 + c0 + 192) * nj + (t1 + c1)];
                      if (nj >= t1 + c1 + 33) {
                        private_C[6][1] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 32)];
                        if (nj >= t1 + c1 + 65) {
                          private_C[6][2] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 64)];
                          if (nj >= t1 + c1 + 97) {
                            private_C[6][3] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 96)];
                            if (nj >= t1 + c1 + 129) {
                              private_C[6][4] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 128)];
                              if (nj >= t1 + c1 + 161) {
                                private_C[6][5] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 160)];
                                if (nj >= t1 + c1 + 193) {
                                  private_C[6][6] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 192)];
                                  if (nj >= t1 + c1 + 225) {
                                    private_C[6][7] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 224)];
                                    if (nj >= t1 + c1 + 257) {
                                      private_C[6][8] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 256)];
                                      if (nj >= t1 + c1 + 289) {
                                        private_C[6][9] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 288)];
                                        if (nj >= t1 + c1 + 321) {
                                          private_C[6][10] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 320)];
                                          if (nj >= t1 + c1 + 353) {
                                            private_C[6][11] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 352)];
                                            if (nj >= t1 + c1 + 385) {
                                              private_C[6][12] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 384)];
                                              if (nj >= t1 + c1 + 417) {
                                                private_C[6][13] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 416)];
                                                if (nj >= t1 + c1 + 449) {
                                                  private_C[6][14] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 448)];
                                                  if (nj >= t1 + c1 + 481)
                                                    private_C[6][15] = C[(t0 + c0 + 192) * nj + (t1 + c1 + 480)];
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                      if (ni >= t0 + c0 + 225) {
                        private_C[7][0] = C[(t0 + c0 + 224) * nj + (t1 + c1)];
                        if (nj >= t1 + c1 + 33) {
                          private_C[7][1] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 32)];
                          if (nj >= t1 + c1 + 65) {
                            private_C[7][2] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 64)];
                            if (nj >= t1 + c1 + 97) {
                              private_C[7][3] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 96)];
                              if (nj >= t1 + c1 + 129) {
                                private_C[7][4] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 128)];
                                if (nj >= t1 + c1 + 161) {
                                  private_C[7][5] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 160)];
                                  if (nj >= t1 + c1 + 193) {
                                    private_C[7][6] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 192)];
                                    if (nj >= t1 + c1 + 225) {
                                      private_C[7][7] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 224)];
                                      if (nj >= t1 + c1 + 257) {
                                        private_C[7][8] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 256)];
                                        if (nj >= t1 + c1 + 289) {
                                          private_C[7][9] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 288)];
                                          if (nj >= t1 + c1 + 321) {
                                            private_C[7][10] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 320)];
                                            if (nj >= t1 + c1 + 353) {
                                              private_C[7][11] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 352)];
                                              if (nj >= t1 + c1 + 385) {
                                                private_C[7][12] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 384)];
                                                if (nj >= t1 + c1 + 417) {
                                                  private_C[7][13] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 416)];
                                                  if (nj >= t1 + c1 + 449) {
                                                    private_C[7][14] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 448)];
                                                    if (nj >= t1 + c1 + 481)
                                                      private_C[7][15] = C[(t0 + c0 + 224) * nj + (t1 + c1 + 480)];
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                        if (ni >= t0 + c0 + 257) {
                          private_C[8][0] = C[(t0 + c0 + 256) * nj + (t1 + c1)];
                          if (nj >= t1 + c1 + 33) {
                            private_C[8][1] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 32)];
                            if (nj >= t1 + c1 + 65) {
                              private_C[8][2] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 64)];
                              if (nj >= t1 + c1 + 97) {
                                private_C[8][3] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 96)];
                                if (nj >= t1 + c1 + 129) {
                                  private_C[8][4] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 128)];
                                  if (nj >= t1 + c1 + 161) {
                                    private_C[8][5] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 160)];
                                    if (nj >= t1 + c1 + 193) {
                                      private_C[8][6] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 192)];
                                      if (nj >= t1 + c1 + 225) {
                                        private_C[8][7] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 224)];
                                        if (nj >= t1 + c1 + 257) {
                                          private_C[8][8] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 256)];
                                          if (nj >= t1 + c1 + 289) {
                                            private_C[8][9] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 288)];
                                            if (nj >= t1 + c1 + 321) {
                                              private_C[8][10] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 320)];
                                              if (nj >= t1 + c1 + 353) {
                                                private_C[8][11] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 352)];
                                                if (nj >= t1 + c1 + 385) {
                                                  private_C[8][12] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 384)];
                                                  if (nj >= t1 + c1 + 417) {
                                                    private_C[8][13] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 416)];
                                                    if (nj >= t1 + c1 + 449) {
                                                      private_C[8][14] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 448)];
                                                      if (nj >= t1 + c1 + 481)
                                                        private_C[8][15] = C[(t0 + c0 + 256) * nj + (t1 + c1 + 480)];
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                          if (ni >= t0 + c0 + 289) {
                            private_C[9][0] = C[(t0 + c0 + 288) * nj + (t1 + c1)];
                            if (nj >= t1 + c1 + 33) {
                              private_C[9][1] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 32)];
                              if (nj >= t1 + c1 + 65) {
                                private_C[9][2] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 64)];
                                if (nj >= t1 + c1 + 97) {
                                  private_C[9][3] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 96)];
                                  if (nj >= t1 + c1 + 129) {
                                    private_C[9][4] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 128)];
                                    if (nj >= t1 + c1 + 161) {
                                      private_C[9][5] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 160)];
                                      if (nj >= t1 + c1 + 193) {
                                        private_C[9][6] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 192)];
                                        if (nj >= t1 + c1 + 225) {
                                          private_C[9][7] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 224)];
                                          if (nj >= t1 + c1 + 257) {
                                            private_C[9][8] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 256)];
                                            if (nj >= t1 + c1 + 289) {
                                              private_C[9][9] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 288)];
                                              if (nj >= t1 + c1 + 321) {
                                                private_C[9][10] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 320)];
                                                if (nj >= t1 + c1 + 353) {
                                                  private_C[9][11] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 352)];
                                                  if (nj >= t1 + c1 + 385) {
                                                    private_C[9][12] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 384)];
                                                    if (nj >= t1 + c1 + 417) {
                                                      private_C[9][13] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 416)];
                                                      if (nj >= t1 + c1 + 449) {
                                                        private_C[9][14] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 448)];
                                                        if (nj >= t1 + c1 + 481)
                                                          private_C[9][15] = C[(t0 + c0 + 288) * nj + (t1 + c1 + 480)];
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                            if (ni >= t0 + c0 + 321) {
                              private_C[10][0] = C[(t0 + c0 + 320) * nj + (t1 + c1)];
                              if (nj >= t1 + c1 + 33) {
                                private_C[10][1] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 32)];
                                if (nj >= t1 + c1 + 65) {
                                  private_C[10][2] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 64)];
                                  if (nj >= t1 + c1 + 97) {
                                    private_C[10][3] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 96)];
                                    if (nj >= t1 + c1 + 129) {
                                      private_C[10][4] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 128)];
                                      if (nj >= t1 + c1 + 161) {
                                        private_C[10][5] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 160)];
                                        if (nj >= t1 + c1 + 193) {
                                          private_C[10][6] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 192)];
                                          if (nj >= t1 + c1 + 225) {
                                            private_C[10][7] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 224)];
                                            if (nj >= t1 + c1 + 257) {
                                              private_C[10][8] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 256)];
                                              if (nj >= t1 + c1 + 289) {
                                                private_C[10][9] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 288)];
                                                if (nj >= t1 + c1 + 321) {
                                                  private_C[10][10] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 320)];
                                                  if (nj >= t1 + c1 + 353) {
                                                    private_C[10][11] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 352)];
                                                    if (nj >= t1 + c1 + 385) {
                                                      private_C[10][12] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 384)];
                                                      if (nj >= t1 + c1 + 417) {
                                                        private_C[10][13] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 416)];
                                                        if (nj >= t1 + c1 + 449) {
                                                          private_C[10][14] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 448)];
                                                          if (nj >= t1 + c1 + 481)
                                                            private_C[10][15] = C[(t0 + c0 + 320) * nj + (t1 + c1 + 480)];
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                              if (ni >= t0 + c0 + 353) {
                                private_C[11][0] = C[(t0 + c0 + 352) * nj + (t1 + c1)];
                                if (nj >= t1 + c1 + 33) {
                                  private_C[11][1] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 32)];
                                  if (nj >= t1 + c1 + 65) {
                                    private_C[11][2] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 64)];
                                    if (nj >= t1 + c1 + 97) {
                                      private_C[11][3] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 96)];
                                      if (nj >= t1 + c1 + 129) {
                                        private_C[11][4] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 128)];
                                        if (nj >= t1 + c1 + 161) {
                                          private_C[11][5] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 160)];
                                          if (nj >= t1 + c1 + 193) {
                                            private_C[11][6] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 192)];
                                            if (nj >= t1 + c1 + 225) {
                                              private_C[11][7] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 224)];
                                              if (nj >= t1 + c1 + 257) {
                                                private_C[11][8] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 256)];
                                                if (nj >= t1 + c1 + 289) {
                                                  private_C[11][9] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 288)];
                                                  if (nj >= t1 + c1 + 321) {
                                                    private_C[11][10] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 320)];
                                                    if (nj >= t1 + c1 + 353) {
                                                      private_C[11][11] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 352)];
                                                      if (nj >= t1 + c1 + 385) {
                                                        private_C[11][12] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 384)];
                                                        if (nj >= t1 + c1 + 417) {
                                                          private_C[11][13] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 416)];
                                                          if (nj >= t1 + c1 + 449) {
                                                            private_C[11][14] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 448)];
                                                            if (nj >= t1 + c1 + 481)
                                                              private_C[11][15] = C[(t0 + c0 + 352) * nj + (t1 + c1 + 480)];
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                                if (ni >= t0 + c0 + 385) {
                                  private_C[12][0] = C[(t0 + c0 + 384) * nj + (t1 + c1)];
                                  if (nj >= t1 + c1 + 33) {
                                    private_C[12][1] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 32)];
                                    if (nj >= t1 + c1 + 65) {
                                      private_C[12][2] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 64)];
                                      if (nj >= t1 + c1 + 97) {
                                        private_C[12][3] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 96)];
                                        if (nj >= t1 + c1 + 129) {
                                          private_C[12][4] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 128)];
                                          if (nj >= t1 + c1 + 161) {
                                            private_C[12][5] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 160)];
                                            if (nj >= t1 + c1 + 193) {
                                              private_C[12][6] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 192)];
                                              if (nj >= t1 + c1 + 225) {
                                                private_C[12][7] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 224)];
                                                if (nj >= t1 + c1 + 257) {
                                                  private_C[12][8] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 256)];
                                                  if (nj >= t1 + c1 + 289) {
                                                    private_C[12][9] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 288)];
                                                    if (nj >= t1 + c1 + 321) {
                                                      private_C[12][10] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 320)];
                                                      if (nj >= t1 + c1 + 353) {
                                                        private_C[12][11] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 352)];
                                                        if (nj >= t1 + c1 + 385) {
                                                          private_C[12][12] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 384)];
                                                          if (nj >= t1 + c1 + 417) {
                                                            private_C[12][13] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 416)];
                                                            if (nj >= t1 + c1 + 449) {
                                                              private_C[12][14] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 448)];
                                                              if (nj >= t1 + c1 + 481)
                                                                private_C[12][15] = C[(t0 + c0 + 384) * nj + (t1 + c1 + 480)];
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                  if (ni >= t0 + c0 + 417) {
                                    private_C[13][0] = C[(t0 + c0 + 416) * nj + (t1 + c1)];
                                    if (nj >= t1 + c1 + 33) {
                                      private_C[13][1] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 32)];
                                      if (nj >= t1 + c1 + 65) {
                                        private_C[13][2] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 64)];
                                        if (nj >= t1 + c1 + 97) {
                                          private_C[13][3] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 96)];
                                          if (nj >= t1 + c1 + 129) {
                                            private_C[13][4] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 128)];
                                            if (nj >= t1 + c1 + 161) {
                                              private_C[13][5] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 160)];
                                              if (nj >= t1 + c1 + 193) {
                                                private_C[13][6] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 192)];
                                                if (nj >= t1 + c1 + 225) {
                                                  private_C[13][7] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 224)];
                                                  if (nj >= t1 + c1 + 257) {
                                                    private_C[13][8] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 256)];
                                                    if (nj >= t1 + c1 + 289) {
                                                      private_C[13][9] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 288)];
                                                      if (nj >= t1 + c1 + 321) {
                                                        private_C[13][10] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 320)];
                                                        if (nj >= t1 + c1 + 353) {
                                                          private_C[13][11] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 352)];
                                                          if (nj >= t1 + c1 + 385) {
                                                            private_C[13][12] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 384)];
                                                            if (nj >= t1 + c1 + 417) {
                                                              private_C[13][13] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 416)];
                                                              if (nj >= t1 + c1 + 449) {
                                                                private_C[13][14] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 448)];
                                                                if (nj >= t1 + c1 + 481)
                                                                  private_C[13][15] = C[(t0 + c0 + 416) * nj + (t1 + c1 + 480)];
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                    if (ni >= t0 + c0 + 449) {
                                      private_C[14][0] = C[(t0 + c0 + 448) * nj + (t1 + c1)];
                                      if (nj >= t1 + c1 + 33) {
                                        private_C[14][1] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 32)];
                                        if (nj >= t1 + c1 + 65) {
                                          private_C[14][2] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 64)];
                                          if (nj >= t1 + c1 + 97) {
                                            private_C[14][3] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 96)];
                                            if (nj >= t1 + c1 + 129) {
                                              private_C[14][4] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 128)];
                                              if (nj >= t1 + c1 + 161) {
                                                private_C[14][5] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 160)];
                                                if (nj >= t1 + c1 + 193) {
                                                  private_C[14][6] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 192)];
                                                  if (nj >= t1 + c1 + 225) {
                                                    private_C[14][7] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 224)];
                                                    if (nj >= t1 + c1 + 257) {
                                                      private_C[14][8] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 256)];
                                                      if (nj >= t1 + c1 + 289) {
                                                        private_C[14][9] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 288)];
                                                        if (nj >= t1 + c1 + 321) {
                                                          private_C[14][10] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 320)];
                                                          if (nj >= t1 + c1 + 353) {
                                                            private_C[14][11] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 352)];
                                                            if (nj >= t1 + c1 + 385) {
                                                              private_C[14][12] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 384)];
                                                              if (nj >= t1 + c1 + 417) {
                                                                private_C[14][13] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 416)];
                                                                if (nj >= t1 + c1 + 449) {
                                                                  private_C[14][14] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 448)];
                                                                  if (nj >= t1 + c1 + 481)
                                                                    private_C[14][15] = C[(t0 + c0 + 448) * nj + (t1 + c1 + 480)];
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                      if (ni >= t0 + c0 + 481) {
                                        private_C[15][0] = C[(t0 + c0 + 480) * nj + (t1 + c1)];
                                        if (nj >= t1 + c1 + 33) {
                                          private_C[15][1] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 32)];
                                          if (nj >= t1 + c1 + 65) {
                                            private_C[15][2] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 64)];
                                            if (nj >= t1 + c1 + 97) {
                                              private_C[15][3] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 96)];
                                              if (nj >= t1 + c1 + 129) {
                                                private_C[15][4] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 128)];
                                                if (nj >= t1 + c1 + 161) {
                                                  private_C[15][5] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 160)];
                                                  if (nj >= t1 + c1 + 193) {
                                                    private_C[15][6] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 192)];
                                                    if (nj >= t1 + c1 + 225) {
                                                      private_C[15][7] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 224)];
                                                      if (nj >= t1 + c1 + 257) {
                                                        private_C[15][8] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 256)];
                                                        if (nj >= t1 + c1 + 289) {
                                                          private_C[15][9] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 288)];
                                                          if (nj >= t1 + c1 + 321) {
                                                            private_C[15][10] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 320)];
                                                            if (nj >= t1 + c1 + 353) {
                                                              private_C[15][11] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 352)];
                                                              if (nj >= t1 + c1 + 385) {
                                                                private_C[15][12] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 384)];
                                                                if (nj >= t1 + c1 + 417) {
                                                                  private_C[15][13] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 416)];
                                                                  if (nj >= t1 + c1 + 449) {
                                                                    private_C[15][14] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 448)];
                                                                    if (nj >= t1 + c1 + 481)
                                                                      private_C[15][15] = C[(t0 + c0 + 480) * nj + (t1 + c1 + 480)];
                                                                  }
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          private_C[0][0] *= beta;
          if (nj >= t1 + c1 + 33) {
            private_C[0][1] *= beta;
            if (nj >= t1 + c1 + 65) {
              private_C[0][2] *= beta;
              if (nj >= t1 + c1 + 97) {
                private_C[0][3] *= beta;
                if (nj >= t1 + c1 + 129) {
                  private_C[0][4] *= beta;
                  if (nj >= t1 + c1 + 161) {
                    private_C[0][5] *= beta;
                    if (nj >= t1 + c1 + 193) {
                      private_C[0][6] *= beta;
                      if (nj >= t1 + c1 + 225) {
                        private_C[0][7] *= beta;
                        if (nj >= t1 + c1 + 257) {
                          private_C[0][8] *= beta;
                          if (nj >= t1 + c1 + 289) {
                            private_C[0][9] *= beta;
                            if (nj >= t1 + c1 + 321) {
                              private_C[0][10] *= beta;
                              if (nj >= t1 + c1 + 353) {
                                private_C[0][11] *= beta;
                                if (nj >= t1 + c1 + 385) {
                                  private_C[0][12] *= beta;
                                  if (nj >= t1 + c1 + 417) {
                                    private_C[0][13] *= beta;
                                    if (nj >= t1 + c1 + 449) {
                                      private_C[0][14] *= beta;
                                      if (nj >= t1 + c1 + 481)
                                        private_C[0][15] *= beta;
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          if (ni >= t0 + c0 + 33) {
            private_C[1][0] *= beta;
            if (nj >= t1 + c1 + 33) {
              private_C[1][1] *= beta;
              if (nj >= t1 + c1 + 65) {
                private_C[1][2] *= beta;
                if (nj >= t1 + c1 + 97) {
                  private_C[1][3] *= beta;
                  if (nj >= t1 + c1 + 129) {
                    private_C[1][4] *= beta;
                    if (nj >= t1 + c1 + 161) {
                      private_C[1][5] *= beta;
                      if (nj >= t1 + c1 + 193) {
                        private_C[1][6] *= beta;
                        if (nj >= t1 + c1 + 225) {
                          private_C[1][7] *= beta;
                          if (nj >= t1 + c1 + 257) {
                            private_C[1][8] *= beta;
                            if (nj >= t1 + c1 + 289) {
                              private_C[1][9] *= beta;
                              if (nj >= t1 + c1 + 321) {
                                private_C[1][10] *= beta;
                                if (nj >= t1 + c1 + 353) {
                                  private_C[1][11] *= beta;
                                  if (nj >= t1 + c1 + 385) {
                                    private_C[1][12] *= beta;
                                    if (nj >= t1 + c1 + 417) {
                                      private_C[1][13] *= beta;
                                      if (nj >= t1 + c1 + 449) {
                                        private_C[1][14] *= beta;
                                        if (nj >= t1 + c1 + 481)
                                          private_C[1][15] *= beta;
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
            if (ni >= t0 + c0 + 65) {
              private_C[2][0] *= beta;
              if (nj >= t1 + c1 + 33) {
                private_C[2][1] *= beta;
                if (nj >= t1 + c1 + 65) {
                  private_C[2][2] *= beta;
                  if (nj >= t1 + c1 + 97) {
                    private_C[2][3] *= beta;
                    if (nj >= t1 + c1 + 129) {
                      private_C[2][4] *= beta;
                      if (nj >= t1 + c1 + 161) {
                        private_C[2][5] *= beta;
                        if (nj >= t1 + c1 + 193) {
                          private_C[2][6] *= beta;
                          if (nj >= t1 + c1 + 225) {
                            private_C[2][7] *= beta;
                            if (nj >= t1 + c1 + 257) {
                              private_C[2][8] *= beta;
                              if (nj >= t1 + c1 + 289) {
                                private_C[2][9] *= beta;
                                if (nj >= t1 + c1 + 321) {
                                  private_C[2][10] *= beta;
                                  if (nj >= t1 + c1 + 353) {
                                    private_C[2][11] *= beta;
                                    if (nj >= t1 + c1 + 385) {
                                      private_C[2][12] *= beta;
                                      if (nj >= t1 + c1 + 417) {
                                        private_C[2][13] *= beta;
                                        if (nj >= t1 + c1 + 449) {
                                          private_C[2][14] *= beta;
                                          if (nj >= t1 + c1 + 481)
                                            private_C[2][15] *= beta;
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
              if (ni >= t0 + c0 + 97) {
                private_C[3][0] *= beta;
                if (nj >= t1 + c1 + 33) {
                  private_C[3][1] *= beta;
                  if (nj >= t1 + c1 + 65) {
                    private_C[3][2] *= beta;
                    if (nj >= t1 + c1 + 97) {
                      private_C[3][3] *= beta;
                      if (nj >= t1 + c1 + 129) {
                        private_C[3][4] *= beta;
                        if (nj >= t1 + c1 + 161) {
                          private_C[3][5] *= beta;
                          if (nj >= t1 + c1 + 193) {
                            private_C[3][6] *= beta;
                            if (nj >= t1 + c1 + 225) {
                              private_C[3][7] *= beta;
                              if (nj >= t1 + c1 + 257) {
                                private_C[3][8] *= beta;
                                if (nj >= t1 + c1 + 289) {
                                  private_C[3][9] *= beta;
                                  if (nj >= t1 + c1 + 321) {
                                    private_C[3][10] *= beta;
                                    if (nj >= t1 + c1 + 353) {
                                      private_C[3][11] *= beta;
                                      if (nj >= t1 + c1 + 385) {
                                        private_C[3][12] *= beta;
                                        if (nj >= t1 + c1 + 417) {
                                          private_C[3][13] *= beta;
                                          if (nj >= t1 + c1 + 449) {
                                            private_C[3][14] *= beta;
                                            if (nj >= t1 + c1 + 481)
                                              private_C[3][15] *= beta;
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
                if (ni >= t0 + c0 + 129) {
                  private_C[4][0] *= beta;
                  if (nj >= t1 + c1 + 33) {
                    private_C[4][1] *= beta;
                    if (nj >= t1 + c1 + 65) {
                      private_C[4][2] *= beta;
                      if (nj >= t1 + c1 + 97) {
                        private_C[4][3] *= beta;
                        if (nj >= t1 + c1 + 129) {
                          private_C[4][4] *= beta;
                          if (nj >= t1 + c1 + 161) {
                            private_C[4][5] *= beta;
                            if (nj >= t1 + c1 + 193) {
                              private_C[4][6] *= beta;
                              if (nj >= t1 + c1 + 225) {
                                private_C[4][7] *= beta;
                                if (nj >= t1 + c1 + 257) {
                                  private_C[4][8] *= beta;
                                  if (nj >= t1 + c1 + 289) {
                                    private_C[4][9] *= beta;
                                    if (nj >= t1 + c1 + 321) {
                                      private_C[4][10] *= beta;
                                      if (nj >= t1 + c1 + 353) {
                                        private_C[4][11] *= beta;
                                        if (nj >= t1 + c1 + 385) {
                                          private_C[4][12] *= beta;
                                          if (nj >= t1 + c1 + 417) {
                                            private_C[4][13] *= beta;
                                            if (nj >= t1 + c1 + 449) {
                                              private_C[4][14] *= beta;
                                              if (nj >= t1 + c1 + 481)
                                                private_C[4][15] *= beta;
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                  if (ni >= t0 + c0 + 161) {
                    private_C[5][0] *= beta;
                    if (nj >= t1 + c1 + 33) {
                      private_C[5][1] *= beta;
                      if (nj >= t1 + c1 + 65) {
                        private_C[5][2] *= beta;
                        if (nj >= t1 + c1 + 97) {
                          private_C[5][3] *= beta;
                          if (nj >= t1 + c1 + 129) {
                            private_C[5][4] *= beta;
                            if (nj >= t1 + c1 + 161) {
                              private_C[5][5] *= beta;
                              if (nj >= t1 + c1 + 193) {
                                private_C[5][6] *= beta;
                                if (nj >= t1 + c1 + 225) {
                                  private_C[5][7] *= beta;
                                  if (nj >= t1 + c1 + 257) {
                                    private_C[5][8] *= beta;
                                    if (nj >= t1 + c1 + 289) {
                                      private_C[5][9] *= beta;
                                      if (nj >= t1 + c1 + 321) {
                                        private_C[5][10] *= beta;
                                        if (nj >= t1 + c1 + 353) {
                                          private_C[5][11] *= beta;
                                          if (nj >= t1 + c1 + 385) {
                                            private_C[5][12] *= beta;
                                            if (nj >= t1 + c1 + 417) {
                                              private_C[5][13] *= beta;
                                              if (nj >= t1 + c1 + 449) {
                                                private_C[5][14] *= beta;
                                                if (nj >= t1 + c1 + 481)
                                                  private_C[5][15] *= beta;
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                    if (ni >= t0 + c0 + 193) {
                      private_C[6][0] *= beta;
                      if (nj >= t1 + c1 + 33) {
                        private_C[6][1] *= beta;
                        if (nj >= t1 + c1 + 65) {
                          private_C[6][2] *= beta;
                          if (nj >= t1 + c1 + 97) {
                            private_C[6][3] *= beta;
                            if (nj >= t1 + c1 + 129) {
                              private_C[6][4] *= beta;
                              if (nj >= t1 + c1 + 161) {
                                private_C[6][5] *= beta;
                                if (nj >= t1 + c1 + 193) {
                                  private_C[6][6] *= beta;
                                  if (nj >= t1 + c1 + 225) {
                                    private_C[6][7] *= beta;
                                    if (nj >= t1 + c1 + 257) {
                                      private_C[6][8] *= beta;
                                      if (nj >= t1 + c1 + 289) {
                                        private_C[6][9] *= beta;
                                        if (nj >= t1 + c1 + 321) {
                                          private_C[6][10] *= beta;
                                          if (nj >= t1 + c1 + 353) {
                                            private_C[6][11] *= beta;
                                            if (nj >= t1 + c1 + 385) {
                                              private_C[6][12] *= beta;
                                              if (nj >= t1 + c1 + 417) {
                                                private_C[6][13] *= beta;
                                                if (nj >= t1 + c1 + 449) {
                                                  private_C[6][14] *= beta;
                                                  if (nj >= t1 + c1 + 481)
                                                    private_C[6][15] *= beta;
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                      if (ni >= t0 + c0 + 225) {
                        private_C[7][0] *= beta;
                        if (nj >= t1 + c1 + 33) {
                          private_C[7][1] *= beta;
                          if (nj >= t1 + c1 + 65) {
                            private_C[7][2] *= beta;
                            if (nj >= t1 + c1 + 97) {
                              private_C[7][3] *= beta;
                              if (nj >= t1 + c1 + 129) {
                                private_C[7][4] *= beta;
                                if (nj >= t1 + c1 + 161) {
                                  private_C[7][5] *= beta;
                                  if (nj >= t1 + c1 + 193) {
                                    private_C[7][6] *= beta;
                                    if (nj >= t1 + c1 + 225) {
                                      private_C[7][7] *= beta;
                                      if (nj >= t1 + c1 + 257) {
                                        private_C[7][8] *= beta;
                                        if (nj >= t1 + c1 + 289) {
                                          private_C[7][9] *= beta;
                                          if (nj >= t1 + c1 + 321) {
                                            private_C[7][10] *= beta;
                                            if (nj >= t1 + c1 + 353) {
                                              private_C[7][11] *= beta;
                                              if (nj >= t1 + c1 + 385) {
                                                private_C[7][12] *= beta;
                                                if (nj >= t1 + c1 + 417) {
                                                  private_C[7][13] *= beta;
                                                  if (nj >= t1 + c1 + 449) {
                                                    private_C[7][14] *= beta;
                                                    if (nj >= t1 + c1 + 481)
                                                      private_C[7][15] *= beta;
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                        if (ni >= t0 + c0 + 257) {
                          private_C[8][0] *= beta;
                          if (nj >= t1 + c1 + 33) {
                            private_C[8][1] *= beta;
                            if (nj >= t1 + c1 + 65) {
                              private_C[8][2] *= beta;
                              if (nj >= t1 + c1 + 97) {
                                private_C[8][3] *= beta;
                                if (nj >= t1 + c1 + 129) {
                                  private_C[8][4] *= beta;
                                  if (nj >= t1 + c1 + 161) {
                                    private_C[8][5] *= beta;
                                    if (nj >= t1 + c1 + 193) {
                                      private_C[8][6] *= beta;
                                      if (nj >= t1 + c1 + 225) {
                                        private_C[8][7] *= beta;
                                        if (nj >= t1 + c1 + 257) {
                                          private_C[8][8] *= beta;
                                          if (nj >= t1 + c1 + 289) {
                                            private_C[8][9] *= beta;
                                            if (nj >= t1 + c1 + 321) {
                                              private_C[8][10] *= beta;
                                              if (nj >= t1 + c1 + 353) {
                                                private_C[8][11] *= beta;
                                                if (nj >= t1 + c1 + 385) {
                                                  private_C[8][12] *= beta;
                                                  if (nj >= t1 + c1 + 417) {
                                                    private_C[8][13] *= beta;
                                                    if (nj >= t1 + c1 + 449) {
                                                      private_C[8][14] *= beta;
                                                      if (nj >= t1 + c1 + 481)
                                                        private_C[8][15] *= beta;
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                          if (ni >= t0 + c0 + 289) {
                            private_C[9][0] *= beta;
                            if (nj >= t1 + c1 + 33) {
                              private_C[9][1] *= beta;
                              if (nj >= t1 + c1 + 65) {
                                private_C[9][2] *= beta;
                                if (nj >= t1 + c1 + 97) {
                                  private_C[9][3] *= beta;
                                  if (nj >= t1 + c1 + 129) {
                                    private_C[9][4] *= beta;
                                    if (nj >= t1 + c1 + 161) {
                                      private_C[9][5] *= beta;
                                      if (nj >= t1 + c1 + 193) {
                                        private_C[9][6] *= beta;
                                        if (nj >= t1 + c1 + 225) {
                                          private_C[9][7] *= beta;
                                          if (nj >= t1 + c1 + 257) {
                                            private_C[9][8] *= beta;
                                            if (nj >= t1 + c1 + 289) {
                                              private_C[9][9] *= beta;
                                              if (nj >= t1 + c1 + 321) {
                                                private_C[9][10] *= beta;
                                                if (nj >= t1 + c1 + 353) {
                                                  private_C[9][11] *= beta;
                                                  if (nj >= t1 + c1 + 385) {
                                                    private_C[9][12] *= beta;
                                                    if (nj >= t1 + c1 + 417) {
                                                      private_C[9][13] *= beta;
                                                      if (nj >= t1 + c1 + 449) {
                                                        private_C[9][14] *= beta;
                                                        if (nj >= t1 + c1 + 481)
                                                          private_C[9][15] *= beta;
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                            if (ni >= t0 + c0 + 321) {
                              private_C[10][0] *= beta;
                              if (nj >= t1 + c1 + 33) {
                                private_C[10][1] *= beta;
                                if (nj >= t1 + c1 + 65) {
                                  private_C[10][2] *= beta;
                                  if (nj >= t1 + c1 + 97) {
                                    private_C[10][3] *= beta;
                                    if (nj >= t1 + c1 + 129) {
                                      private_C[10][4] *= beta;
                                      if (nj >= t1 + c1 + 161) {
                                        private_C[10][5] *= beta;
                                        if (nj >= t1 + c1 + 193) {
                                          private_C[10][6] *= beta;
                                          if (nj >= t1 + c1 + 225) {
                                            private_C[10][7] *= beta;
                                            if (nj >= t1 + c1 + 257) {
                                              private_C[10][8] *= beta;
                                              if (nj >= t1 + c1 + 289) {
                                                private_C[10][9] *= beta;
                                                if (nj >= t1 + c1 + 321) {
                                                  private_C[10][10] *= beta;
                                                  if (nj >= t1 + c1 + 353) {
                                                    private_C[10][11] *= beta;
                                                    if (nj >= t1 + c1 + 385) {
                                                      private_C[10][12] *= beta;
                                                      if (nj >= t1 + c1 + 417) {
                                                        private_C[10][13] *= beta;
                                                        if (nj >= t1 + c1 + 449) {
                                                          private_C[10][14] *= beta;
                                                          if (nj >= t1 + c1 + 481)
                                                            private_C[10][15] *= beta;
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                              if (ni >= t0 + c0 + 353) {
                                private_C[11][0] *= beta;
                                if (nj >= t1 + c1 + 33) {
                                  private_C[11][1] *= beta;
                                  if (nj >= t1 + c1 + 65) {
                                    private_C[11][2] *= beta;
                                    if (nj >= t1 + c1 + 97) {
                                      private_C[11][3] *= beta;
                                      if (nj >= t1 + c1 + 129) {
                                        private_C[11][4] *= beta;
                                        if (nj >= t1 + c1 + 161) {
                                          private_C[11][5] *= beta;
                                          if (nj >= t1 + c1 + 193) {
                                            private_C[11][6] *= beta;
                                            if (nj >= t1 + c1 + 225) {
                                              private_C[11][7] *= beta;
                                              if (nj >= t1 + c1 + 257) {
                                                private_C[11][8] *= beta;
                                                if (nj >= t1 + c1 + 289) {
                                                  private_C[11][9] *= beta;
                                                  if (nj >= t1 + c1 + 321) {
                                                    private_C[11][10] *= beta;
                                                    if (nj >= t1 + c1 + 353) {
                                                      private_C[11][11] *= beta;
                                                      if (nj >= t1 + c1 + 385) {
                                                        private_C[11][12] *= beta;
                                                        if (nj >= t1 + c1 + 417) {
                                                          private_C[11][13] *= beta;
                                                          if (nj >= t1 + c1 + 449) {
                                                            private_C[11][14] *= beta;
                                                            if (nj >= t1 + c1 + 481)
                                                              private_C[11][15] *= beta;
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                                if (ni >= t0 + c0 + 385) {
                                  private_C[12][0] *= beta;
                                  if (nj >= t1 + c1 + 33) {
                                    private_C[12][1] *= beta;
                                    if (nj >= t1 + c1 + 65) {
                                      private_C[12][2] *= beta;
                                      if (nj >= t1 + c1 + 97) {
                                        private_C[12][3] *= beta;
                                        if (nj >= t1 + c1 + 129) {
                                          private_C[12][4] *= beta;
                                          if (nj >= t1 + c1 + 161) {
                                            private_C[12][5] *= beta;
                                            if (nj >= t1 + c1 + 193) {
                                              private_C[12][6] *= beta;
                                              if (nj >= t1 + c1 + 225) {
                                                private_C[12][7] *= beta;
                                                if (nj >= t1 + c1 + 257) {
                                                  private_C[12][8] *= beta;
                                                  if (nj >= t1 + c1 + 289) {
                                                    private_C[12][9] *= beta;
                                                    if (nj >= t1 + c1 + 321) {
                                                      private_C[12][10] *= beta;
                                                      if (nj >= t1 + c1 + 353) {
                                                        private_C[12][11] *= beta;
                                                        if (nj >= t1 + c1 + 385) {
                                                          private_C[12][12] *= beta;
                                                          if (nj >= t1 + c1 + 417) {
                                                            private_C[12][13] *= beta;
                                                            if (nj >= t1 + c1 + 449) {
                                                              private_C[12][14] *= beta;
                                                              if (nj >= t1 + c1 + 481)
                                                                private_C[12][15] *= beta;
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                  if (ni >= t0 + c0 + 417) {
                                    private_C[13][0] *= beta;
                                    if (nj >= t1 + c1 + 33) {
                                      private_C[13][1] *= beta;
                                      if (nj >= t1 + c1 + 65) {
                                        private_C[13][2] *= beta;
                                        if (nj >= t1 + c1 + 97) {
                                          private_C[13][3] *= beta;
                                          if (nj >= t1 + c1 + 129) {
                                            private_C[13][4] *= beta;
                                            if (nj >= t1 + c1 + 161) {
                                              private_C[13][5] *= beta;
                                              if (nj >= t1 + c1 + 193) {
                                                private_C[13][6] *= beta;
                                                if (nj >= t1 + c1 + 225) {
                                                  private_C[13][7] *= beta;
                                                  if (nj >= t1 + c1 + 257) {
                                                    private_C[13][8] *= beta;
                                                    if (nj >= t1 + c1 + 289) {
                                                      private_C[13][9] *= beta;
                                                      if (nj >= t1 + c1 + 321) {
                                                        private_C[13][10] *= beta;
                                                        if (nj >= t1 + c1 + 353) {
                                                          private_C[13][11] *= beta;
                                                          if (nj >= t1 + c1 + 385) {
                                                            private_C[13][12] *= beta;
                                                            if (nj >= t1 + c1 + 417) {
                                                              private_C[13][13] *= beta;
                                                              if (nj >= t1 + c1 + 449) {
                                                                private_C[13][14] *= beta;
                                                                if (nj >= t1 + c1 + 481)
                                                                  private_C[13][15] *= beta;
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                    if (ni >= t0 + c0 + 449) {
                                      private_C[14][0] *= beta;
                                      if (nj >= t1 + c1 + 33) {
                                        private_C[14][1] *= beta;
                                        if (nj >= t1 + c1 + 65) {
                                          private_C[14][2] *= beta;
                                          if (nj >= t1 + c1 + 97) {
                                            private_C[14][3] *= beta;
                                            if (nj >= t1 + c1 + 129) {
                                              private_C[14][4] *= beta;
                                              if (nj >= t1 + c1 + 161) {
                                                private_C[14][5] *= beta;
                                                if (nj >= t1 + c1 + 193) {
                                                  private_C[14][6] *= beta;
                                                  if (nj >= t1 + c1 + 225) {
                                                    private_C[14][7] *= beta;
                                                    if (nj >= t1 + c1 + 257) {
                                                      private_C[14][8] *= beta;
                                                      if (nj >= t1 + c1 + 289) {
                                                        private_C[14][9] *= beta;
                                                        if (nj >= t1 + c1 + 321) {
                                                          private_C[14][10] *= beta;
                                                          if (nj >= t1 + c1 + 353) {
                                                            private_C[14][11] *= beta;
                                                            if (nj >= t1 + c1 + 385) {
                                                              private_C[14][12] *= beta;
                                                              if (nj >= t1 + c1 + 417) {
                                                                private_C[14][13] *= beta;
                                                                if (nj >= t1 + c1 + 449) {
                                                                  private_C[14][14] *= beta;
                                                                  if (nj >= t1 + c1 + 481)
                                                                    private_C[14][15] *= beta;
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                      if (ni >= t0 + c0 + 481) {
                                        private_C[15][0] *= beta;
                                        if (nj >= t1 + c1 + 33) {
                                          private_C[15][1] *= beta;
                                          if (nj >= t1 + c1 + 65) {
                                            private_C[15][2] *= beta;
                                            if (nj >= t1 + c1 + 97) {
                                              private_C[15][3] *= beta;
                                              if (nj >= t1 + c1 + 129) {
                                                private_C[15][4] *= beta;
                                                if (nj >= t1 + c1 + 161) {
                                                  private_C[15][5] *= beta;
                                                  if (nj >= t1 + c1 + 193) {
                                                    private_C[15][6] *= beta;
                                                    if (nj >= t1 + c1 + 225) {
                                                      private_C[15][7] *= beta;
                                                      if (nj >= t1 + c1 + 257) {
                                                        private_C[15][8] *= beta;
                                                        if (nj >= t1 + c1 + 289) {
                                                          private_C[15][9] *= beta;
                                                          if (nj >= t1 + c1 + 321) {
                                                            private_C[15][10] *= beta;
                                                            if (nj >= t1 + c1 + 353) {
                                                              private_C[15][11] *= beta;
                                                              if (nj >= t1 + c1 + 385) {
                                                                private_C[15][12] *= beta;
                                                                if (nj >= t1 + c1 + 417) {
                                                                  private_C[15][13] *= beta;
                                                                  if (nj >= t1 + c1 + 449) {
                                                                    private_C[15][14] *= beta;
                                                                    if (nj >= t1 + c1 + 481)
                                                                      private_C[15][15] *= beta;
                                                                  }
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          for (int c2 = 0; c2 < nk; c2 += 512)
            for (int c3 = 0; c3 <= ppcg_min(511, nk - c2 - 1); c3 += 1) {
              private_C[0][0] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
              if (nj >= t1 + c1 + 33) {
                private_C[0][1] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                if (nj >= t1 + c1 + 65) {
                  private_C[0][2] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                  if (nj >= t1 + c1 + 97) {
                    private_C[0][3] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                    if (nj >= t1 + c1 + 129) {
                      private_C[0][4] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                      if (nj >= t1 + c1 + 161) {
                        private_C[0][5] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                        if (nj >= t1 + c1 + 193) {
                          private_C[0][6] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                          if (nj >= t1 + c1 + 225) {
                            private_C[0][7] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                            if (nj >= t1 + c1 + 257) {
                              private_C[0][8] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                              if (nj >= t1 + c1 + 289) {
                                private_C[0][9] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                if (nj >= t1 + c1 + 321) {
                                  private_C[0][10] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                  if (nj >= t1 + c1 + 353) {
                                    private_C[0][11] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                    if (nj >= t1 + c1 + 385) {
                                      private_C[0][12] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                      if (nj >= t1 + c1 + 417) {
                                        private_C[0][13] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                        if (nj >= t1 + c1 + 449) {
                                          private_C[0][14] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                          if (nj >= t1 + c1 + 481)
                                            private_C[0][15] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
              if (ni >= t0 + c0 + 33) {
                private_C[1][0] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                if (nj >= t1 + c1 + 33) {
                  private_C[1][1] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                  if (nj >= t1 + c1 + 65) {
                    private_C[1][2] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                    if (nj >= t1 + c1 + 97) {
                      private_C[1][3] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                      if (nj >= t1 + c1 + 129) {
                        private_C[1][4] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                        if (nj >= t1 + c1 + 161) {
                          private_C[1][5] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                          if (nj >= t1 + c1 + 193) {
                            private_C[1][6] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                            if (nj >= t1 + c1 + 225) {
                              private_C[1][7] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                              if (nj >= t1 + c1 + 257) {
                                private_C[1][8] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                if (nj >= t1 + c1 + 289) {
                                  private_C[1][9] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                  if (nj >= t1 + c1 + 321) {
                                    private_C[1][10] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                    if (nj >= t1 + c1 + 353) {
                                      private_C[1][11] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                      if (nj >= t1 + c1 + 385) {
                                        private_C[1][12] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                        if (nj >= t1 + c1 + 417) {
                                          private_C[1][13] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                          if (nj >= t1 + c1 + 449) {
                                            private_C[1][14] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                            if (nj >= t1 + c1 + 481)
                                              private_C[1][15] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
                if (ni >= t0 + c0 + 65) {
                  private_C[2][0] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                  if (nj >= t1 + c1 + 33) {
                    private_C[2][1] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                    if (nj >= t1 + c1 + 65) {
                      private_C[2][2] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                      if (nj >= t1 + c1 + 97) {
                        private_C[2][3] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                        if (nj >= t1 + c1 + 129) {
                          private_C[2][4] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                          if (nj >= t1 + c1 + 161) {
                            private_C[2][5] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                            if (nj >= t1 + c1 + 193) {
                              private_C[2][6] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                              if (nj >= t1 + c1 + 225) {
                                private_C[2][7] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                if (nj >= t1 + c1 + 257) {
                                  private_C[2][8] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                  if (nj >= t1 + c1 + 289) {
                                    private_C[2][9] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                    if (nj >= t1 + c1 + 321) {
                                      private_C[2][10] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                      if (nj >= t1 + c1 + 353) {
                                        private_C[2][11] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                        if (nj >= t1 + c1 + 385) {
                                          private_C[2][12] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                          if (nj >= t1 + c1 + 417) {
                                            private_C[2][13] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                            if (nj >= t1 + c1 + 449) {
                                              private_C[2][14] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                              if (nj >= t1 + c1 + 481)
                                                private_C[2][15] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                  if (ni >= t0 + c0 + 97) {
                    private_C[3][0] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                    if (nj >= t1 + c1 + 33) {
                      private_C[3][1] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                      if (nj >= t1 + c1 + 65) {
                        private_C[3][2] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                        if (nj >= t1 + c1 + 97) {
                          private_C[3][3] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                          if (nj >= t1 + c1 + 129) {
                            private_C[3][4] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                            if (nj >= t1 + c1 + 161) {
                              private_C[3][5] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                              if (nj >= t1 + c1 + 193) {
                                private_C[3][6] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                                if (nj >= t1 + c1 + 225) {
                                  private_C[3][7] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                  if (nj >= t1 + c1 + 257) {
                                    private_C[3][8] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                    if (nj >= t1 + c1 + 289) {
                                      private_C[3][9] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                      if (nj >= t1 + c1 + 321) {
                                        private_C[3][10] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                        if (nj >= t1 + c1 + 353) {
                                          private_C[3][11] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                          if (nj >= t1 + c1 + 385) {
                                            private_C[3][12] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                            if (nj >= t1 + c1 + 417) {
                                              private_C[3][13] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                              if (nj >= t1 + c1 + 449) {
                                                private_C[3][14] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                                if (nj >= t1 + c1 + 481)
                                                  private_C[3][15] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                    if (ni >= t0 + c0 + 129) {
                      private_C[4][0] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                      if (nj >= t1 + c1 + 33) {
                        private_C[4][1] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                        if (nj >= t1 + c1 + 65) {
                          private_C[4][2] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                          if (nj >= t1 + c1 + 97) {
                            private_C[4][3] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                            if (nj >= t1 + c1 + 129) {
                              private_C[4][4] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                              if (nj >= t1 + c1 + 161) {
                                private_C[4][5] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                                if (nj >= t1 + c1 + 193) {
                                  private_C[4][6] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                                  if (nj >= t1 + c1 + 225) {
                                    private_C[4][7] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                    if (nj >= t1 + c1 + 257) {
                                      private_C[4][8] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                      if (nj >= t1 + c1 + 289) {
                                        private_C[4][9] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                        if (nj >= t1 + c1 + 321) {
                                          private_C[4][10] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                          if (nj >= t1 + c1 + 353) {
                                            private_C[4][11] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                            if (nj >= t1 + c1 + 385) {
                                              private_C[4][12] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                              if (nj >= t1 + c1 + 417) {
                                                private_C[4][13] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                                if (nj >= t1 + c1 + 449) {
                                                  private_C[4][14] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                                  if (nj >= t1 + c1 + 481)
                                                    private_C[4][15] += ((alpha * A[(t0 + c0 + 128) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                      if (ni >= t0 + c0 + 161) {
                        private_C[5][0] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                        if (nj >= t1 + c1 + 33) {
                          private_C[5][1] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                          if (nj >= t1 + c1 + 65) {
                            private_C[5][2] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                            if (nj >= t1 + c1 + 97) {
                              private_C[5][3] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                              if (nj >= t1 + c1 + 129) {
                                private_C[5][4] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                                if (nj >= t1 + c1 + 161) {
                                  private_C[5][5] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                                  if (nj >= t1 + c1 + 193) {
                                    private_C[5][6] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                                    if (nj >= t1 + c1 + 225) {
                                      private_C[5][7] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                      if (nj >= t1 + c1 + 257) {
                                        private_C[5][8] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                        if (nj >= t1 + c1 + 289) {
                                          private_C[5][9] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                          if (nj >= t1 + c1 + 321) {
                                            private_C[5][10] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                            if (nj >= t1 + c1 + 353) {
                                              private_C[5][11] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                              if (nj >= t1 + c1 + 385) {
                                                private_C[5][12] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                                if (nj >= t1 + c1 + 417) {
                                                  private_C[5][13] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                                  if (nj >= t1 + c1 + 449) {
                                                    private_C[5][14] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                                    if (nj >= t1 + c1 + 481)
                                                      private_C[5][15] += ((alpha * A[(t0 + c0 + 160) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                        if (ni >= t0 + c0 + 193) {
                          private_C[6][0] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                          if (nj >= t1 + c1 + 33) {
                            private_C[6][1] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                            if (nj >= t1 + c1 + 65) {
                              private_C[6][2] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                              if (nj >= t1 + c1 + 97) {
                                private_C[6][3] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                if (nj >= t1 + c1 + 129) {
                                  private_C[6][4] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                                  if (nj >= t1 + c1 + 161) {
                                    private_C[6][5] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                                    if (nj >= t1 + c1 + 193) {
                                      private_C[6][6] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                                      if (nj >= t1 + c1 + 225) {
                                        private_C[6][7] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                        if (nj >= t1 + c1 + 257) {
                                          private_C[6][8] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                          if (nj >= t1 + c1 + 289) {
                                            private_C[6][9] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                            if (nj >= t1 + c1 + 321) {
                                              private_C[6][10] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                              if (nj >= t1 + c1 + 353) {
                                                private_C[6][11] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                                if (nj >= t1 + c1 + 385) {
                                                  private_C[6][12] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                                  if (nj >= t1 + c1 + 417) {
                                                    private_C[6][13] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                                    if (nj >= t1 + c1 + 449) {
                                                      private_C[6][14] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                                      if (nj >= t1 + c1 + 481)
                                                        private_C[6][15] += ((alpha * A[(t0 + c0 + 192) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                          if (ni >= t0 + c0 + 225) {
                            private_C[7][0] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                            if (nj >= t1 + c1 + 33) {
                              private_C[7][1] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                              if (nj >= t1 + c1 + 65) {
                                private_C[7][2] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                if (nj >= t1 + c1 + 97) {
                                  private_C[7][3] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                  if (nj >= t1 + c1 + 129) {
                                    private_C[7][4] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                                    if (nj >= t1 + c1 + 161) {
                                      private_C[7][5] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                                      if (nj >= t1 + c1 + 193) {
                                        private_C[7][6] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                                        if (nj >= t1 + c1 + 225) {
                                          private_C[7][7] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                          if (nj >= t1 + c1 + 257) {
                                            private_C[7][8] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                            if (nj >= t1 + c1 + 289) {
                                              private_C[7][9] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                              if (nj >= t1 + c1 + 321) {
                                                private_C[7][10] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                                if (nj >= t1 + c1 + 353) {
                                                  private_C[7][11] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                                  if (nj >= t1 + c1 + 385) {
                                                    private_C[7][12] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                                    if (nj >= t1 + c1 + 417) {
                                                      private_C[7][13] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                                      if (nj >= t1 + c1 + 449) {
                                                        private_C[7][14] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                                        if (nj >= t1 + c1 + 481)
                                                          private_C[7][15] += ((alpha * A[(t0 + c0 + 224) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                            if (ni >= t0 + c0 + 257) {
                              private_C[8][0] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                              if (nj >= t1 + c1 + 33) {
                                private_C[8][1] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                if (nj >= t1 + c1 + 65) {
                                  private_C[8][2] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                  if (nj >= t1 + c1 + 97) {
                                    private_C[8][3] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                    if (nj >= t1 + c1 + 129) {
                                      private_C[8][4] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                                      if (nj >= t1 + c1 + 161) {
                                        private_C[8][5] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                                        if (nj >= t1 + c1 + 193) {
                                          private_C[8][6] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                                          if (nj >= t1 + c1 + 225) {
                                            private_C[8][7] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                            if (nj >= t1 + c1 + 257) {
                                              private_C[8][8] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                              if (nj >= t1 + c1 + 289) {
                                                private_C[8][9] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                                if (nj >= t1 + c1 + 321) {
                                                  private_C[8][10] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                                  if (nj >= t1 + c1 + 353) {
                                                    private_C[8][11] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                                    if (nj >= t1 + c1 + 385) {
                                                      private_C[8][12] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                                      if (nj >= t1 + c1 + 417) {
                                                        private_C[8][13] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                                        if (nj >= t1 + c1 + 449) {
                                                          private_C[8][14] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                                          if (nj >= t1 + c1 + 481)
                                                            private_C[8][15] += ((alpha * A[(t0 + c0 + 256) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                              if (ni >= t0 + c0 + 289) {
                                private_C[9][0] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                if (nj >= t1 + c1 + 33) {
                                  private_C[9][1] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                  if (nj >= t1 + c1 + 65) {
                                    private_C[9][2] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                    if (nj >= t1 + c1 + 97) {
                                      private_C[9][3] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                      if (nj >= t1 + c1 + 129) {
                                        private_C[9][4] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                                        if (nj >= t1 + c1 + 161) {
                                          private_C[9][5] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                                          if (nj >= t1 + c1 + 193) {
                                            private_C[9][6] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                                            if (nj >= t1 + c1 + 225) {
                                              private_C[9][7] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                              if (nj >= t1 + c1 + 257) {
                                                private_C[9][8] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                                if (nj >= t1 + c1 + 289) {
                                                  private_C[9][9] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                                  if (nj >= t1 + c1 + 321) {
                                                    private_C[9][10] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                                    if (nj >= t1 + c1 + 353) {
                                                      private_C[9][11] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                                      if (nj >= t1 + c1 + 385) {
                                                        private_C[9][12] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                                        if (nj >= t1 + c1 + 417) {
                                                          private_C[9][13] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                                          if (nj >= t1 + c1 + 449) {
                                                            private_C[9][14] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                                            if (nj >= t1 + c1 + 481)
                                                              private_C[9][15] += ((alpha * A[(t0 + c0 + 288) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                                if (ni >= t0 + c0 + 321) {
                                  private_C[10][0] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                  if (nj >= t1 + c1 + 33) {
                                    private_C[10][1] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                    if (nj >= t1 + c1 + 65) {
                                      private_C[10][2] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                      if (nj >= t1 + c1 + 97) {
                                        private_C[10][3] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                        if (nj >= t1 + c1 + 129) {
                                          private_C[10][4] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                                          if (nj >= t1 + c1 + 161) {
                                            private_C[10][5] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                                            if (nj >= t1 + c1 + 193) {
                                              private_C[10][6] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                                              if (nj >= t1 + c1 + 225) {
                                                private_C[10][7] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                                if (nj >= t1 + c1 + 257) {
                                                  private_C[10][8] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                                  if (nj >= t1 + c1 + 289) {
                                                    private_C[10][9] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                                    if (nj >= t1 + c1 + 321) {
                                                      private_C[10][10] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                                      if (nj >= t1 + c1 + 353) {
                                                        private_C[10][11] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                                        if (nj >= t1 + c1 + 385) {
                                                          private_C[10][12] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                                          if (nj >= t1 + c1 + 417) {
                                                            private_C[10][13] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                                            if (nj >= t1 + c1 + 449) {
                                                              private_C[10][14] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                                              if (nj >= t1 + c1 + 481)
                                                                private_C[10][15] += ((alpha * A[(t0 + c0 + 320) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                  if (ni >= t0 + c0 + 353) {
                                    private_C[11][0] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                    if (nj >= t1 + c1 + 33) {
                                      private_C[11][1] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                      if (nj >= t1 + c1 + 65) {
                                        private_C[11][2] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                        if (nj >= t1 + c1 + 97) {
                                          private_C[11][3] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                          if (nj >= t1 + c1 + 129) {
                                            private_C[11][4] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                                            if (nj >= t1 + c1 + 161) {
                                              private_C[11][5] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                                              if (nj >= t1 + c1 + 193) {
                                                private_C[11][6] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                                                if (nj >= t1 + c1 + 225) {
                                                  private_C[11][7] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                                  if (nj >= t1 + c1 + 257) {
                                                    private_C[11][8] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                                    if (nj >= t1 + c1 + 289) {
                                                      private_C[11][9] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                                      if (nj >= t1 + c1 + 321) {
                                                        private_C[11][10] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                                        if (nj >= t1 + c1 + 353) {
                                                          private_C[11][11] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                                          if (nj >= t1 + c1 + 385) {
                                                            private_C[11][12] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                                            if (nj >= t1 + c1 + 417) {
                                                              private_C[11][13] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                                              if (nj >= t1 + c1 + 449) {
                                                                private_C[11][14] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                                                if (nj >= t1 + c1 + 481)
                                                                  private_C[11][15] += ((alpha * A[(t0 + c0 + 352) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                    if (ni >= t0 + c0 + 385) {
                                      private_C[12][0] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                      if (nj >= t1 + c1 + 33) {
                                        private_C[12][1] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                        if (nj >= t1 + c1 + 65) {
                                          private_C[12][2] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                          if (nj >= t1 + c1 + 97) {
                                            private_C[12][3] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                            if (nj >= t1 + c1 + 129) {
                                              private_C[12][4] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                                              if (nj >= t1 + c1 + 161) {
                                                private_C[12][5] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                                                if (nj >= t1 + c1 + 193) {
                                                  private_C[12][6] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                                                  if (nj >= t1 + c1 + 225) {
                                                    private_C[12][7] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                                    if (nj >= t1 + c1 + 257) {
                                                      private_C[12][8] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                                      if (nj >= t1 + c1 + 289) {
                                                        private_C[12][9] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                                        if (nj >= t1 + c1 + 321) {
                                                          private_C[12][10] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                                          if (nj >= t1 + c1 + 353) {
                                                            private_C[12][11] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                                            if (nj >= t1 + c1 + 385) {
                                                              private_C[12][12] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                                              if (nj >= t1 + c1 + 417) {
                                                                private_C[12][13] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                                                if (nj >= t1 + c1 + 449) {
                                                                  private_C[12][14] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                                                  if (nj >= t1 + c1 + 481)
                                                                    private_C[12][15] += ((alpha * A[(t0 + c0 + 384) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                      if (ni >= t0 + c0 + 417) {
                                        private_C[13][0] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                        if (nj >= t1 + c1 + 33) {
                                          private_C[13][1] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                          if (nj >= t1 + c1 + 65) {
                                            private_C[13][2] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                            if (nj >= t1 + c1 + 97) {
                                              private_C[13][3] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                              if (nj >= t1 + c1 + 129) {
                                                private_C[13][4] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                                                if (nj >= t1 + c1 + 161) {
                                                  private_C[13][5] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                                                  if (nj >= t1 + c1 + 193) {
                                                    private_C[13][6] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                                                    if (nj >= t1 + c1 + 225) {
                                                      private_C[13][7] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                                      if (nj >= t1 + c1 + 257) {
                                                        private_C[13][8] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                                        if (nj >= t1 + c1 + 289) {
                                                          private_C[13][9] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                                          if (nj >= t1 + c1 + 321) {
                                                            private_C[13][10] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                                            if (nj >= t1 + c1 + 353) {
                                                              private_C[13][11] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                                              if (nj >= t1 + c1 + 385) {
                                                                private_C[13][12] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                                                if (nj >= t1 + c1 + 417) {
                                                                  private_C[13][13] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                                                  if (nj >= t1 + c1 + 449) {
                                                                    private_C[13][14] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                                                    if (nj >= t1 + c1 + 481)
                                                                      private_C[13][15] += ((alpha * A[(t0 + c0 + 416) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                                                  }
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                        if (ni >= t0 + c0 + 449) {
                                          private_C[14][0] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                          if (nj >= t1 + c1 + 33) {
                                            private_C[14][1] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                            if (nj >= t1 + c1 + 65) {
                                              private_C[14][2] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                              if (nj >= t1 + c1 + 97) {
                                                private_C[14][3] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                                if (nj >= t1 + c1 + 129) {
                                                  private_C[14][4] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                                                  if (nj >= t1 + c1 + 161) {
                                                    private_C[14][5] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                                                    if (nj >= t1 + c1 + 193) {
                                                      private_C[14][6] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                                                      if (nj >= t1 + c1 + 225) {
                                                        private_C[14][7] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                                        if (nj >= t1 + c1 + 257) {
                                                          private_C[14][8] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                                          if (nj >= t1 + c1 + 289) {
                                                            private_C[14][9] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                                            if (nj >= t1 + c1 + 321) {
                                                              private_C[14][10] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                                              if (nj >= t1 + c1 + 353) {
                                                                private_C[14][11] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                                                if (nj >= t1 + c1 + 385) {
                                                                  private_C[14][12] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                                                  if (nj >= t1 + c1 + 417) {
                                                                    private_C[14][13] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                                                    if (nj >= t1 + c1 + 449) {
                                                                      private_C[14][14] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                                                      if (nj >= t1 + c1 + 481)
                                                                        private_C[14][15] += ((alpha * A[(t0 + c0 + 448) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                                                    }
                                                                  }
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                          if (ni >= t0 + c0 + 481) {
                                            private_C[15][0] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                                            if (nj >= t1 + c1 + 33) {
                                              private_C[15][1] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                                              if (nj >= t1 + c1 + 65) {
                                                private_C[15][2] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 64)]);
                                                if (nj >= t1 + c1 + 97) {
                                                  private_C[15][3] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 96)]);
                                                  if (nj >= t1 + c1 + 129) {
                                                    private_C[15][4] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 128)]);
                                                    if (nj >= t1 + c1 + 161) {
                                                      private_C[15][5] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 160)]);
                                                      if (nj >= t1 + c1 + 193) {
                                                        private_C[15][6] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 192)]);
                                                        if (nj >= t1 + c1 + 225) {
                                                          private_C[15][7] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 224)]);
                                                          if (nj >= t1 + c1 + 257) {
                                                            private_C[15][8] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 256)]);
                                                            if (nj >= t1 + c1 + 289) {
                                                              private_C[15][9] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 288)]);
                                                              if (nj >= t1 + c1 + 321) {
                                                                private_C[15][10] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 320)]);
                                                                if (nj >= t1 + c1 + 353) {
                                                                  private_C[15][11] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 352)]);
                                                                  if (nj >= t1 + c1 + 385) {
                                                                    private_C[15][12] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 384)]);
                                                                    if (nj >= t1 + c1 + 417) {
                                                                      private_C[15][13] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 416)]);
                                                                      if (nj >= t1 + c1 + 449) {
                                                                        private_C[15][14] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 448)]);
                                                                        if (nj >= t1 + c1 + 481)
                                                                          private_C[15][15] += ((alpha * A[(t0 + c0 + 480) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 480)]);
                                                                      }
                                                                    }
                                                                  }
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          C[(t0 + c0) * nj + (t1 + c1)] = private_C[0][0];
          if (nj >= t1 + c1 + 33) {
            C[(t0 + c0) * nj + (t1 + c1 + 32)] = private_C[0][1];
            if (nj >= t1 + c1 + 65) {
              C[(t0 + c0) * nj + (t1 + c1 + 64)] = private_C[0][2];
              if (nj >= t1 + c1 + 97) {
                C[(t0 + c0) * nj + (t1 + c1 + 96)] = private_C[0][3];
                if (nj >= t1 + c1 + 129) {
                  C[(t0 + c0) * nj + (t1 + c1 + 128)] = private_C[0][4];
                  if (nj >= t1 + c1 + 161) {
                    C[(t0 + c0) * nj + (t1 + c1 + 160)] = private_C[0][5];
                    if (nj >= t1 + c1 + 193) {
                      C[(t0 + c0) * nj + (t1 + c1 + 192)] = private_C[0][6];
                      if (nj >= t1 + c1 + 225) {
                        C[(t0 + c0) * nj + (t1 + c1 + 224)] = private_C[0][7];
                        if (nj >= t1 + c1 + 257) {
                          C[(t0 + c0) * nj + (t1 + c1 + 256)] = private_C[0][8];
                          if (nj >= t1 + c1 + 289) {
                            C[(t0 + c0) * nj + (t1 + c1 + 288)] = private_C[0][9];
                            if (nj >= t1 + c1 + 321) {
                              C[(t0 + c0) * nj + (t1 + c1 + 320)] = private_C[0][10];
                              if (nj >= t1 + c1 + 353) {
                                C[(t0 + c0) * nj + (t1 + c1 + 352)] = private_C[0][11];
                                if (nj >= t1 + c1 + 385) {
                                  C[(t0 + c0) * nj + (t1 + c1 + 384)] = private_C[0][12];
                                  if (nj >= t1 + c1 + 417) {
                                    C[(t0 + c0) * nj + (t1 + c1 + 416)] = private_C[0][13];
                                    if (nj >= t1 + c1 + 449) {
                                      C[(t0 + c0) * nj + (t1 + c1 + 448)] = private_C[0][14];
                                      if (nj >= t1 + c1 + 481)
                                        C[(t0 + c0) * nj + (t1 + c1 + 480)] = private_C[0][15];
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
          if (ni >= t0 + c0 + 33) {
            C[(t0 + c0 + 32) * nj + (t1 + c1)] = private_C[1][0];
            if (nj >= t1 + c1 + 33) {
              C[(t0 + c0 + 32) * nj + (t1 + c1 + 32)] = private_C[1][1];
              if (nj >= t1 + c1 + 65) {
                C[(t0 + c0 + 32) * nj + (t1 + c1 + 64)] = private_C[1][2];
                if (nj >= t1 + c1 + 97) {
                  C[(t0 + c0 + 32) * nj + (t1 + c1 + 96)] = private_C[1][3];
                  if (nj >= t1 + c1 + 129) {
                    C[(t0 + c0 + 32) * nj + (t1 + c1 + 128)] = private_C[1][4];
                    if (nj >= t1 + c1 + 161) {
                      C[(t0 + c0 + 32) * nj + (t1 + c1 + 160)] = private_C[1][5];
                      if (nj >= t1 + c1 + 193) {
                        C[(t0 + c0 + 32) * nj + (t1 + c1 + 192)] = private_C[1][6];
                        if (nj >= t1 + c1 + 225) {
                          C[(t0 + c0 + 32) * nj + (t1 + c1 + 224)] = private_C[1][7];
                          if (nj >= t1 + c1 + 257) {
                            C[(t0 + c0 + 32) * nj + (t1 + c1 + 256)] = private_C[1][8];
                            if (nj >= t1 + c1 + 289) {
                              C[(t0 + c0 + 32) * nj + (t1 + c1 + 288)] = private_C[1][9];
                              if (nj >= t1 + c1 + 321) {
                                C[(t0 + c0 + 32) * nj + (t1 + c1 + 320)] = private_C[1][10];
                                if (nj >= t1 + c1 + 353) {
                                  C[(t0 + c0 + 32) * nj + (t1 + c1 + 352)] = private_C[1][11];
                                  if (nj >= t1 + c1 + 385) {
                                    C[(t0 + c0 + 32) * nj + (t1 + c1 + 384)] = private_C[1][12];
                                    if (nj >= t1 + c1 + 417) {
                                      C[(t0 + c0 + 32) * nj + (t1 + c1 + 416)] = private_C[1][13];
                                      if (nj >= t1 + c1 + 449) {
                                        C[(t0 + c0 + 32) * nj + (t1 + c1 + 448)] = private_C[1][14];
                                        if (nj >= t1 + c1 + 481)
                                          C[(t0 + c0 + 32) * nj + (t1 + c1 + 480)] = private_C[1][15];
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
            if (ni >= t0 + c0 + 65) {
              C[(t0 + c0 + 64) * nj + (t1 + c1)] = private_C[2][0];
              if (nj >= t1 + c1 + 33) {
                C[(t0 + c0 + 64) * nj + (t1 + c1 + 32)] = private_C[2][1];
                if (nj >= t1 + c1 + 65) {
                  C[(t0 + c0 + 64) * nj + (t1 + c1 + 64)] = private_C[2][2];
                  if (nj >= t1 + c1 + 97) {
                    C[(t0 + c0 + 64) * nj + (t1 + c1 + 96)] = private_C[2][3];
                    if (nj >= t1 + c1 + 129) {
                      C[(t0 + c0 + 64) * nj + (t1 + c1 + 128)] = private_C[2][4];
                      if (nj >= t1 + c1 + 161) {
                        C[(t0 + c0 + 64) * nj + (t1 + c1 + 160)] = private_C[2][5];
                        if (nj >= t1 + c1 + 193) {
                          C[(t0 + c0 + 64) * nj + (t1 + c1 + 192)] = private_C[2][6];
                          if (nj >= t1 + c1 + 225) {
                            C[(t0 + c0 + 64) * nj + (t1 + c1 + 224)] = private_C[2][7];
                            if (nj >= t1 + c1 + 257) {
                              C[(t0 + c0 + 64) * nj + (t1 + c1 + 256)] = private_C[2][8];
                              if (nj >= t1 + c1 + 289) {
                                C[(t0 + c0 + 64) * nj + (t1 + c1 + 288)] = private_C[2][9];
                                if (nj >= t1 + c1 + 321) {
                                  C[(t0 + c0 + 64) * nj + (t1 + c1 + 320)] = private_C[2][10];
                                  if (nj >= t1 + c1 + 353) {
                                    C[(t0 + c0 + 64) * nj + (t1 + c1 + 352)] = private_C[2][11];
                                    if (nj >= t1 + c1 + 385) {
                                      C[(t0 + c0 + 64) * nj + (t1 + c1 + 384)] = private_C[2][12];
                                      if (nj >= t1 + c1 + 417) {
                                        C[(t0 + c0 + 64) * nj + (t1 + c1 + 416)] = private_C[2][13];
                                        if (nj >= t1 + c1 + 449) {
                                          C[(t0 + c0 + 64) * nj + (t1 + c1 + 448)] = private_C[2][14];
                                          if (nj >= t1 + c1 + 481)
                                            C[(t0 + c0 + 64) * nj + (t1 + c1 + 480)] = private_C[2][15];
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
              if (ni >= t0 + c0 + 97) {
                C[(t0 + c0 + 96) * nj + (t1 + c1)] = private_C[3][0];
                if (nj >= t1 + c1 + 33) {
                  C[(t0 + c0 + 96) * nj + (t1 + c1 + 32)] = private_C[3][1];
                  if (nj >= t1 + c1 + 65) {
                    C[(t0 + c0 + 96) * nj + (t1 + c1 + 64)] = private_C[3][2];
                    if (nj >= t1 + c1 + 97) {
                      C[(t0 + c0 + 96) * nj + (t1 + c1 + 96)] = private_C[3][3];
                      if (nj >= t1 + c1 + 129) {
                        C[(t0 + c0 + 96) * nj + (t1 + c1 + 128)] = private_C[3][4];
                        if (nj >= t1 + c1 + 161) {
                          C[(t0 + c0 + 96) * nj + (t1 + c1 + 160)] = private_C[3][5];
                          if (nj >= t1 + c1 + 193) {
                            C[(t0 + c0 + 96) * nj + (t1 + c1 + 192)] = private_C[3][6];
                            if (nj >= t1 + c1 + 225) {
                              C[(t0 + c0 + 96) * nj + (t1 + c1 + 224)] = private_C[3][7];
                              if (nj >= t1 + c1 + 257) {
                                C[(t0 + c0 + 96) * nj + (t1 + c1 + 256)] = private_C[3][8];
                                if (nj >= t1 + c1 + 289) {
                                  C[(t0 + c0 + 96) * nj + (t1 + c1 + 288)] = private_C[3][9];
                                  if (nj >= t1 + c1 + 321) {
                                    C[(t0 + c0 + 96) * nj + (t1 + c1 + 320)] = private_C[3][10];
                                    if (nj >= t1 + c1 + 353) {
                                      C[(t0 + c0 + 96) * nj + (t1 + c1 + 352)] = private_C[3][11];
                                      if (nj >= t1 + c1 + 385) {
                                        C[(t0 + c0 + 96) * nj + (t1 + c1 + 384)] = private_C[3][12];
                                        if (nj >= t1 + c1 + 417) {
                                          C[(t0 + c0 + 96) * nj + (t1 + c1 + 416)] = private_C[3][13];
                                          if (nj >= t1 + c1 + 449) {
                                            C[(t0 + c0 + 96) * nj + (t1 + c1 + 448)] = private_C[3][14];
                                            if (nj >= t1 + c1 + 481)
                                              C[(t0 + c0 + 96) * nj + (t1 + c1 + 480)] = private_C[3][15];
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
                if (ni >= t0 + c0 + 129) {
                  C[(t0 + c0 + 128) * nj + (t1 + c1)] = private_C[4][0];
                  if (nj >= t1 + c1 + 33) {
                    C[(t0 + c0 + 128) * nj + (t1 + c1 + 32)] = private_C[4][1];
                    if (nj >= t1 + c1 + 65) {
                      C[(t0 + c0 + 128) * nj + (t1 + c1 + 64)] = private_C[4][2];
                      if (nj >= t1 + c1 + 97) {
                        C[(t0 + c0 + 128) * nj + (t1 + c1 + 96)] = private_C[4][3];
                        if (nj >= t1 + c1 + 129) {
                          C[(t0 + c0 + 128) * nj + (t1 + c1 + 128)] = private_C[4][4];
                          if (nj >= t1 + c1 + 161) {
                            C[(t0 + c0 + 128) * nj + (t1 + c1 + 160)] = private_C[4][5];
                            if (nj >= t1 + c1 + 193) {
                              C[(t0 + c0 + 128) * nj + (t1 + c1 + 192)] = private_C[4][6];
                              if (nj >= t1 + c1 + 225) {
                                C[(t0 + c0 + 128) * nj + (t1 + c1 + 224)] = private_C[4][7];
                                if (nj >= t1 + c1 + 257) {
                                  C[(t0 + c0 + 128) * nj + (t1 + c1 + 256)] = private_C[4][8];
                                  if (nj >= t1 + c1 + 289) {
                                    C[(t0 + c0 + 128) * nj + (t1 + c1 + 288)] = private_C[4][9];
                                    if (nj >= t1 + c1 + 321) {
                                      C[(t0 + c0 + 128) * nj + (t1 + c1 + 320)] = private_C[4][10];
                                      if (nj >= t1 + c1 + 353) {
                                        C[(t0 + c0 + 128) * nj + (t1 + c1 + 352)] = private_C[4][11];
                                        if (nj >= t1 + c1 + 385) {
                                          C[(t0 + c0 + 128) * nj + (t1 + c1 + 384)] = private_C[4][12];
                                          if (nj >= t1 + c1 + 417) {
                                            C[(t0 + c0 + 128) * nj + (t1 + c1 + 416)] = private_C[4][13];
                                            if (nj >= t1 + c1 + 449) {
                                              C[(t0 + c0 + 128) * nj + (t1 + c1 + 448)] = private_C[4][14];
                                              if (nj >= t1 + c1 + 481)
                                                C[(t0 + c0 + 128) * nj + (t1 + c1 + 480)] = private_C[4][15];
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                  if (ni >= t0 + c0 + 161) {
                    C[(t0 + c0 + 160) * nj + (t1 + c1)] = private_C[5][0];
                    if (nj >= t1 + c1 + 33) {
                      C[(t0 + c0 + 160) * nj + (t1 + c1 + 32)] = private_C[5][1];
                      if (nj >= t1 + c1 + 65) {
                        C[(t0 + c0 + 160) * nj + (t1 + c1 + 64)] = private_C[5][2];
                        if (nj >= t1 + c1 + 97) {
                          C[(t0 + c0 + 160) * nj + (t1 + c1 + 96)] = private_C[5][3];
                          if (nj >= t1 + c1 + 129) {
                            C[(t0 + c0 + 160) * nj + (t1 + c1 + 128)] = private_C[5][4];
                            if (nj >= t1 + c1 + 161) {
                              C[(t0 + c0 + 160) * nj + (t1 + c1 + 160)] = private_C[5][5];
                              if (nj >= t1 + c1 + 193) {
                                C[(t0 + c0 + 160) * nj + (t1 + c1 + 192)] = private_C[5][6];
                                if (nj >= t1 + c1 + 225) {
                                  C[(t0 + c0 + 160) * nj + (t1 + c1 + 224)] = private_C[5][7];
                                  if (nj >= t1 + c1 + 257) {
                                    C[(t0 + c0 + 160) * nj + (t1 + c1 + 256)] = private_C[5][8];
                                    if (nj >= t1 + c1 + 289) {
                                      C[(t0 + c0 + 160) * nj + (t1 + c1 + 288)] = private_C[5][9];
                                      if (nj >= t1 + c1 + 321) {
                                        C[(t0 + c0 + 160) * nj + (t1 + c1 + 320)] = private_C[5][10];
                                        if (nj >= t1 + c1 + 353) {
                                          C[(t0 + c0 + 160) * nj + (t1 + c1 + 352)] = private_C[5][11];
                                          if (nj >= t1 + c1 + 385) {
                                            C[(t0 + c0 + 160) * nj + (t1 + c1 + 384)] = private_C[5][12];
                                            if (nj >= t1 + c1 + 417) {
                                              C[(t0 + c0 + 160) * nj + (t1 + c1 + 416)] = private_C[5][13];
                                              if (nj >= t1 + c1 + 449) {
                                                C[(t0 + c0 + 160) * nj + (t1 + c1 + 448)] = private_C[5][14];
                                                if (nj >= t1 + c1 + 481)
                                                  C[(t0 + c0 + 160) * nj + (t1 + c1 + 480)] = private_C[5][15];
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                    if (ni >= t0 + c0 + 193) {
                      C[(t0 + c0 + 192) * nj + (t1 + c1)] = private_C[6][0];
                      if (nj >= t1 + c1 + 33) {
                        C[(t0 + c0 + 192) * nj + (t1 + c1 + 32)] = private_C[6][1];
                        if (nj >= t1 + c1 + 65) {
                          C[(t0 + c0 + 192) * nj + (t1 + c1 + 64)] = private_C[6][2];
                          if (nj >= t1 + c1 + 97) {
                            C[(t0 + c0 + 192) * nj + (t1 + c1 + 96)] = private_C[6][3];
                            if (nj >= t1 + c1 + 129) {
                              C[(t0 + c0 + 192) * nj + (t1 + c1 + 128)] = private_C[6][4];
                              if (nj >= t1 + c1 + 161) {
                                C[(t0 + c0 + 192) * nj + (t1 + c1 + 160)] = private_C[6][5];
                                if (nj >= t1 + c1 + 193) {
                                  C[(t0 + c0 + 192) * nj + (t1 + c1 + 192)] = private_C[6][6];
                                  if (nj >= t1 + c1 + 225) {
                                    C[(t0 + c0 + 192) * nj + (t1 + c1 + 224)] = private_C[6][7];
                                    if (nj >= t1 + c1 + 257) {
                                      C[(t0 + c0 + 192) * nj + (t1 + c1 + 256)] = private_C[6][8];
                                      if (nj >= t1 + c1 + 289) {
                                        C[(t0 + c0 + 192) * nj + (t1 + c1 + 288)] = private_C[6][9];
                                        if (nj >= t1 + c1 + 321) {
                                          C[(t0 + c0 + 192) * nj + (t1 + c1 + 320)] = private_C[6][10];
                                          if (nj >= t1 + c1 + 353) {
                                            C[(t0 + c0 + 192) * nj + (t1 + c1 + 352)] = private_C[6][11];
                                            if (nj >= t1 + c1 + 385) {
                                              C[(t0 + c0 + 192) * nj + (t1 + c1 + 384)] = private_C[6][12];
                                              if (nj >= t1 + c1 + 417) {
                                                C[(t0 + c0 + 192) * nj + (t1 + c1 + 416)] = private_C[6][13];
                                                if (nj >= t1 + c1 + 449) {
                                                  C[(t0 + c0 + 192) * nj + (t1 + c1 + 448)] = private_C[6][14];
                                                  if (nj >= t1 + c1 + 481)
                                                    C[(t0 + c0 + 192) * nj + (t1 + c1 + 480)] = private_C[6][15];
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                      if (ni >= t0 + c0 + 225) {
                        C[(t0 + c0 + 224) * nj + (t1 + c1)] = private_C[7][0];
                        if (nj >= t1 + c1 + 33) {
                          C[(t0 + c0 + 224) * nj + (t1 + c1 + 32)] = private_C[7][1];
                          if (nj >= t1 + c1 + 65) {
                            C[(t0 + c0 + 224) * nj + (t1 + c1 + 64)] = private_C[7][2];
                            if (nj >= t1 + c1 + 97) {
                              C[(t0 + c0 + 224) * nj + (t1 + c1 + 96)] = private_C[7][3];
                              if (nj >= t1 + c1 + 129) {
                                C[(t0 + c0 + 224) * nj + (t1 + c1 + 128)] = private_C[7][4];
                                if (nj >= t1 + c1 + 161) {
                                  C[(t0 + c0 + 224) * nj + (t1 + c1 + 160)] = private_C[7][5];
                                  if (nj >= t1 + c1 + 193) {
                                    C[(t0 + c0 + 224) * nj + (t1 + c1 + 192)] = private_C[7][6];
                                    if (nj >= t1 + c1 + 225) {
                                      C[(t0 + c0 + 224) * nj + (t1 + c1 + 224)] = private_C[7][7];
                                      if (nj >= t1 + c1 + 257) {
                                        C[(t0 + c0 + 224) * nj + (t1 + c1 + 256)] = private_C[7][8];
                                        if (nj >= t1 + c1 + 289) {
                                          C[(t0 + c0 + 224) * nj + (t1 + c1 + 288)] = private_C[7][9];
                                          if (nj >= t1 + c1 + 321) {
                                            C[(t0 + c0 + 224) * nj + (t1 + c1 + 320)] = private_C[7][10];
                                            if (nj >= t1 + c1 + 353) {
                                              C[(t0 + c0 + 224) * nj + (t1 + c1 + 352)] = private_C[7][11];
                                              if (nj >= t1 + c1 + 385) {
                                                C[(t0 + c0 + 224) * nj + (t1 + c1 + 384)] = private_C[7][12];
                                                if (nj >= t1 + c1 + 417) {
                                                  C[(t0 + c0 + 224) * nj + (t1 + c1 + 416)] = private_C[7][13];
                                                  if (nj >= t1 + c1 + 449) {
                                                    C[(t0 + c0 + 224) * nj + (t1 + c1 + 448)] = private_C[7][14];
                                                    if (nj >= t1 + c1 + 481)
                                                      C[(t0 + c0 + 224) * nj + (t1 + c1 + 480)] = private_C[7][15];
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                        if (ni >= t0 + c0 + 257) {
                          C[(t0 + c0 + 256) * nj + (t1 + c1)] = private_C[8][0];
                          if (nj >= t1 + c1 + 33) {
                            C[(t0 + c0 + 256) * nj + (t1 + c1 + 32)] = private_C[8][1];
                            if (nj >= t1 + c1 + 65) {
                              C[(t0 + c0 + 256) * nj + (t1 + c1 + 64)] = private_C[8][2];
                              if (nj >= t1 + c1 + 97) {
                                C[(t0 + c0 + 256) * nj + (t1 + c1 + 96)] = private_C[8][3];
                                if (nj >= t1 + c1 + 129) {
                                  C[(t0 + c0 + 256) * nj + (t1 + c1 + 128)] = private_C[8][4];
                                  if (nj >= t1 + c1 + 161) {
                                    C[(t0 + c0 + 256) * nj + (t1 + c1 + 160)] = private_C[8][5];
                                    if (nj >= t1 + c1 + 193) {
                                      C[(t0 + c0 + 256) * nj + (t1 + c1 + 192)] = private_C[8][6];
                                      if (nj >= t1 + c1 + 225) {
                                        C[(t0 + c0 + 256) * nj + (t1 + c1 + 224)] = private_C[8][7];
                                        if (nj >= t1 + c1 + 257) {
                                          C[(t0 + c0 + 256) * nj + (t1 + c1 + 256)] = private_C[8][8];
                                          if (nj >= t1 + c1 + 289) {
                                            C[(t0 + c0 + 256) * nj + (t1 + c1 + 288)] = private_C[8][9];
                                            if (nj >= t1 + c1 + 321) {
                                              C[(t0 + c0 + 256) * nj + (t1 + c1 + 320)] = private_C[8][10];
                                              if (nj >= t1 + c1 + 353) {
                                                C[(t0 + c0 + 256) * nj + (t1 + c1 + 352)] = private_C[8][11];
                                                if (nj >= t1 + c1 + 385) {
                                                  C[(t0 + c0 + 256) * nj + (t1 + c1 + 384)] = private_C[8][12];
                                                  if (nj >= t1 + c1 + 417) {
                                                    C[(t0 + c0 + 256) * nj + (t1 + c1 + 416)] = private_C[8][13];
                                                    if (nj >= t1 + c1 + 449) {
                                                      C[(t0 + c0 + 256) * nj + (t1 + c1 + 448)] = private_C[8][14];
                                                      if (nj >= t1 + c1 + 481)
                                                        C[(t0 + c0 + 256) * nj + (t1 + c1 + 480)] = private_C[8][15];
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                          if (ni >= t0 + c0 + 289) {
                            C[(t0 + c0 + 288) * nj + (t1 + c1)] = private_C[9][0];
                            if (nj >= t1 + c1 + 33) {
                              C[(t0 + c0 + 288) * nj + (t1 + c1 + 32)] = private_C[9][1];
                              if (nj >= t1 + c1 + 65) {
                                C[(t0 + c0 + 288) * nj + (t1 + c1 + 64)] = private_C[9][2];
                                if (nj >= t1 + c1 + 97) {
                                  C[(t0 + c0 + 288) * nj + (t1 + c1 + 96)] = private_C[9][3];
                                  if (nj >= t1 + c1 + 129) {
                                    C[(t0 + c0 + 288) * nj + (t1 + c1 + 128)] = private_C[9][4];
                                    if (nj >= t1 + c1 + 161) {
                                      C[(t0 + c0 + 288) * nj + (t1 + c1 + 160)] = private_C[9][5];
                                      if (nj >= t1 + c1 + 193) {
                                        C[(t0 + c0 + 288) * nj + (t1 + c1 + 192)] = private_C[9][6];
                                        if (nj >= t1 + c1 + 225) {
                                          C[(t0 + c0 + 288) * nj + (t1 + c1 + 224)] = private_C[9][7];
                                          if (nj >= t1 + c1 + 257) {
                                            C[(t0 + c0 + 288) * nj + (t1 + c1 + 256)] = private_C[9][8];
                                            if (nj >= t1 + c1 + 289) {
                                              C[(t0 + c0 + 288) * nj + (t1 + c1 + 288)] = private_C[9][9];
                                              if (nj >= t1 + c1 + 321) {
                                                C[(t0 + c0 + 288) * nj + (t1 + c1 + 320)] = private_C[9][10];
                                                if (nj >= t1 + c1 + 353) {
                                                  C[(t0 + c0 + 288) * nj + (t1 + c1 + 352)] = private_C[9][11];
                                                  if (nj >= t1 + c1 + 385) {
                                                    C[(t0 + c0 + 288) * nj + (t1 + c1 + 384)] = private_C[9][12];
                                                    if (nj >= t1 + c1 + 417) {
                                                      C[(t0 + c0 + 288) * nj + (t1 + c1 + 416)] = private_C[9][13];
                                                      if (nj >= t1 + c1 + 449) {
                                                        C[(t0 + c0 + 288) * nj + (t1 + c1 + 448)] = private_C[9][14];
                                                        if (nj >= t1 + c1 + 481)
                                                          C[(t0 + c0 + 288) * nj + (t1 + c1 + 480)] = private_C[9][15];
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                            if (ni >= t0 + c0 + 321) {
                              C[(t0 + c0 + 320) * nj + (t1 + c1)] = private_C[10][0];
                              if (nj >= t1 + c1 + 33) {
                                C[(t0 + c0 + 320) * nj + (t1 + c1 + 32)] = private_C[10][1];
                                if (nj >= t1 + c1 + 65) {
                                  C[(t0 + c0 + 320) * nj + (t1 + c1 + 64)] = private_C[10][2];
                                  if (nj >= t1 + c1 + 97) {
                                    C[(t0 + c0 + 320) * nj + (t1 + c1 + 96)] = private_C[10][3];
                                    if (nj >= t1 + c1 + 129) {
                                      C[(t0 + c0 + 320) * nj + (t1 + c1 + 128)] = private_C[10][4];
                                      if (nj >= t1 + c1 + 161) {
                                        C[(t0 + c0 + 320) * nj + (t1 + c1 + 160)] = private_C[10][5];
                                        if (nj >= t1 + c1 + 193) {
                                          C[(t0 + c0 + 320) * nj + (t1 + c1 + 192)] = private_C[10][6];
                                          if (nj >= t1 + c1 + 225) {
                                            C[(t0 + c0 + 320) * nj + (t1 + c1 + 224)] = private_C[10][7];
                                            if (nj >= t1 + c1 + 257) {
                                              C[(t0 + c0 + 320) * nj + (t1 + c1 + 256)] = private_C[10][8];
                                              if (nj >= t1 + c1 + 289) {
                                                C[(t0 + c0 + 320) * nj + (t1 + c1 + 288)] = private_C[10][9];
                                                if (nj >= t1 + c1 + 321) {
                                                  C[(t0 + c0 + 320) * nj + (t1 + c1 + 320)] = private_C[10][10];
                                                  if (nj >= t1 + c1 + 353) {
                                                    C[(t0 + c0 + 320) * nj + (t1 + c1 + 352)] = private_C[10][11];
                                                    if (nj >= t1 + c1 + 385) {
                                                      C[(t0 + c0 + 320) * nj + (t1 + c1 + 384)] = private_C[10][12];
                                                      if (nj >= t1 + c1 + 417) {
                                                        C[(t0 + c0 + 320) * nj + (t1 + c1 + 416)] = private_C[10][13];
                                                        if (nj >= t1 + c1 + 449) {
                                                          C[(t0 + c0 + 320) * nj + (t1 + c1 + 448)] = private_C[10][14];
                                                          if (nj >= t1 + c1 + 481)
                                                            C[(t0 + c0 + 320) * nj + (t1 + c1 + 480)] = private_C[10][15];
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                              if (ni >= t0 + c0 + 353) {
                                C[(t0 + c0 + 352) * nj + (t1 + c1)] = private_C[11][0];
                                if (nj >= t1 + c1 + 33) {
                                  C[(t0 + c0 + 352) * nj + (t1 + c1 + 32)] = private_C[11][1];
                                  if (nj >= t1 + c1 + 65) {
                                    C[(t0 + c0 + 352) * nj + (t1 + c1 + 64)] = private_C[11][2];
                                    if (nj >= t1 + c1 + 97) {
                                      C[(t0 + c0 + 352) * nj + (t1 + c1 + 96)] = private_C[11][3];
                                      if (nj >= t1 + c1 + 129) {
                                        C[(t0 + c0 + 352) * nj + (t1 + c1 + 128)] = private_C[11][4];
                                        if (nj >= t1 + c1 + 161) {
                                          C[(t0 + c0 + 352) * nj + (t1 + c1 + 160)] = private_C[11][5];
                                          if (nj >= t1 + c1 + 193) {
                                            C[(t0 + c0 + 352) * nj + (t1 + c1 + 192)] = private_C[11][6];
                                            if (nj >= t1 + c1 + 225) {
                                              C[(t0 + c0 + 352) * nj + (t1 + c1 + 224)] = private_C[11][7];
                                              if (nj >= t1 + c1 + 257) {
                                                C[(t0 + c0 + 352) * nj + (t1 + c1 + 256)] = private_C[11][8];
                                                if (nj >= t1 + c1 + 289) {
                                                  C[(t0 + c0 + 352) * nj + (t1 + c1 + 288)] = private_C[11][9];
                                                  if (nj >= t1 + c1 + 321) {
                                                    C[(t0 + c0 + 352) * nj + (t1 + c1 + 320)] = private_C[11][10];
                                                    if (nj >= t1 + c1 + 353) {
                                                      C[(t0 + c0 + 352) * nj + (t1 + c1 + 352)] = private_C[11][11];
                                                      if (nj >= t1 + c1 + 385) {
                                                        C[(t0 + c0 + 352) * nj + (t1 + c1 + 384)] = private_C[11][12];
                                                        if (nj >= t1 + c1 + 417) {
                                                          C[(t0 + c0 + 352) * nj + (t1 + c1 + 416)] = private_C[11][13];
                                                          if (nj >= t1 + c1 + 449) {
                                                            C[(t0 + c0 + 352) * nj + (t1 + c1 + 448)] = private_C[11][14];
                                                            if (nj >= t1 + c1 + 481)
                                                              C[(t0 + c0 + 352) * nj + (t1 + c1 + 480)] = private_C[11][15];
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                                if (ni >= t0 + c0 + 385) {
                                  C[(t0 + c0 + 384) * nj + (t1 + c1)] = private_C[12][0];
                                  if (nj >= t1 + c1 + 33) {
                                    C[(t0 + c0 + 384) * nj + (t1 + c1 + 32)] = private_C[12][1];
                                    if (nj >= t1 + c1 + 65) {
                                      C[(t0 + c0 + 384) * nj + (t1 + c1 + 64)] = private_C[12][2];
                                      if (nj >= t1 + c1 + 97) {
                                        C[(t0 + c0 + 384) * nj + (t1 + c1 + 96)] = private_C[12][3];
                                        if (nj >= t1 + c1 + 129) {
                                          C[(t0 + c0 + 384) * nj + (t1 + c1 + 128)] = private_C[12][4];
                                          if (nj >= t1 + c1 + 161) {
                                            C[(t0 + c0 + 384) * nj + (t1 + c1 + 160)] = private_C[12][5];
                                            if (nj >= t1 + c1 + 193) {
                                              C[(t0 + c0 + 384) * nj + (t1 + c1 + 192)] = private_C[12][6];
                                              if (nj >= t1 + c1 + 225) {
                                                C[(t0 + c0 + 384) * nj + (t1 + c1 + 224)] = private_C[12][7];
                                                if (nj >= t1 + c1 + 257) {
                                                  C[(t0 + c0 + 384) * nj + (t1 + c1 + 256)] = private_C[12][8];
                                                  if (nj >= t1 + c1 + 289) {
                                                    C[(t0 + c0 + 384) * nj + (t1 + c1 + 288)] = private_C[12][9];
                                                    if (nj >= t1 + c1 + 321) {
                                                      C[(t0 + c0 + 384) * nj + (t1 + c1 + 320)] = private_C[12][10];
                                                      if (nj >= t1 + c1 + 353) {
                                                        C[(t0 + c0 + 384) * nj + (t1 + c1 + 352)] = private_C[12][11];
                                                        if (nj >= t1 + c1 + 385) {
                                                          C[(t0 + c0 + 384) * nj + (t1 + c1 + 384)] = private_C[12][12];
                                                          if (nj >= t1 + c1 + 417) {
                                                            C[(t0 + c0 + 384) * nj + (t1 + c1 + 416)] = private_C[12][13];
                                                            if (nj >= t1 + c1 + 449) {
                                                              C[(t0 + c0 + 384) * nj + (t1 + c1 + 448)] = private_C[12][14];
                                                              if (nj >= t1 + c1 + 481)
                                                                C[(t0 + c0 + 384) * nj + (t1 + c1 + 480)] = private_C[12][15];
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                  if (ni >= t0 + c0 + 417) {
                                    C[(t0 + c0 + 416) * nj + (t1 + c1)] = private_C[13][0];
                                    if (nj >= t1 + c1 + 33) {
                                      C[(t0 + c0 + 416) * nj + (t1 + c1 + 32)] = private_C[13][1];
                                      if (nj >= t1 + c1 + 65) {
                                        C[(t0 + c0 + 416) * nj + (t1 + c1 + 64)] = private_C[13][2];
                                        if (nj >= t1 + c1 + 97) {
                                          C[(t0 + c0 + 416) * nj + (t1 + c1 + 96)] = private_C[13][3];
                                          if (nj >= t1 + c1 + 129) {
                                            C[(t0 + c0 + 416) * nj + (t1 + c1 + 128)] = private_C[13][4];
                                            if (nj >= t1 + c1 + 161) {
                                              C[(t0 + c0 + 416) * nj + (t1 + c1 + 160)] = private_C[13][5];
                                              if (nj >= t1 + c1 + 193) {
                                                C[(t0 + c0 + 416) * nj + (t1 + c1 + 192)] = private_C[13][6];
                                                if (nj >= t1 + c1 + 225) {
                                                  C[(t0 + c0 + 416) * nj + (t1 + c1 + 224)] = private_C[13][7];
                                                  if (nj >= t1 + c1 + 257) {
                                                    C[(t0 + c0 + 416) * nj + (t1 + c1 + 256)] = private_C[13][8];
                                                    if (nj >= t1 + c1 + 289) {
                                                      C[(t0 + c0 + 416) * nj + (t1 + c1 + 288)] = private_C[13][9];
                                                      if (nj >= t1 + c1 + 321) {
                                                        C[(t0 + c0 + 416) * nj + (t1 + c1 + 320)] = private_C[13][10];
                                                        if (nj >= t1 + c1 + 353) {
                                                          C[(t0 + c0 + 416) * nj + (t1 + c1 + 352)] = private_C[13][11];
                                                          if (nj >= t1 + c1 + 385) {
                                                            C[(t0 + c0 + 416) * nj + (t1 + c1 + 384)] = private_C[13][12];
                                                            if (nj >= t1 + c1 + 417) {
                                                              C[(t0 + c0 + 416) * nj + (t1 + c1 + 416)] = private_C[13][13];
                                                              if (nj >= t1 + c1 + 449) {
                                                                C[(t0 + c0 + 416) * nj + (t1 + c1 + 448)] = private_C[13][14];
                                                                if (nj >= t1 + c1 + 481)
                                                                  C[(t0 + c0 + 416) * nj + (t1 + c1 + 480)] = private_C[13][15];
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                    if (ni >= t0 + c0 + 449) {
                                      C[(t0 + c0 + 448) * nj + (t1 + c1)] = private_C[14][0];
                                      if (nj >= t1 + c1 + 33) {
                                        C[(t0 + c0 + 448) * nj + (t1 + c1 + 32)] = private_C[14][1];
                                        if (nj >= t1 + c1 + 65) {
                                          C[(t0 + c0 + 448) * nj + (t1 + c1 + 64)] = private_C[14][2];
                                          if (nj >= t1 + c1 + 97) {
                                            C[(t0 + c0 + 448) * nj + (t1 + c1 + 96)] = private_C[14][3];
                                            if (nj >= t1 + c1 + 129) {
                                              C[(t0 + c0 + 448) * nj + (t1 + c1 + 128)] = private_C[14][4];
                                              if (nj >= t1 + c1 + 161) {
                                                C[(t0 + c0 + 448) * nj + (t1 + c1 + 160)] = private_C[14][5];
                                                if (nj >= t1 + c1 + 193) {
                                                  C[(t0 + c0 + 448) * nj + (t1 + c1 + 192)] = private_C[14][6];
                                                  if (nj >= t1 + c1 + 225) {
                                                    C[(t0 + c0 + 448) * nj + (t1 + c1 + 224)] = private_C[14][7];
                                                    if (nj >= t1 + c1 + 257) {
                                                      C[(t0 + c0 + 448) * nj + (t1 + c1 + 256)] = private_C[14][8];
                                                      if (nj >= t1 + c1 + 289) {
                                                        C[(t0 + c0 + 448) * nj + (t1 + c1 + 288)] = private_C[14][9];
                                                        if (nj >= t1 + c1 + 321) {
                                                          C[(t0 + c0 + 448) * nj + (t1 + c1 + 320)] = private_C[14][10];
                                                          if (nj >= t1 + c1 + 353) {
                                                            C[(t0 + c0 + 448) * nj + (t1 + c1 + 352)] = private_C[14][11];
                                                            if (nj >= t1 + c1 + 385) {
                                                              C[(t0 + c0 + 448) * nj + (t1 + c1 + 384)] = private_C[14][12];
                                                              if (nj >= t1 + c1 + 417) {
                                                                C[(t0 + c0 + 448) * nj + (t1 + c1 + 416)] = private_C[14][13];
                                                                if (nj >= t1 + c1 + 449) {
                                                                  C[(t0 + c0 + 448) * nj + (t1 + c1 + 448)] = private_C[14][14];
                                                                  if (nj >= t1 + c1 + 481)
                                                                    C[(t0 + c0 + 448) * nj + (t1 + c1 + 480)] = private_C[14][15];
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                      if (ni >= t0 + c0 + 481) {
                                        C[(t0 + c0 + 480) * nj + (t1 + c1)] = private_C[15][0];
                                        if (nj >= t1 + c1 + 33) {
                                          C[(t0 + c0 + 480) * nj + (t1 + c1 + 32)] = private_C[15][1];
                                          if (nj >= t1 + c1 + 65) {
                                            C[(t0 + c0 + 480) * nj + (t1 + c1 + 64)] = private_C[15][2];
                                            if (nj >= t1 + c1 + 97) {
                                              C[(t0 + c0 + 480) * nj + (t1 + c1 + 96)] = private_C[15][3];
                                              if (nj >= t1 + c1 + 129) {
                                                C[(t0 + c0 + 480) * nj + (t1 + c1 + 128)] = private_C[15][4];
                                                if (nj >= t1 + c1 + 161) {
                                                  C[(t0 + c0 + 480) * nj + (t1 + c1 + 160)] = private_C[15][5];
                                                  if (nj >= t1 + c1 + 193) {
                                                    C[(t0 + c0 + 480) * nj + (t1 + c1 + 192)] = private_C[15][6];
                                                    if (nj >= t1 + c1 + 225) {
                                                      C[(t0 + c0 + 480) * nj + (t1 + c1 + 224)] = private_C[15][7];
                                                      if (nj >= t1 + c1 + 257) {
                                                        C[(t0 + c0 + 480) * nj + (t1 + c1 + 256)] = private_C[15][8];
                                                        if (nj >= t1 + c1 + 289) {
                                                          C[(t0 + c0 + 480) * nj + (t1 + c1 + 288)] = private_C[15][9];
                                                          if (nj >= t1 + c1 + 321) {
                                                            C[(t0 + c0 + 480) * nj + (t1 + c1 + 320)] = private_C[15][10];
                                                            if (nj >= t1 + c1 + 353) {
                                                              C[(t0 + c0 + 480) * nj + (t1 + c1 + 352)] = private_C[15][11];
                                                              if (nj >= t1 + c1 + 385) {
                                                                C[(t0 + c0 + 480) * nj + (t1 + c1 + 384)] = private_C[15][12];
                                                                if (nj >= t1 + c1 + 417) {
                                                                  C[(t0 + c0 + 480) * nj + (t1 + c1 + 416)] = private_C[15][13];
                                                                  if (nj >= t1 + c1 + 449) {
                                                                    C[(t0 + c0 + 480) * nj + (t1 + c1 + 448)] = private_C[15][14];
                                                                    if (nj >= t1 + c1 + 481)
                                                                      C[(t0 + c0 + 480) * nj + (t1 + c1 + 480)] = private_C[15][15];
                                                                  }
                                                                }
                                                              }
                                                            }
                                                          }
                                                        }
                                                      }
                                                    }
                                                  }
                                                }
                                              }
                                            }
                                          }
                                        }
                                      }
                                    }
                                  }
                                }
                              }
                            }
                          }
                        }
                      }
                    }
                  }
                }
              }
            }
          }
        }
        __syncthreads();
      }
}
