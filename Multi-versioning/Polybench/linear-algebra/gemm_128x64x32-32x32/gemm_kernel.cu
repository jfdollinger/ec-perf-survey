#include "hip/hip_runtime.h"
#include "gemm_kernel.hu"
__global__ void kernel0(double *A, double *B, double *C, double alpha, double beta, int nj, int nk, int ni)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    double private_C[4][2];

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 128 * b0; c0 < ni; c0 += 32768)
      for (int c1 = 64 * b1; c1 < nj; c1 += 16384) {
        if (ni >= t0 + c0 + 1 && nj >= t1 + c1 + 1) {
          private_C[0][0] = C[(t0 + c0) * nj + (t1 + c1)];
          if (nj >= t1 + c1 + 33)
            private_C[0][1] = C[(t0 + c0) * nj + (t1 + c1 + 32)];
          if (ni >= t0 + c0 + 33) {
            private_C[1][0] = C[(t0 + c0 + 32) * nj + (t1 + c1)];
            if (nj >= t1 + c1 + 33)
              private_C[1][1] = C[(t0 + c0 + 32) * nj + (t1 + c1 + 32)];
            if (ni >= t0 + c0 + 65) {
              private_C[2][0] = C[(t0 + c0 + 64) * nj + (t1 + c1)];
              if (nj >= t1 + c1 + 33)
                private_C[2][1] = C[(t0 + c0 + 64) * nj + (t1 + c1 + 32)];
              if (ni >= t0 + c0 + 97) {
                private_C[3][0] = C[(t0 + c0 + 96) * nj + (t1 + c1)];
                if (nj >= t1 + c1 + 33)
                  private_C[3][1] = C[(t0 + c0 + 96) * nj + (t1 + c1 + 32)];
              }
            }
          }
          private_C[0][0] *= beta;
          if (nj >= t1 + c1 + 33)
            private_C[0][1] *= beta;
          if (ni >= t0 + c0 + 33) {
            private_C[1][0] *= beta;
            if (nj >= t1 + c1 + 33)
              private_C[1][1] *= beta;
            if (ni >= t0 + c0 + 65) {
              private_C[2][0] *= beta;
              if (nj >= t1 + c1 + 33)
                private_C[2][1] *= beta;
              if (ni >= t0 + c0 + 97) {
                private_C[3][0] *= beta;
                if (nj >= t1 + c1 + 33)
                  private_C[3][1] *= beta;
              }
            }
          }
          for (int c2 = 0; c2 < nk; c2 += 32)
            for (int c3 = 0; c3 <= ppcg_min(31, nk - c2 - 1); c3 += 1) {
              private_C[0][0] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
              if (nj >= t1 + c1 + 33)
                private_C[0][1] += ((alpha * A[(t0 + c0) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
              if (ni >= t0 + c0 + 33) {
                private_C[1][0] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                if (nj >= t1 + c1 + 33)
                  private_C[1][1] += ((alpha * A[(t0 + c0 + 32) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                if (ni >= t0 + c0 + 65) {
                  private_C[2][0] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                  if (nj >= t1 + c1 + 33)
                    private_C[2][1] += ((alpha * A[(t0 + c0 + 64) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                  if (ni >= t0 + c0 + 97) {
                    private_C[3][0] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1)]);
                    if (nj >= t1 + c1 + 33)
                      private_C[3][1] += ((alpha * A[(t0 + c0 + 96) * nk + (c2 + c3)]) * B[(c2 + c3) * nj + (t1 + c1 + 32)]);
                  }
                }
              }
            }
          C[(t0 + c0) * nj + (t1 + c1)] = private_C[0][0];
          if (nj >= t1 + c1 + 33)
            C[(t0 + c0) * nj + (t1 + c1 + 32)] = private_C[0][1];
          if (ni >= t0 + c0 + 33) {
            C[(t0 + c0 + 32) * nj + (t1 + c1)] = private_C[1][0];
            if (nj >= t1 + c1 + 33)
              C[(t0 + c0 + 32) * nj + (t1 + c1 + 32)] = private_C[1][1];
            if (ni >= t0 + c0 + 65) {
              C[(t0 + c0 + 64) * nj + (t1 + c1)] = private_C[2][0];
              if (nj >= t1 + c1 + 33)
                C[(t0 + c0 + 64) * nj + (t1 + c1 + 32)] = private_C[2][1];
              if (ni >= t0 + c0 + 97) {
                C[(t0 + c0 + 96) * nj + (t1 + c1)] = private_C[3][0];
                if (nj >= t1 + c1 + 33)
                  C[(t0 + c0 + 96) * nj + (t1 + c1 + 32)] = private_C[3][1];
              }
            }
          }
        }
        __syncthreads();
      }
}
